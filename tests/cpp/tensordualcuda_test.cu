#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cassert>
#include "../../src/cpp/cudatensordense.cu"
// Include your VectorBool implementation here
using namespace janus;


template <typename T>
thrust::complex<T> generate_random() {
    return thrust::complex<T>(rand() % 100, rand() % 100);
}

//Generate N random complex nunbers
template <typename T>
std::vector<thrust::complex<T>> generate_random_vector(int N) {
    std::vector<thrust::complex<T>> vec(N);
    for (int i = 0; i < N; i++) {
        vec[i] = generate_random<T>();
    }
    return vec;
}

class VectorBoolTest : public ::testing::Test {
public:
    VectorBool input, output, subvector;

    void SetUp() override {
        // Initialize vectors
        int size = 10;
        hipError_t err;

        // Allocate and initialize `input`
        input.size_ = size;
        err = hipMalloc(&input.data_, size * sizeof(bool));
        ASSERT_EQ(err, hipSuccess) << "hipMalloc failed for input: " << hipGetErrorString(err);

        bool host_data[10] = {true, false, true, false, true, false, true, false, true, false};
        err = hipMemcpy(input.data_, host_data, size * sizeof(bool), hipMemcpyHostToDevice);
        ASSERT_EQ(err, hipSuccess) << "hipMemcpy failed for input: " << hipGetErrorString(err);

        // Allocate `output`
        output.size_ = size;
        err = hipMalloc(&output.data_, size * sizeof(bool));
        ASSERT_EQ(err, hipSuccess) << "hipMalloc failed for output: " << hipGetErrorString(err);

        // Allocate and initialize `subvector`
        subvector.size_ = size;
        err = hipMalloc(&subvector.data_, size * sizeof(bool));
        ASSERT_EQ(err, hipSuccess) << "hipMalloc failed for subvector: " << hipGetErrorString(err);
    }

    void TearDown() override {
        if (input.data_ != nullptr) hipFree(input.data_);
        if (output.data_ != nullptr) hipFree(output.data_);
        if (subvector.data_ != nullptr) hipFree(subvector.data_);
    }
};

// Test case for boolIndexGet
TEST_F(VectorBoolTest, BoolIndexGetTest) {
    int start = 2, end = 5;
    int range = end - start;

    // Launch kernel

    boolIndexGetKernel<<<1, 10>>>(input.data_, start, end, output.data_);
    hipDeviceSynchronize();

    // Check for kernel errors
    hipError_t err = hipGetLastError();
    EXPECT_EQ(err, hipSuccess) << "Kernel launch failed: " << hipGetErrorString(err);

    // Copy result back to host
    bool result[range];
    hipMemcpy(result, output.data_, range * sizeof(bool), hipMemcpyDeviceToHost);

    // Expected values
    bool expected[] = {true, false, true};
    for (int i = 0; i < range; i++) {
        EXPECT_EQ(result[i], expected[i]) << "Mismatch at index " << i;
    }
}

// Test case for indexPut
TEST_F(VectorBoolTest, IndexPutTest) {
    int start = 2, end = 5;
    int range = end - start;

    // Prepare subvector on the host
    bool host_subvector[] = {false, true, false};
    hipMemcpy(subvector.data_, host_subvector, range * sizeof(bool), hipMemcpyHostToDevice);

    // Launch kernel
    boolIndexPutKernel<<<1, range>>>(input.data_, start, end, subvector.data_);
    hipDeviceSynchronize();

    // Copy result back to host
    bool result[10];
    hipMemcpy(result, input.data_, 10 * sizeof(bool), hipMemcpyDeviceToHost);

    // Expected values
    bool expected[] = {true, false, false, true, false, false, true, false, true, false};
    for (int i = 0; i < 10; i++) {
        EXPECT_EQ(result[i], expected[i]) << "Mismatch at index " << i;
    }
}


// Test fixture class
class VectorAddTest : public ::testing::Test {
protected:
    const int size = 5;
    thrust::complex<float> h_a[5] = {
        {1.0f, 2.0f}, {3.0f, 4.0f}, {5.0f, 6.0f}, {7.0f, 8.0f}, {9.0f, 10.0f}};
    thrust::complex<float> h_b[5] = {
        {10.0f, 20.0f}, {30.0f, 40.0f}, {50.0f, 60.0f}, {70.0f, 80.0f}, {90.0f, 100.0f}};
    thrust::complex<float> h_result[5];

    thrust::complex<float> *d_a, *d_b, *d_result;

    void SetUp() override {
        hipMalloc(&d_a, size * sizeof(thrust::complex<float>));
        hipMalloc(&d_b, size * sizeof(thrust::complex<float>));
        hipMalloc(&d_result, size * sizeof(thrust::complex<float>));

        hipMemcpy(d_a, h_a, size * sizeof(thrust::complex<float>), hipMemcpyHostToDevice);
        hipMemcpy(d_b, h_b, size * sizeof(thrust::complex<float>), hipMemcpyHostToDevice);
    }

    void TearDown() override {
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_result);
    }
};

// Test case
TEST_F(VectorAddTest, ElementwiseAddition) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    VectorElementwiseAddKernel<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, size, d_result);
    hipMemcpy(h_result, d_result, size * sizeof(thrust::complex<float>), hipMemcpyDeviceToHost);

    for (int i = 0; i < size; ++i) {
        thrust::complex<float> expected = h_a[i] + h_b[i];
        ASSERT_EQ(h_result[i], expected) << "Mismatch at index " << i;
    }
}




// Test fixture class
class VectorTest : public ::testing::Test {
protected:
    const int size = 5;
    thrust::complex<float> h_a[5] = {
        {1.0f, 2.0f}, {-3.0f, 4.0f}, {5.0f, -6.0f}, {-7.0f, -8.0f}, {0.0f, 10.0f}};
    thrust::complex<float> h_b[5] = {
        {10.0f, 20.0f}, {-30.0f, 40.0f}, {50.0f, -60.0f}, {-70.0f, 80.0f}, {90.0f, -100.0f}};
    thrust::complex<float> h_result[5];

    thrust::complex<float> *d_a, *d_b, *d_result;

    void SetUp() override {
        hipMalloc(&d_a, size * sizeof(thrust::complex<float>));
        hipMalloc(&d_b, size * sizeof(thrust::complex<float>));
        hipMalloc(&d_result, size * sizeof(thrust::complex<float>));

        hipMemcpy(d_a, h_a, size * sizeof(thrust::complex<float>), hipMemcpyHostToDevice);
        hipMemcpy(d_b, h_b, size * sizeof(thrust::complex<float>), hipMemcpyHostToDevice);
    }

    void TearDown() override {
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_result);
    }
};

// Test case for VectorElementwiseAdd
TEST_F(VectorTest, ElementwiseAddition) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    VectorElementwiseAddKernel<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, size, d_result);
    hipMemcpy(h_result, d_result, size * sizeof(thrust::complex<float>), hipMemcpyDeviceToHost);

    for (int i = 0; i < size; ++i) {
        thrust::complex<float> expected = h_a[i] + h_b[i];
        ASSERT_EQ(h_result[i], expected) << "Mismatch at index " << i;
    }
}

// Test case for VectorSigncond
TEST_F(VectorTest, SignCondition) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    VectorSigncondKernel<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, size, d_result, 1.0e-6);
    hipMemcpy(h_result, d_result, size * sizeof(thrust::complex<float>), hipMemcpyDeviceToHost);

    for (int i = 0; i < size; ++i) {
        thrust::complex<float> expected;
        int a_sign = (fabs(h_a[i].real()) >= 1.0e-6) ? (h_a[i].real() >= 0 ? 1 : -1) : 1;
        int b_sign = (fabs(h_b[i].real()) >= 1.0e-6) ? (h_b[i].real() >= 0 ? 1 : -1) : 1;

        if (b_sign >= 0) {
            expected = (a_sign >= 0) ? h_a[i] : -h_a[i];
        } else {
            expected = (a_sign >= 0) ? -h_a[i] : h_a[i];
        }

        ASSERT_EQ(h_result[i], expected) << "Mismatch at index " << i;
    }
}


// Test case for VectorSquare
TEST_F(VectorTest, Square) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    VectorSquareKernel<<<blocksPerGrid, threadsPerBlock>>>(d_a, size, d_result);
    hipMemcpy(h_result, d_result, size * sizeof(thrust::complex<float>), hipMemcpyDeviceToHost);

    for (int i = 0; i < size; ++i) {
        thrust::complex<float> expected = h_a[i] * h_a[i];
        ASSERT_NEAR(h_result[i].real(), expected.real(), 1.0e-12f) << "Mismatch at index " << i;
        ASSERT_NEAR(h_result[i].imag(), expected.imag(), 1.0e-12f) << "Mismatch at index " << i;
    }
}



// Test case for VectorScalarMultiply
TEST_F(VectorTest, ScalarMultiply) {
    float scalar = 2.0f;
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    VectorScalarMultiplyKernel<<<blocksPerGrid, threadsPerBlock>>>(d_a, scalar, size, d_result);
    hipMemcpy(h_result, d_result, size * sizeof(thrust::complex<float>), hipMemcpyDeviceToHost);

    for (int i = 0; i < size; ++i) {
        thrust::complex<float> expected = h_a[i] * scalar;
        ASSERT_EQ(h_result[i], expected) << "Mismatch at index " << i;
    }
}


// Test case for VectorReciprocal
TEST_F(VectorTest, Reciprocal) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    VectorReciprocalKernel<<<blocksPerGrid, threadsPerBlock>>>(d_a, size, d_result);
    hipMemcpy(h_result, d_result, size * sizeof(thrust::complex<float>), hipMemcpyDeviceToHost);

    for (int i = 0; i < size; ++i) {
        thrust::complex<float> expected = 1.0f / h_a[i];
        ASSERT_NEAR(h_result[i].real(), expected.real(), 1.0e-6) << "Mismatch at index " << i;
        ASSERT_NEAR(h_result[i].imag(), expected.imag(), 1.0e-6) << "Mismatch at index " << i;
    }
}


// Test case for VectorElementwiseMultiply
TEST_F(VectorTest, ElementwiseMultiply) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    VectorElementwiseMultiplyKernel<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, size, d_result);
    hipMemcpy(h_result, d_result, size * sizeof(thrust::complex<float>), hipMemcpyDeviceToHost);

    for (int i = 0; i < size; ++i) {
        thrust::complex<float> expected = h_a[i] * h_b[i];
        ASSERT_EQ(h_result[i], expected) << "Mismatch at index " << i;
    }
}


// Test case for VectorSqrt
TEST_F(VectorTest, ElementwiseSqrt) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    VectorSqrtKernel<<<blocksPerGrid, threadsPerBlock>>>(d_a, size, d_result);
    hipMemcpy(h_result, d_result, size * sizeof(thrust::complex<float>), hipMemcpyDeviceToHost);

    for (int i = 0; i < size; ++i) {
        thrust::complex<float> expected = thrust::sqrt(h_a[i]);
        ASSERT_EQ(h_result[i], expected) << "Mismatch at index " << i;
    }
}

// Test case for VectorPow
TEST_F(VectorTest, ElementwisePow) {
    float power = 2.0f;
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    VectorPowKernel<<<blocksPerGrid, threadsPerBlock>>>(d_a, power, size, d_result);
    hipMemcpy(h_result, d_result, size * sizeof(thrust::complex<float>), hipMemcpyDeviceToHost);

    for (int i = 0; i < size; ++i) {
        thrust::complex<float> expected = pow(h_a[i], power);
        ASSERT_NEAR(h_result[i].real(), expected.real(), 1.0e-5) << "Mismatch at index " << i;
        ASSERT_NEAR(h_result[i].imag(), expected.imag(), 1.0e-5) << "Mismatch at index " << i;
    }
}


// Test case for VectorReduce
TEST_F(VectorTest, Reduce) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    // Allocate device memory for block-level results
    thrust::complex<float>* d_block_results;
    hipMalloc(&d_block_results, blocksPerGrid * sizeof(thrust::complex<float>));

    // Launch the kernel
    VectorReduceKernel<<<blocksPerGrid, threadsPerBlock, threadsPerBlock * sizeof(thrust::complex<float>)>>>(d_a, size, d_block_results);

    // Copy the block-level results to host
    thrust::complex<float> h_block_results[blocksPerGrid];
    hipMemcpy(h_block_results, d_block_results, blocksPerGrid * sizeof(thrust::complex<float>), hipMemcpyDeviceToHost);

    // Final reduction on the host
    thrust::complex<float> final_result(0, 0);
    for (int i = 0; i < blocksPerGrid; ++i) {
        final_result += h_block_results[i];
    }

    // Expected result
    thrust::complex<float> expected(0, 0);
    for (int i = 0; i < size; ++i) {
        expected += h_a[i];
    }

    ASSERT_EQ(final_result, expected) << "Mismatch in reduction result.";

    // Free device memory
    hipFree(d_block_results);
}


// Test case for VectorIndexGet
TEST_F(VectorTest, IndexGet) {
    const int start = 1;
    const int end = 4;
    const int result_size = end - start;

    // Allocate device memory for the result
    thrust::complex<float>* d_index_result;
    hipMalloc(&d_index_result, result_size * sizeof(thrust::complex<float>));

    // Launch the kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    VectorIndexGetKernel<<<blocksPerGrid, threadsPerBlock>>>(d_a, start, end, size, d_index_result);

    // Copy the result to host
    thrust::complex<float> h_index_result[result_size];
    hipMemcpy(h_index_result, d_index_result, result_size * sizeof(thrust::complex<float>), hipMemcpyDeviceToHost);

    // Expected result
    for (int i = 0; i < result_size; ++i) {
        ASSERT_EQ(h_index_result[i], h_a[start + i]) << "Mismatch at index " << i;
    }

    // Free device memory
    hipFree(d_index_result);
}

// Test case for VectorIndexPut
TEST_F(VectorTest, IndexPut) {
    const int start = 1;
    const int end = 4;
    const int result_size = size;

    // Allocate device memory for the result
    thrust::complex<float>* d_index_result;
    hipMalloc(&d_index_result, result_size * sizeof(thrust::complex<float>));

    // Initialize the result array on the device to zeros
    thrust::complex<float> h_zeros[result_size] = {thrust::complex<float>(0, 0)};
    hipMemcpy(d_index_result, h_zeros, result_size * sizeof(thrust::complex<float>), hipMemcpyHostToDevice);

    // Allocate input subarray on the device
    const int input_size = end - start;
    thrust::complex<float> h_input[input_size] = {
        h_a[start], h_a[start + 1], h_a[start + 2]
    };
    thrust::complex<float>* d_input;
    hipMalloc(&d_input, input_size * sizeof(thrust::complex<float>));
    hipMemcpy(d_input, h_input, input_size * sizeof(thrust::complex<float>), hipMemcpyHostToDevice);

    // Launch the kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (result_size + threadsPerBlock - 1) / threadsPerBlock;
    VectorIndexPutKernel<<<blocksPerGrid, threadsPerBlock>>>(d_input, start, end, result_size, d_index_result);

    // Copy the result to host
    hipMemcpy(h_result, d_index_result, result_size * sizeof(thrust::complex<float>), hipMemcpyDeviceToHost);

    // Verify the results
    for (int i = 0; i < result_size; ++i) {
        if (i >= start && i < end) {
            ASSERT_EQ(h_result[i], h_a[i]) << "Mismatch at index " << i;
        } else {
            ASSERT_EQ(h_result[i], thrust::complex<float>(0, 0)) << "Mismatch at index " << i;
        }
    }

    // Free device memory
    hipFree(d_index_result);
    hipFree(d_input);
}


template <typename T>
void AllocateAndCopy(const std::vector<thrust::complex<T>>& host_data, thrust::complex<T>** device_data) {
    size_t size = host_data.size() * sizeof(thrust::complex<T>);
    hipMalloc(device_data, size);
    hipMemcpy(*device_data, host_data.data(), size, hipMemcpyHostToDevice);
}

template <typename T>
std::vector<thrust::complex<T>> CopyToHost(const thrust::complex<T>* device_data, size_t size) {
    std::vector<thrust::complex<T>> host_data(size);
    hipMemcpy(host_data.data(), device_data, size * sizeof(thrust::complex<T>), hipMemcpyDeviceToHost);
    return host_data;
}

TEST(VectorDualTest, RealDualProduct) {
    using T = float;

    // Input size
    int real_size = 3;
    int dual_size = 2;

    // Host input data
    std::vector<thrust::complex<T>> a_real = {{1, 2}, {3, 4}, {5, 6}};
    std::vector<thrust::complex<T>> a_dual = {{0.1, 0.2}, {0.3, 0.4}, {0.5, 0.6}};
    std::vector<thrust::complex<T>> b_real = {{7, 8}, {9, 10}, {11, 12}};
    std::vector<thrust::complex<T>> b_dual = {{0.7, 0.8}, {0.9, 1.0}, {1.1, 1.2}};

    // Allocate device memory
    thrust::complex<T> *d_a_real, *d_a_dual, *d_b_real, *d_b_dual;
    AllocateAndCopy(a_real, &d_a_real);
    AllocateAndCopy(a_dual, &d_a_dual);
    AllocateAndCopy(b_real, &d_b_real);
    AllocateAndCopy(b_dual, &d_b_dual);

    // Output memory
    thrust::complex<T> *d_result_real, *d_result_dual;
    hipMalloc(&d_result_real, real_size * sizeof(thrust::complex<T>));
    hipMalloc(&d_result_dual, dual_size * sizeof(thrust::complex<T>));

    // Launch kernel
    VectorRealDualProductKernel<T><<<1, real_size * dual_size>>>(
        d_a_real, d_a_dual, d_b_real, d_b_dual, real_size, dual_size, d_result_real, d_result_dual);

    // Copy results to host
    auto result_real = CopyToHost(d_result_real, real_size);
    auto result_dual = CopyToHost(d_result_dual, dual_size);

    // Free device memory
    hipFree(d_a_real);
    hipFree(d_a_dual);
    hipFree(d_b_real);
    hipFree(d_b_dual);
    hipFree(d_result_real);
    hipFree(d_result_dual);

    // Validate results
    EXPECT_EQ(result_real[0], a_real[0] * b_real[0]); // Example validation
    EXPECT_EQ(result_dual[0], a_real[0] * b_dual[0] + b_real[0] * a_dual[0]);
}

TEST(VectorDualTest, ElementwiseAdd) {
    using T = float;

    int real_size = 3;
    int dual_size = 3;

    std::vector<thrust::complex<T>> a_real = {{1, 1}, {2, 2}, {3, 3}};
    std::vector<thrust::complex<T>> a_dual = {{0.1, 0.1}, {0.2, 0.2}, {0.3, 0.3}};
    std::vector<thrust::complex<T>> b_real = {{4, 4}, {5, 5}, {6, 6}};
    std::vector<thrust::complex<T>> b_dual = {{0.4, 0.4}, {0.5, 0.5}, {0.6, 0.6}};

    thrust::complex<T> *d_a_real, *d_a_dual, *d_b_real, *d_b_dual;
    AllocateAndCopy(a_real, &d_a_real);
    AllocateAndCopy(a_dual, &d_a_dual);
    AllocateAndCopy(b_real, &d_b_real);
    AllocateAndCopy(b_dual, &d_b_dual);

    thrust::complex<T> *d_result_real, *d_result_dual;
    hipMalloc(&d_result_real, real_size * sizeof(thrust::complex<T>));
    hipMalloc(&d_result_dual, dual_size * sizeof(thrust::complex<T>));

    VectorDualElementwiseAddKernel<T><<<1, real_size>>>(
        d_a_real, d_a_dual, d_b_real, d_b_dual, real_size, dual_size, d_result_real, d_result_dual);

    auto result_real = CopyToHost(d_result_real, real_size);
    auto result_dual = CopyToHost(d_result_dual, dual_size);

    hipFree(d_a_real);
    hipFree(d_a_dual);
    hipFree(d_b_real);
    hipFree(d_b_dual);
    hipFree(d_result_real);
    hipFree(d_result_dual);

    for (int i = 0; i < real_size; ++i) {
        EXPECT_EQ(result_real[i], a_real[i] + b_real[i]);
    }
    for (int i = 0; i < dual_size; ++i) {
        EXPECT_EQ(result_dual[i], a_dual[i] + b_dual[i]);
    }
}

TEST(VectorDualTest, ElementwiseMultiply) {
    using T = float;

    int real_size = 3;
    int dual_size = 2;  //There are dual_size dual numbers for each real number

    std::vector<thrust::complex<T>> a_real = {{1, 1}, {2, 2}, {3, 3}};
    std::vector<thrust::complex<T>> a_dual = generate_random_vector<T>(real_size*dual_size);
    std::vector<thrust::complex<T>> b_real = {{4, 4}, {5, 5}, {6, 6}};
    std::vector<thrust::complex<T>> b_dual = generate_random_vector<T>(real_size*dual_size);

    thrust::complex<T> *d_a_real, *d_a_dual, *d_b_real, *d_b_dual;
    AllocateAndCopy(a_real, &d_a_real);
    AllocateAndCopy(a_dual, &d_a_dual);
    AllocateAndCopy(b_real, &d_b_real);
    AllocateAndCopy(b_dual, &d_b_dual);

    thrust::complex<T> *d_result_real, *d_result_dual;
    hipMalloc(&d_result_real, real_size * sizeof(thrust::complex<T>));
    hipMalloc(&d_result_dual, real_size*dual_size * sizeof(thrust::complex<T>));

    VectorDualElementwiseMultiplyKernel<T><<<1, real_size*dual_size>>>(
        d_a_real, d_a_dual, d_b_real, d_b_dual, real_size, dual_size, d_result_real, d_result_dual);

    auto result_real = CopyToHost(d_result_real, real_size);
    auto result_dual = CopyToHost(d_result_dual, real_size*dual_size);

    hipFree(d_a_real);
    hipFree(d_a_dual);
    hipFree(d_b_real);
    hipFree(d_b_dual);
    hipFree(d_result_real);
    hipFree(d_result_dual);

    for (int i = 0; i < real_size; ++i) {
        EXPECT_EQ(result_real[i], a_real[i] * b_real[i]);
    }
    std::vector<thrust::complex<T>> check_dual(real_size*dual_size);
    for (int i = 0; i < real_size; ++i) {
        for (int j = 0; j < dual_size; ++j) {
            int off = i*dual_size + j;
            check_dual[off] = a_real[i] * b_dual[off] + 
                                           a_dual[off] * b_real[i];
        }
    }
    for (int i = 0; i < real_size*dual_size; ++i) {
        EXPECT_EQ(result_dual[i], check_dual[i]);
    }
}


TEST(VectorDualTest, IndexGet) {
    using T = float;

    // Input sizes and ranges
    int real_size = 6;
    int dual_size = 4;
    int start = 2, end = 5;

    // Host input data
    std::vector<thrust::complex<T>> a_real = generate_random_vector<T>(real_size);
    std::vector<thrust::complex<T>> a_dual = generate_random_vector<T>(real_size*dual_size);

    // Allocate device memory
    thrust::complex<T> *d_a_real, *d_a_dual;
    AllocateAndCopy(a_real, &d_a_real);
    AllocateAndCopy(a_dual, &d_a_dual);

    // Output memory
    thrust::complex<T> *d_result_real, *d_result_dual;
    hipMalloc(&d_result_real, (end - start) * sizeof(thrust::complex<T>));
    hipMalloc(&d_result_dual, (end - start) * dual_size * sizeof(thrust::complex<T>));

    // Launch kernel
    VectorDualIndexGetKernel<T><<<1, (end - start) * dual_size>>>(
        d_a_real, d_a_dual, real_size, dual_size, start, end, d_result_real, d_result_dual);

    // Copy results to host
    auto result_real = CopyToHost(d_result_real, end - start);
    auto result_dual = CopyToHost(d_result_dual, (end - start)*dual_size);

    // Free device memory
    hipFree(d_a_real);
    hipFree(d_a_dual);
    hipFree(d_result_real);
    hipFree(d_result_dual);

    // Validate results
    std::vector<thrust::complex<T>> expected_real = {a_real[2], a_real[3], a_real[4]};
    std::vector<thrust::complex<T>> expected_dual((end-start)*dual_size);
    int count=0;
    for (int i = start; i < end; ++i) {
        for ( int j=0; j<dual_size; ++j) {
            int off = i*dual_size + j;
            expected_dual[count] = a_dual[off];
            count++;
        }
    }

    EXPECT_EQ(result_real.size(), expected_real.size());
    EXPECT_EQ(result_dual.size(), expected_dual.size());

    for (size_t i = 0; i < result_real.size(); ++i) {
        EXPECT_EQ(result_real[i], expected_real[i]) << "Mismatch at index " << i << " in real part.";
    }
    for (size_t i = 0; i < result_dual.size(); ++i) {
        EXPECT_EQ(result_dual[i], expected_dual[i]) << "Mismatch at index " << i << " in dual part.";
    }
}

TEST(VectorDualTest, IndexPut) {
    using T = float;

    // Input sizes and ranges
    int real_size = 6;
    int dual_size = 6;
    int start = 2, end = 5;

    // Host input data (values to insert)
    std::vector<thrust::complex<T>> input_real = generate_random_vector<T>(end - start);
    std::vector<thrust::complex<T>> input_dual = generate_random_vector<T>(dual_size*(end - start));
    // Host output data (initial values of result arrays)
    std::vector<thrust::complex<T>> result_real= generate_random_vector<T>(real_size);
    std::vector<thrust::complex<T>> result_dual= generate_random_vector<T>(real_size*dual_size);

    // Allocate device memory
    thrust::complex<T> *d_input_real, *d_input_dual, *d_result_real, *d_result_dual;
    AllocateAndCopy(input_real, &d_input_real);
    AllocateAndCopy(input_dual, &d_input_dual);
    AllocateAndCopy(result_real, &d_result_real);
    AllocateAndCopy(result_dual, &d_result_dual);

    // Launch kernel
    VectorDualIndexPutKernel<T><<<1, (end - start)*dual_size>>>(
        d_input_real, d_input_dual, start, end, d_result_real, d_result_dual, real_size, dual_size);

    // Copy results to host
    auto result_real_host = CopyToHost(d_result_real, real_size);
    auto result_dual_host = CopyToHost(d_result_dual, real_size*dual_size);

    // Free device memory
    hipFree(d_input_real);
    hipFree(d_input_dual);
    hipFree(d_result_real);
    hipFree(d_result_dual);

    // Validate results
    std::vector<thrust::complex<T>> expected_real(result_real);
    //Substitute the values in the range [start, end) with the input values
    for (int i = start; i < end; ++i) {
        expected_real[i] = input_real[i-start];
    }

    std::vector<thrust::complex<T>> expected_dual(result_dual);

    //Substitute the values in the range [start, end) with the input values
    int count=0;
    for (int i = start; i < end; ++i) {
        for (int j = 0; j < dual_size; ++j) {
            int off = i*dual_size + j;
            expected_dual[off] = input_dual[count];
            count++;
        }
    }

    EXPECT_EQ(result_real_host.size(), expected_real.size());

    EXPECT_EQ(result_dual_host.size(), expected_dual.size());

    for (size_t i = 0; i < result_real.size(); ++i) {
        EXPECT_EQ(result_real_host[i], expected_real[i]) << "Mismatch at index " << i << " in real part.";
    }
    for (size_t i = 0; i < result_dual_host.size(); ++i) {
        EXPECT_EQ(result_dual_host[i], expected_dual[i]) << "Mismatch at index " << i << " in dual part.";
    }
}

TEST(VectorDualTest, Square) {
    using T = float;

    // Input size
    int real_size = 3;
    int dual_size = 6; //There are two dual numbers for each real number

    // Host input data
    std::vector<thrust::complex<T>> a_real = generate_random_vector<T>(real_size);
    std::vector<thrust::complex<T>> a_dual = generate_random_vector<T>(real_size*dual_size);

    // Allocate device memory
    thrust::complex<T> *d_a_real, *d_a_dual, *d_result_real, *d_result_dual;
    AllocateAndCopy(a_real, &d_a_real);
    AllocateAndCopy(a_dual, &d_a_dual);
    hipMalloc(&d_result_real, real_size * sizeof(thrust::complex<T>));
    hipMalloc(&d_result_dual, real_size*dual_size * sizeof(thrust::complex<T>));

    // Launch kernel
    VectorDualSquareKernel<T><<<1, real_size*dual_size>>>(
        d_a_real, d_a_dual, real_size, dual_size, d_result_real, d_result_dual);

    // Copy results to host
    auto result_real = CopyToHost(d_result_real, real_size);
    auto result_dual = CopyToHost(d_result_dual, real_size*dual_size);

    // Free device memory
    hipFree(d_a_real);
    hipFree(d_a_dual);
    hipFree(d_result_real);
    hipFree(d_result_dual);

    // Validate results
    std::vector<thrust::complex<T>> expected_real(real_size);
    for (int i = 0; i < real_size; ++i) {
        expected_real[i] = a_real[i] * a_real[i];
    }

    std::vector<thrust::complex<T>> expected_dual(real_size*dual_size);
    for (int i = 0; i < real_size; ++i) {
        for (int j = 0; j < dual_size; ++j) {
            int off = i*dual_size + j;
            expected_dual[off] = a_real[i] * a_dual[off] + a_dual[off] * a_real[i];
        }
    }

    EXPECT_EQ(result_real.size(), expected_real.size());
    EXPECT_EQ(result_dual.size(), expected_dual.size());

    for (size_t i = 0; i < result_real.size(); ++i) {
        EXPECT_EQ(result_real[i], expected_real[i]) << "Mismatch at index " << i << " in real part.";
    }
    for (size_t i = 0; i < result_dual.size(); ++i) {
        EXPECT_EQ(result_dual[i], expected_dual[i]) << "Mismatch at index " << i << " in dual part.";
    }
}


TEST(VectorDualTest, Pow) {
    using T = float;

    // Input size and power
    int real_size = 3;
    int dual_size = 3;
    T power = 1.5;  // Example power

    // Host input data
    std::vector<thrust::complex<T>> a_real = {{1, 2}, {3, 4}, {5, 6}};
    std::vector<thrust::complex<T>> a_dual = generate_random_vector<T>(real_size*dual_size);

    // Allocate device memory
    thrust::complex<T> *d_a_real, *d_a_dual, *d_result_real, *d_result_dual;
    AllocateAndCopy(a_real, &d_a_real);
    AllocateAndCopy(a_dual, &d_a_dual);
    hipMalloc(&d_result_real, real_size * sizeof(thrust::complex<T>));
    hipMalloc(&d_result_dual, real_size * dual_size * sizeof(thrust::complex<T>));

    // Launch kernel
    VectorDualPowKernel<T><<<1, real_size * dual_size>>>(
        d_a_real, d_a_dual, power, real_size, dual_size, d_result_real, d_result_dual);

    // Copy results to host
    auto result_real = CopyToHost(d_result_real, real_size);
    auto result_dual = CopyToHost(d_result_dual, real_size * dual_size);

    // Free device memory
    hipFree(d_a_real);
    hipFree(d_a_dual);
    hipFree(d_result_real);
    hipFree(d_result_dual);

    // Validate results
    std::vector<thrust::complex<T>> expected_real = {
        pow(a_real[0], power), pow(a_real[1], power), pow(a_real[2], power)};
    std::vector<thrust::complex<T>> expected_dual(real_size * dual_size);

    for (int i = 0; i < real_size; ++i) {
        for (int j = 0; j < dual_size; ++j) {
            int idx = i * dual_size + j;
            expected_dual[idx] = power * pow(a_real[i], power - 1) * a_dual[idx];
        }
    }

    EXPECT_EQ(result_real.size(), expected_real.size());
    EXPECT_EQ(result_dual.size(), expected_dual.size());

    for (size_t i = 0; i < result_real.size(); ++i) {
        EXPECT_NEAR(result_real[i].real(), expected_real[i].real(), 1.0e-4) << "Mismatch at index " << i << " in real part.";
        EXPECT_NEAR(result_real[i].imag(), expected_real[i].imag(), 1.0e-4) << "Mismatch at index " << i << " in real part.";
    }
    for (size_t i = 0; i < result_dual.size(); ++i) {
        EXPECT_NEAR(result_dual[i].real(), expected_dual[i].real(), 1.0e-4) << "Mismatch at index " << i << " in dual part.";
        EXPECT_NEAR(result_dual[i].imag(), expected_dual[i].imag(), 1.0e-4) << "Mismatch at index " << i << " in dual part.";
    }
}

TEST(VectorDualTest, Sqrt) {
    using T = float;

    // Input size
    int real_size = 3;
    int dual_size = 3;

    // Host input data
    std::vector<thrust::complex<T>> a_real = {{4, 0}, {9, 0}, {16, 0}};
    std::vector<thrust::complex<T>> a_dual = generate_random_vector<T>(real_size*dual_size);

    // Allocate device memory
    thrust::complex<T> *d_a_real, *d_a_dual, *d_result_real, *d_result_dual;
    AllocateAndCopy(a_real, &d_a_real);
    AllocateAndCopy(a_dual, &d_a_dual);
    hipMalloc(&d_result_real, real_size * sizeof(thrust::complex<T>));
    hipMalloc(&d_result_dual, real_size * dual_size * sizeof(thrust::complex<T>));

    // Launch kernel
    VectorDualSqrtKernel<T><<<1, real_size * dual_size>>>(
        d_a_real, d_a_dual, real_size, dual_size, d_result_real, d_result_dual);

    // Copy results to host
    auto result_real = CopyToHost(d_result_real, real_size);
    auto result_dual = CopyToHost(d_result_dual, real_size * dual_size);

    // Free device memory
    hipFree(d_a_real);
    hipFree(d_a_dual);
    hipFree(d_result_real);
    hipFree(d_result_dual);

    // Validate results
    std::vector<thrust::complex<T>> expected_real = {
        sqrt(a_real[0]), sqrt(a_real[1]), sqrt(a_real[2])};
    std::vector<thrust::complex<T>> expected_dual(real_size * dual_size);

    for (int i = 0; i < real_size; ++i) {
        for (int j = 0; j < dual_size; ++j) {
            int idx = i * dual_size + j;
            expected_dual[idx] = 0.5 * pow(a_real[i], -0.5) * a_dual[idx];
        }
    }

    EXPECT_EQ(result_real.size(), expected_real.size());
    EXPECT_EQ(result_dual.size(), expected_dual.size());

    for (size_t i = 0; i < result_real.size(); ++i) {
        EXPECT_NEAR(result_real[i].real(), expected_real[i].real(), 1.0e-4) << "Mismatch at index " << i << " in real part.";
        EXPECT_NEAR(result_real[i].imag(), expected_real[i].imag(), 1.0e-4) << "Mismatch at index " << i << " in real part.";

    }
    for (size_t i = 0; i < result_dual.size(); ++i) {
        EXPECT_NEAR(result_dual[i].real(), expected_dual[i].real(), 1.0e-4) << "Mismatch at index " << i << " in dual part.";
        EXPECT_NEAR(result_dual[i].imag(), expected_dual[i].imag(), 1.0e-4) << "Mismatch at index " << i << " in dual part.";

    }
}



// Test case for the get_hyperdual_vector_offsets_kernel
TEST(GetHyperdualVectorOffsetsKernel, ComputesOffsetsCorrectly) {
    // Input parameters
    int i = 2;
    int k = 1;
    int l = 3;
    int rows = 5; // Not used in the kernel but retained for completeness
    int dual = 4;

    // Device pointers for outputs
    int *d_off_i, *d_off_k, *d_off_l;

    // Host pointers for validation
    int h_off_i, h_off_k, h_off_l;

    // Allocate device memory
    ASSERT_EQ(hipMalloc((void **)&d_off_i, sizeof(int)), hipSuccess);
    ASSERT_EQ(hipMalloc((void **)&d_off_k, sizeof(int)), hipSuccess);
    ASSERT_EQ(hipMalloc((void **)&d_off_l, sizeof(int)), hipSuccess);

    // Launch the kernel (1 thread)
    get_hyperdual_vector_offsets_kernel<<<1, 1>>>(i, k, l, rows, dual, d_off_i, d_off_k, d_off_l);

    // Copy results back to host
    ASSERT_EQ(hipMemcpy(&h_off_i, d_off_i, sizeof(int), hipMemcpyDeviceToHost), hipSuccess);
    ASSERT_EQ(hipMemcpy(&h_off_k, d_off_k, sizeof(int), hipMemcpyDeviceToHost), hipSuccess);
    ASSERT_EQ(hipMemcpy(&h_off_l, d_off_l, sizeof(int), hipMemcpyDeviceToHost), hipSuccess);

    // Free device memory
    hipFree(d_off_i);
    hipFree(d_off_k);
    hipFree(d_off_l);

    // Expected results
    int expected_off_i = i;
    int expected_off_k = i * dual + k;
    int expected_off_l = i * dual * dual + k * dual + l;

    // Validate results
    EXPECT_EQ(h_off_i, expected_off_i);
    EXPECT_EQ(h_off_k, expected_off_k);
    EXPECT_EQ(h_off_l, expected_off_l);
}

TEST(VectorHyperDualTest, IndexGet) {
    using T = float;
    // Input sizes and ranges
    int real_size = 6;
    int dual_size = 6;

    int start_real = 1, end_real = 4;

    // Host input data
    std::vector<thrust::complex<T>> a_real = generate_random_vector<T>(real_size);
    std::vector<thrust::complex<T>> a_dual = generate_random_vector<T>(real_size*dual_size);
    std::vector<thrust::complex<T>> a_hyper = generate_random_vector<T>(real_size*dual_size*dual_size);

    // Allocate device memory
    thrust::complex<T> *d_a_real, *d_a_dual, *d_a_hyper;
    AllocateAndCopy(a_real,  &d_a_real);
    AllocateAndCopy(a_dual,  &d_a_dual);
    AllocateAndCopy(a_hyper, &d_a_hyper);

    // Output memory
    thrust::complex<T> *d_result_real, *d_result_dual, *d_result_hyper;
    hipMalloc(&d_result_real, (end_real - start_real) * sizeof(thrust::complex<T>));
    hipMalloc(&d_result_dual, dual_size*(end_real - start_real) * sizeof(thrust::complex<T>));
    hipMalloc(&d_result_hyper, dual_size*dual_size*(end_real - start_real) * sizeof(thrust::complex<T>));
    int result_real_size = end_real - start_real;
    int result_dual_size = (end_real - start_real)*dual_size;
    int result_hyper_size = (end_real - start_real)*dual_size*dual_size;
    // Launch kernel
    //We need to launch at least (end_real - start_real)*(end_dual - start_dual)*(end_dual - start_dual) threads
    VectorHyperDualIndexGetKernel<T><<<1, result_hyper_size>>>(
        d_a_real, d_a_dual, d_a_hyper, real_size, dual_size, start_real, end_real,  
        d_result_real, d_result_dual, d_result_hyper);

    // Copy results to host
    auto result_real = CopyToHost(d_result_real, end_real - start_real);
    auto result_dual = CopyToHost(d_result_dual, (end_real - start_real)*dual_size);
    auto result_hyper = CopyToHost(d_result_hyper, (end_real - start_real)*dual_size*dual_size);

    // Free device memory
    hipFree(d_a_real);
    hipFree(d_a_dual);
    hipFree(d_a_hyper);
    hipFree(d_result_real);
    hipFree(d_result_dual);
    hipFree(d_result_hyper);

    // Validate results
    std::vector<thrust::complex<T>> expected_real(result_real_size);
    std::vector<thrust::complex<T>> expected_dual(result_dual_size);
    std::vector<thrust::complex<T>> expected_hyper(result_hyper_size);

    EXPECT_EQ(result_real.size(), expected_real.size());
    EXPECT_EQ(result_dual.size(), expected_dual.size());
    EXPECT_EQ(result_hyper.size(), expected_hyper.size());
    for ( int i=0; i<result_real_size; ++i) {
        expected_real[i] = a_real[start_real + i];
    }
    int count=0;
    for (int i = start_real; i < end_real; ++i) {
        for ( int j=0; j<dual_size; ++j) {
            int off = i*dual_size + j;
            expected_dual[count] = a_dual[off];
            count++;
        }
    }
    count=0;
    for (int i = start_real; i < end_real; ++i) {
        for ( int j=0; j< dual_size; ++j) {
            for ( int k=0; k<dual_size; ++k) {
                int off = i*dual_size*dual_size + j*dual_size + k;
                expected_hyper[count] = a_hyper[off];
                count++;
            }
        }
    }

    for (size_t i = 0; i < result_real.size(); ++i) {
        EXPECT_EQ(result_real[i], expected_real[i]) << "Mismatch at index " << i << " in real part.";
    }
    for (size_t i = 0; i < result_dual.size(); ++i) {
        EXPECT_EQ(result_dual[i], expected_dual[i]) << "Mismatch at index " << i << " in dual part.";
    }
    for (size_t i = 0; i < result_hyper.size(); ++i) {
        EXPECT_EQ(result_hyper[i], expected_hyper[i]) << "Mismatch at index " << i << " in hyper part.";
    }
}

TEST(VectorHyperDualTest, IndexPut) {
    using T = float;
    // Input sizes and ranges
    int real_size = 6;
    int dual_size = 6;

    int start_real = 1, end_real = 4;

    // Host output data
    std::vector<thrust::complex<T>> dest_real = generate_random_vector<T>(real_size);
    std::vector<thrust::complex<T>> dest_dual = generate_random_vector<T>(real_size*dual_size);
    std::vector<thrust::complex<T>> dest_hyper = generate_random_vector<T>(real_size*dual_size*dual_size);

    // Host input data (values to insert)
    std::vector<thrust::complex<T>> input_real = generate_random_vector<T>(end_real - start_real);
    std::vector<thrust::complex<T>> input_dual = generate_random_vector<T>(dual_size*(end_real - start_real));
    std::vector<thrust::complex<T>> input_hyper = generate_random_vector<T>(dual_size*dual_size*(end_real - start_real));


    // Copy the original values to the expected values
    std::vector<thrust::complex<T>> expected_real(dest_real);
    std::vector<thrust::complex<T>> expected_dual(dest_dual);
    std::vector<thrust::complex<T>> expected_hyper(dest_hyper);

    // Allocate device memory
    thrust::complex<T> *d_dest_real, *d_dest_dual, *d_dest_hyper, *d_input_real, *d_input_dual, *d_input_hyper;
    AllocateAndCopy(dest_real,  &d_dest_real);
    AllocateAndCopy(dest_dual,  &d_dest_dual);
    AllocateAndCopy(dest_hyper, &d_dest_hyper);
    AllocateAndCopy(input_real,  &d_input_real);
    AllocateAndCopy(input_dual,  &d_input_dual);
    AllocateAndCopy(input_hyper, &d_input_hyper);

    // Launch kernel
    //We need to launch at least (end_real - start_real)*(end_dual - start_dual)*(end_dual - start_dual) threads
    VectorHyperDualIndexPutKernel<T><<<1, input_hyper.size() >>>(
         d_input_real, d_input_dual, d_input_hyper,
         end_real-start_real, dual_size, start_real, end_real,  
         d_dest_real, d_dest_dual, d_dest_hyper);

    // Copy results to host
    auto result_real = CopyToHost(d_dest_real, real_size);
    auto result_dual = CopyToHost(d_dest_dual, real_size*dual_size);
    auto result_hyper = CopyToHost(d_dest_hyper, real_size*dual_size*dual_size);

    // Free device memory
    hipFree(d_dest_real);
    hipFree(d_dest_dual);
    hipFree(d_dest_hyper);
    hipFree(d_input_real);
    hipFree(d_input_dual);
    hipFree(d_input_hyper);


    EXPECT_EQ(result_real.size(), expected_real.size());
    EXPECT_EQ(result_dual.size(), expected_dual.size());
    EXPECT_EQ(result_hyper.size(), expected_hyper.size());
    for ( int i=start_real; i<end_real; ++i) {
        expected_real[i] = input_real[i-start_real];
    }
    for (int i = start_real; i < end_real; ++i) {
        for ( int j=0; j<dual_size; ++j) {
            int off = i*dual_size + j;
            expected_dual[off] = input_dual[(i-start_real)*dual_size + j];
        }
    }
    for (int i = start_real; i < end_real; ++i) {
        for ( int j=0; j< dual_size; ++j) {
            for ( int k=0; k<dual_size; ++k) {
                int off = i*dual_size*dual_size + j*dual_size + k;
                expected_hyper[off] = input_hyper[(i-start_real)*dual_size*dual_size + j*dual_size + k];
            }
        }
    }

    for (size_t i = 0; i < result_real.size(); ++i) {
        EXPECT_EQ(result_real[i], expected_real[i]) << "Mismatch at index " << i << " in real part.";
    }
    for (size_t i = 0; i < result_dual.size(); ++i) {
        EXPECT_EQ(result_dual[i], expected_dual[i]) << "Mismatch at index " << i << " in dual part.";
    }
    for (size_t i = 0; i < result_hyper.size(); ++i) {
        EXPECT_EQ(result_hyper[i], expected_hyper[i]) << "Mismatch at index " << i << " in hyper part.";
    }
}


// A helper macro to check for CUDA errors
#define CUDA_CHECK(call)                                               \
    do {                                                               \
        hipError_t err = call;                                        \
        if (err != hipSuccess) {                                      \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ \
                      << ": " << hipGetErrorString(err) << std::endl; \
            std::exit(EXIT_FAILURE);                                   \
        }                                                              \
    } while (0)


// Google Test for MatrixElementwiseAddKernel
TEST(MatrixElementwiseAddTest, BasicAddition) 
{
    using Complex = thrust::complex<double>;

    // Define the dimensions of the matrix
    const int rows = 2;
    const int cols = 3;
    const int totalSize = rows * cols;

    // Prepare host data for matrix A and B
    // We'll just pick arbitrary values
    std::vector<Complex> h_a = {
        Complex(1.0, 2.0), Complex(3.0, 4.0), Complex(5.0,  6.0), 
        Complex(7.0, 8.0), Complex(9.0, 0.0), Complex(-1.0, 2.5)
    };
    std::vector<Complex> h_b = {
        Complex(0.5,  0.5),  Complex(1.0,  1.0), Complex(2.0,  2.0),
        Complex(-3.0, 1.0), Complex(10.0, 10.0), Complex( 0.0, 0.0)
    };

    // Compute expected result on the host
    std::vector<Complex> h_expected(totalSize);
    for (int i = 0; i < totalSize; ++i) {
        h_expected[i] = h_a[i] + h_b[i];
    }

    // Allocate device memory
    Complex* d_a = nullptr;
    Complex* d_b = nullptr;
    Complex* d_result = nullptr;

    CUDA_CHECK(hipMalloc(&d_a,      totalSize * sizeof(Complex)));
    CUDA_CHECK(hipMalloc(&d_b,      totalSize * sizeof(Complex)));
    CUDA_CHECK(hipMalloc(&d_result, totalSize * sizeof(Complex)));

    // Copy host data to device
    CUDA_CHECK(hipMemcpy(d_a, h_a.data(), totalSize * sizeof(Complex), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b, h_b.data(), totalSize * sizeof(Complex), hipMemcpyHostToDevice));

    // Launch the kernel
    int blockSize = 256; 
    int gridSize  = (totalSize + blockSize - 1) / blockSize; 
    MatrixElementwiseAddKernel<<<gridSize, blockSize>>>(d_a, d_b, rows, cols, d_result);

    // Check for kernel errors
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy the result back to the host
    std::vector<Complex> h_result(totalSize);
    CUDA_CHECK(hipMemcpy(h_result.data(), d_result, totalSize * sizeof(Complex), hipMemcpyDeviceToHost));

    // Verify the result against the expected values
    for (int i = 0; i < totalSize; ++i) {
        // We can use EXPECT_NEAR on real and imag parts, 
        // because Complex is a struct of double. 
        // Alternatively, we can define a tolerance for floating comparisons:
        double tol = 1e-9;
        EXPECT_NEAR(h_expected[i].real(), h_result[i].real(), tol) 
            << "Mismatch at index " << i << " (real part)";
        EXPECT_NEAR(h_expected[i].imag(), h_result[i].imag(), tol)
            << "Mismatch at index " << i << " (imag part)";
    }

    // Cleanup
    CUDA_CHECK(hipFree(d_a));
    CUDA_CHECK(hipFree(d_b));
    CUDA_CHECK(hipFree(d_result));
}


// Test for elementwise matrix multiplication
TEST(MatrixElementwiseMultiplyTest, BasicMultiply)
{
    using Complex = thrust::complex<double>;

    // Define matrix dimensions
    const int rows = 2;
    const int cols = 3;
    const int totalSize = rows * cols;

    // Prepare host data for matrices A and B
    // (Just some example values)
    std::vector<Complex> h_a = {
        Complex(1.0, 2.0),  Complex(3.0,  -1.0), Complex(-1.0, 0.5),
        Complex(2.0, 0.0),  Complex(1.5,  2.5),  Complex( 0.0, -3.0)
    };
    std::vector<Complex> h_b = {
        Complex(2.0,  1.0), Complex( 4.0, 0.5),  Complex(1.0, 2.0),
        Complex(-2.0, 0.5), Complex( 1.0, 1.0),  Complex(2.0,  2.0)
    };

    // Compute the expected result on the host
    // result[i] = a[i] * b[i]
    std::vector<Complex> h_expected(totalSize);
    for (int i = 0; i < totalSize; ++i) {
        h_expected[i] = h_a[i] * h_b[i];
    }

    // Allocate device memory
    Complex* d_a = nullptr;
    Complex* d_b = nullptr;
    Complex* d_result = nullptr;

    CUDA_CHECK(hipMalloc(&d_a,      totalSize * sizeof(Complex)));
    CUDA_CHECK(hipMalloc(&d_b,      totalSize * sizeof(Complex)));
    CUDA_CHECK(hipMalloc(&d_result, totalSize * sizeof(Complex)));

    // Copy host memory to device
    CUDA_CHECK(hipMemcpy(d_a, h_a.data(), totalSize * sizeof(Complex), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b, h_b.data(), totalSize * sizeof(Complex), hipMemcpyHostToDevice));

    // Launch the kernel
    int blockSize = 128;
    int gridSize  = (totalSize + blockSize - 1) / blockSize;
    MatrixElementwiseMultiplyKernel<<<gridSize, blockSize>>>(d_a, d_b, rows, cols, d_result);

    // Check for kernel errors and synchronize
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy the result back to the host
    std::vector<Complex> h_result(totalSize);
    CUDA_CHECK(hipMemcpy(h_result.data(), d_result, totalSize * sizeof(Complex), hipMemcpyDeviceToHost));

    // Compare the GPU result to the expected result
    // We use EXPECT_NEAR for both real and imag parts because these are double values.
    double tol = 1e-9;
    for (int i = 0; i < totalSize; ++i) {
        EXPECT_NEAR(h_expected[i].real(), h_result[i].real(), tol)
            << "Mismatch at index " << i << " (real part)";
        EXPECT_NEAR(h_expected[i].imag(), h_result[i].imag(), tol)
            << "Mismatch at index " << i << " (imag part)";
    }

    // Cleanup
    CUDA_CHECK(hipFree(d_a));
    CUDA_CHECK(hipFree(d_b));
    CUDA_CHECK(hipFree(d_result));
}


// Google Test for MatrixSquareKernel
TEST(MatrixSquareTest, BasicSquare) 
{
    using Complex = thrust::complex<double>;

    // Matrix dimensions
    const int rows = 2;
    const int cols = 3;
    const int totalSize = rows * cols;

    // Initialize a small matrix on the host
    std::vector<Complex> h_a = {
        Complex(1.0, 2.0),  Complex(2.0, -1.0), Complex(-1.0,  3.0),
        Complex(2.0, 0.0),  Complex(0.5,  0.5), Complex(-3.0, -3.0)
    };

    // Compute the expected result (elementwise square) on the host
    std::vector<Complex> h_expected(totalSize);
    for (int i = 0; i < totalSize; ++i) {
        // Square is simply a[i] * a[i]
        h_expected[i] = h_a[i] * h_a[i];
    }

    // Device pointers
    Complex* d_a = nullptr;
    Complex* d_result = nullptr;

    // Allocate device memory
    CUDA_CHECK(hipMalloc(&d_a,      totalSize * sizeof(Complex)));
    CUDA_CHECK(hipMalloc(&d_result, totalSize * sizeof(Complex)));

    // Copy host data to device
    CUDA_CHECK(hipMemcpy(d_a, h_a.data(), totalSize * sizeof(Complex), hipMemcpyHostToDevice));

    // Launch the kernel
    int blockSize = 128;
    int gridSize = (totalSize + blockSize - 1) / blockSize;
    MatrixSquareKernel<<<gridSize, blockSize>>>(d_a, rows, cols, d_result);

    // Check for kernel errors, synchronize
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy result back to host
    std::vector<Complex> h_result(totalSize);
    CUDA_CHECK(hipMemcpy(h_result.data(), d_result, totalSize * sizeof(Complex), hipMemcpyDeviceToHost));

    // Compare with the expected results
    double tol = 1e-9;
    for (int i = 0; i < totalSize; ++i) {
        EXPECT_NEAR(h_expected[i].real(), h_result[i].real(), tol)
            << "Mismatch at index " << i << " (real part)";
        EXPECT_NEAR(h_expected[i].imag(), h_result[i].imag(), tol)
            << "Mismatch at index " << i << " (imag part)";
    }

    // Cleanup
    CUDA_CHECK(hipFree(d_a));
    CUDA_CHECK(hipFree(d_result));
}


// Google Test for MatrixPowKernel
TEST(MatrixPowTest, ElementwisePower)
{
    using Complex = thrust::complex<double>;

    // Define dimensions of the matrix
    const int rows = 2;
    const int cols = 3;
    const int totalSize = rows * cols;

    // Define a matrix with arbitrary complex values
    std::vector<Complex> h_a = {
        Complex( 1.0,  2.0), Complex(-2.0,  3.0), Complex( 3.0, -1.0),
        Complex(-1.0, -1.0), Complex( 2.0,  2.0), Complex( 0.5,  1.5)
    };

    // The exponent we want to apply
    double power = 2.5; // for example

    // Compute the expected result on the host
    // We can use std::pow(...) or thrust::pow(...). Since it's host code,
    // std::pow is typically fine, but we'll cast to thrust::complex<double>.
    std::vector<Complex> h_expected(totalSize);
    for (int i = 0; i < totalSize; ++i) {
        // std::pow(...) can be used for complex numbers if <complex> is included,
        // or we can do something like thrust::pow(h_a[i], power).
        // For demonstration, let's use thrust::pow:
        h_expected[i] = thrust::pow(h_a[i], power);
    }

    // Allocate device memory
    Complex* d_a      = nullptr;
    Complex* d_result = nullptr;

    CUDA_CHECK(hipMalloc(&d_a,      totalSize * sizeof(Complex)));
    CUDA_CHECK(hipMalloc(&d_result, totalSize * sizeof(Complex)));

    // Copy host data to device
    CUDA_CHECK(hipMemcpy(d_a, h_a.data(), totalSize * sizeof(Complex), hipMemcpyHostToDevice));

    // Launch the kernel
    int blockSize = 128;
    int gridSize  = (totalSize + blockSize - 1) / blockSize;
    MatrixPowKernel<<<gridSize, blockSize>>>(d_a, power, rows, cols, d_result);

    // Check for launch errors and synchronize
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy the result back to host
    std::vector<Complex> h_result(totalSize);
    CUDA_CHECK(hipMemcpy(h_result.data(), d_result, totalSize * sizeof(Complex), hipMemcpyDeviceToHost));

    // Validate the results
    double tol = 1e-9;
    for (int i = 0; i < totalSize; ++i) {
        // Compare real and imaginary parts
        EXPECT_NEAR(h_expected[i].real(), h_result[i].real(), tol)
            << "Mismatch at index " << i << " (real part)";
        EXPECT_NEAR(h_expected[i].imag(), h_result[i].imag(), tol)
            << "Mismatch at index " << i << " (imag part)";
    }

    // Free device memory
    CUDA_CHECK(hipFree(d_a));
    CUDA_CHECK(hipFree(d_result));
}

//--------------------------------------------------
// Google Test for MatrixReduceKernel
//--------------------------------------------------
TEST(MatrixReduceTest, BasicReduce)
{
    using Complex = thrust::complex<double>;

    // Define matrix dimensions
    const int rows = 4;
    const int cols = 4;
    const int totalSize = rows * cols;

    // Create a small matrix with known complex values
    // 16 elements in total
    std::vector<Complex> h_a = {
        Complex(1.0,  2.0), Complex(2.0,  1.0), Complex(-1.0,  3.0), Complex(0.5,  -1.5),
        Complex(2.5,  2.5), Complex(-2.0, 1.0), Complex( 1.0, -2.0), Complex(-1.0, -1.0),
        Complex(3.0, -1.0), Complex(0.0,  0.5), Complex( 1.0,  1.0), Complex(-0.5,  2.0),
        Complex(2.0,  0.0), Complex(-3.0, 2.0), Complex( 1.5, -1.5), Complex(0.0,  0.75)
    };

    // Compute the expected sum (entire matrix) on the host
    Complex expectedSum(0.0, 0.0);
    for (int i = 0; i < totalSize; ++i) {
        expectedSum += h_a[i];
    }

    // Allocate device memory
    Complex* d_a = nullptr;
    // Each block writes one partial sum, so we need "gridSize" output elements.
    // We'll allocate enough space for the maximum expected gridSize.
    // We'll compute gridSize after we define blockSize.
    Complex* d_partialSums = nullptr;

    // Copy host vector to device
    CUDA_CHECK(hipMalloc(&d_a, totalSize * sizeof(Complex)));
    CUDA_CHECK(hipMemcpy(d_a, h_a.data(), totalSize * sizeof(Complex), hipMemcpyHostToDevice));

    // Decide on block size and grid size
    int blockSize = 8; // e.g., 8 threads per block
    int gridSize  = (totalSize + blockSize - 1) / blockSize; 
    // We will store each block's result in d_partialSums
    CUDA_CHECK(hipMalloc(&d_partialSums, gridSize * sizeof(Complex)));

    // Launch the kernel
    // IMPORTANT: we need dynamic shared memory size = blockSize * sizeof(Complex)
    size_t sharedMemSize = blockSize * sizeof(Complex);
    MatrixReduceKernel<<<gridSize, blockSize, sharedMemSize>>>(d_a, rows, cols, d_partialSums);

    // Check for launch errors and synchronize
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy partial sums back to host
    std::vector<Complex> h_partialSums(gridSize);
    CUDA_CHECK(hipMemcpy(h_partialSums.data(),
                          d_partialSums,
                          gridSize * sizeof(Complex),
                          hipMemcpyDeviceToHost));

    // Now reduce the partial sums on the host
    Complex finalSum(0.0, 0.0);
    for (int i = 0; i < gridSize; ++i) {
        finalSum += h_partialSums[i];
    }

    // Compare with the reference sum
    double tol = 1e-9;
    EXPECT_NEAR(expectedSum.real(), finalSum.real(), tol)
        << "Mismatch in real part of reduction result";
    EXPECT_NEAR(expectedSum.imag(), finalSum.imag(), tol)
        << "Mismatch in imaginary part of reduction result";

    // Cleanup
    CUDA_CHECK(hipFree(d_a));
    CUDA_CHECK(hipFree(d_partialSums));
}


TEST(MatrixSumTest, SumColumns)
{
    using Complex = thrust::complex<double>;

    // Small 2x3 matrix
    const int rows = 2;
    const int cols = 3;
    const int totalSize = rows * cols;

    // Host data
    // Matrix layout (row-major):
    // Row 0: (1.0,2.0), (2.0,1.0), (3.0,-1.0)
    // Row 1: (4.0,1.5), (0.0,-2.0), (1.0,2.0)
    std::vector<Complex> h_a = {
        Complex(1.0, 2.0),  Complex(2.0, 1.0),  Complex(3.0, -1.0),
        Complex(4.0, 1.5),  Complex(0.0, -2.0), Complex(1.0, 2.0)
    };

    // We want column sums => result length = cols (3)
    // Summation along dim=0 means summing "down" each column.
    // Let's compute the expected sums on the host:
    // Column 0: (1+4, 2+1.5)   = (5.0, 3.5)
    // Column 1: (2+0, 1-2)     = (2.0, -1.0)
    // Column 2: (3+1, -1+2)    = (4.0, 1.0)
    std::vector<Complex> h_expected(cols);
    h_expected[0] = Complex(5.0, 3.5);
    h_expected[1] = Complex(2.0, -1.0);
    h_expected[2] = Complex(4.0, 1.0);

    // Allocate device memory
    Complex* d_a = nullptr;
    Complex* d_result = nullptr;
    CUDA_CHECK(hipMalloc(&d_a,      totalSize * sizeof(Complex)));
    CUDA_CHECK(hipMalloc(&d_result, cols       * sizeof(Complex)));

    // Copy host data to device
    CUDA_CHECK(hipMemcpy(d_a, h_a.data(), totalSize * sizeof(Complex), hipMemcpyHostToDevice));

    // Initialize result array to zero
    std::vector<Complex> h_init(cols, Complex(0.0, 0.0));
    CUDA_CHECK(hipMemcpy(d_result, h_init.data(), cols * sizeof(Complex), hipMemcpyHostToDevice));

    // Launch kernel with a single block (grid=1) so each element is processed by exactly one thread
    int blockSize = totalSize; // i.e., 6 threads
    int gridSize  = 1;
    MatrixSumKernel<<<gridSize, blockSize>>>(d_a, rows, cols, /*dim=*/0, d_result);

    // Check for launch errors, then synchronize
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy result back to host
    std::vector<Complex> h_result(cols);
    CUDA_CHECK(hipMemcpy(h_result.data(), d_result, cols * sizeof(Complex), hipMemcpyDeviceToHost));

    // Check correctness
    double tol = 1e-9;
    for (int c = 0; c < cols; ++c) {
        EXPECT_NEAR(h_expected[c].real(), h_result[c].real(), tol)
            << "Mismatch in real part of column " << c;
        EXPECT_NEAR(h_expected[c].imag(), h_result[c].imag(), tol)
            << "Mismatch in imaginary part of column " << c;
    }

    // Clean up
    CUDA_CHECK(hipFree(d_a));
    CUDA_CHECK(hipFree(d_result));
}

TEST(MatrixSumTest, SumRows)
{
    using Complex = thrust::complex<double>;

    // Same 2x3 matrix as above
    const int rows = 2;
    const int cols = 3;
    const int totalSize = rows * cols;

    std::vector<Complex> h_a = {
        Complex(1.0, 2.0),  Complex(2.0, 1.0),  Complex(3.0, -1.0),
        Complex(4.0, 1.5),  Complex(0.0, -2.0), Complex(1.0, 2.0)
    };

    // Summation along dim=1 => sum across columns, producing a result of length = rows (2).
    // Let's compute the expected sums on the host:
    // Row 0: (1.0+2.0+3.0, 2.0+1.0-1.0) = (6.0, 2.0)
    // Row 1: (4.0+0.0+1.0, 1.5-2.0+2.0) = (5.0, 1.5)
    std::vector<Complex> h_expected(rows);
    h_expected[0] = Complex(6.0, 2.0);
    h_expected[1] = Complex(5.0, 1.5);

    // Allocate device memory
    Complex* d_a = nullptr;
    Complex* d_result = nullptr;
    CUDA_CHECK(hipMalloc(&d_a,      totalSize * sizeof(Complex)));
    CUDA_CHECK(hipMalloc(&d_result, rows       * sizeof(Complex)));

    // Copy matrix to device
    CUDA_CHECK(hipMemcpy(d_a, h_a.data(), totalSize * sizeof(Complex), hipMemcpyHostToDevice));

    // Initialize result array to zero
    std::vector<Complex> h_init(rows, Complex(0.0, 0.0));
    CUDA_CHECK(hipMemcpy(d_result, h_init.data(), rows * sizeof(Complex), hipMemcpyHostToDevice));

    // Launch the kernel
    int blockSize = totalSize; // 6 threads, 1 block
    int gridSize  = 1;
    MatrixSumKernel<<<gridSize, blockSize>>>(d_a, rows, cols, /*dim=*/1, d_result);

    // Check for errors and sync
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy result back to host
    std::vector<Complex> h_result(rows);
    CUDA_CHECK(hipMemcpy(h_result.data(), d_result, rows * sizeof(Complex), hipMemcpyDeviceToHost));

    // Verify correctness
    double tol = 1e-9;
    for (int r = 0; r < rows; ++r) {
        EXPECT_NEAR(h_expected[r].real(), h_result[r].real(), tol)
            << "Mismatch in real part of row " << r;
        EXPECT_NEAR(h_expected[r].imag(), h_result[r].imag(), tol)
            << "Mismatch in imaginary part of row " << r;
    }

    // Clean up
    CUDA_CHECK(hipFree(d_a));
    CUDA_CHECK(hipFree(d_result));
}


TEST(MatrixIndexGetTest, SliceAllElements)
{
    using Complex = thrust::complex<double>;

    // Let's define a small 2x3 matrix => totalSize = 6
    // Flattened in row-major order:
    // idx:  0        1        2        3         4         5
    // val: (1,2), (2,1), (3,3), (4,0.5), (5,-2), (6,6)
    const int rows = 2;
    const int cols = 3;
    const int totalSize = rows * cols;

    // Build a small host array
    std::vector<Complex> h_a = {
        {1.0, 2.0}, {2.0, 1.0}, {3.0, 3.0}, 
        {4.0, 0.5}, {5.0, -2.0}, {6.0, 6.0}
    };

    // We'll slice from start=0 to end=6 => entire array
    int start = 0;
    int end   = totalSize;

    // Expected result is the same as the input
    std::vector<Complex> h_expected(h_a);

    // Allocate device buffers
    Complex* d_a = nullptr;
    Complex* d_result = nullptr;
    CUDA_CHECK(hipMalloc(&d_a, totalSize * sizeof(Complex)));
    CUDA_CHECK(hipMalloc(&d_result, (end - start) * sizeof(Complex)));

    // Copy input to device
    CUDA_CHECK(hipMemcpy(d_a, h_a.data(), totalSize * sizeof(Complex), hipMemcpyHostToDevice));

    // Launch kernel: we need enough threads to cover [0, totalSize).
    int blockSize = 128;
    int gridSize  = (totalSize + blockSize - 1) / blockSize;
    MatrixIndexGetKernel<<<gridSize, blockSize>>>(d_a, start, end, rows, cols, d_result);

    // Check for kernel errors and synchronize
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy slice result back
    std::vector<Complex> h_result(end - start);
    CUDA_CHECK(hipMemcpy(h_result.data(), d_result, (end - start) * sizeof(Complex), hipMemcpyDeviceToHost));

    // Compare results
    double tol = 1e-9;
    for (int i = 0; i < (end - start); ++i) {
        EXPECT_NEAR(h_expected[i].real(), h_result[i].real(), tol)
            << "Mismatch at index " << i << " in real part";
        EXPECT_NEAR(h_expected[i].imag(), h_result[i].imag(), tol)
            << "Mismatch at index " << i << " in imag part";
    }

    // Clean up
    CUDA_CHECK(hipFree(d_a));
    CUDA_CHECK(hipFree(d_result));
}

TEST(MatrixIndexGetTest, PartialSlice)
{
    using Complex = thrust::complex<double>;

    // Same 2x3 matrix => totalSize = 6
    // Indices: 0..5
    // Values: (1.0,2.0), (2.0,1.0), (3.0,3.0), (4.0,0.5), (5.0,-2.0), (6.0,6.0)
    const int rows = 2;
    const int cols = 3;
    const int totalSize = rows * cols;

    std::vector<Complex> h_a = {
        {1.0,  2.0}, {2.0, 1.0}, {3.0, 3.0},
        {4.0,  0.5}, {5.0, -2.0}, {6.0, 6.0}
    };

    // Let's take a slice from idx=1 to idx=4 => includes elements at [1,2,3]
    // That should produce 3 elements
    int start = 1;
    int end   = 4;

    // Expected: a[1], a[2], a[3]
    // => (2.0,1.0), (3.0,3.0), (4.0,0.5)
    std::vector<Complex> h_expected = {
        {2.0, 1.0}, {3.0, 3.0}, {4.0, 0.5}
    };

    // Allocate device memory
    Complex* d_a = nullptr;
    Complex* d_result = nullptr;
    CUDA_CHECK(hipMalloc(&d_a, totalSize * sizeof(Complex)));
    CUDA_CHECK(hipMalloc(&d_result, (end - start) * sizeof(Complex)));

    // Copy input to device
    CUDA_CHECK(hipMemcpy(d_a, h_a.data(), totalSize * sizeof(Complex), hipMemcpyHostToDevice));

    // Launch kernel
    int blockSize = 128;
    int gridSize  = (totalSize + blockSize - 1) / blockSize;
    MatrixIndexGetKernel<<<gridSize, blockSize>>>(d_a, start, end, rows, cols, d_result);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy slice result back
    std::vector<Complex> h_result(end - start);
    CUDA_CHECK(hipMemcpy(h_result.data(), d_result, (end - start) * sizeof(Complex), hipMemcpyDeviceToHost));

    // Compare results
    double tol = 1e-9;
    ASSERT_EQ(h_expected.size(), h_result.size());
    for (int i = 0; i < (end - start); ++i) {
        EXPECT_NEAR(h_expected[i].real(), h_result[i].real(), tol)
            << "Mismatch at slice index " << i << " in real part";
        EXPECT_NEAR(h_expected[i].imag(), h_result[i].imag(), tol)
            << "Mismatch at slice index " << i << " in imag part";
    }

    // Cleanup
    CUDA_CHECK(hipFree(d_a));
    CUDA_CHECK(hipFree(d_result));
}

TEST(MatrixIndexGetTest, EmptySlice)
{
    using Complex = thrust::complex<double>;

    // 2x3 matrix => totalSize = 6
    const int rows = 2;
    const int cols = 3;
    const int totalSize = rows * cols;

    std::vector<Complex> h_a = {
        {1.0,  2.0}, {2.0, 1.0}, {3.0, 3.0},
        {4.0,  0.5}, {5.0, -2.0}, {6.0, 6.0}
    };

    // If we choose start=end=3, it yields zero elements in [3,3).
    int start = 3;
    int end   = 3; // empty

    // We expect an empty result
    std::vector<Complex> h_expected;  // size=0

    // Allocate device memory
    Complex* d_a = nullptr;
    CUDA_CHECK(hipMalloc(&d_a, totalSize * sizeof(Complex)));
    CUDA_CHECK(hipMemcpy(d_a, h_a.data(), totalSize * sizeof(Complex), hipMemcpyHostToDevice));

    // For an empty slice, we can skip allocating d_result 
    // or we can allocate zero bytes for it. Let's do a non-null pointer for safety:
    Complex* d_result = nullptr;
    CUDA_CHECK(hipMalloc(&d_result, 0)); // 0 bytes

    // Launch kernel
    int blockSize = 128;
    int gridSize  = (totalSize + blockSize - 1) / blockSize;
    MatrixIndexGetKernel<<<gridSize, blockSize>>>(d_a, start, end, rows, cols, d_result);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy results: none to copy
    std::vector<Complex> h_result; 
    // We expect it to remain empty.

    // Compare sizes (both should be 0)
    EXPECT_EQ(h_expected.size(), h_result.size());

    // Cleanup
    CUDA_CHECK(hipFree(d_a));
    CUDA_CHECK(hipFree(d_result));
}


//------------------------------------------------------------------------------
// TEST 1: Put the entire input array into the result from index 0 to totalSize
//------------------------------------------------------------------------------
TEST(MatrixIndexPutTest, FullRangePut)
{
    using Complex = thrust::complex<double>;

    // For a 2x3 matrix, totalSize = 6
    const int rows = 2;
    const int cols = 3;
    const int totalSize = rows * cols;

    // Define input array of length totalSize
    std::vector<Complex> h_input = {
        {1.0,  2.0}, {2.0,  1.0}, {3.0, -1.0}, 
        {4.0,  4.5}, {5.0,  2.0}, {6.0,  6.0}
    };

    // We'll put into an initially empty (or zero) result array of length totalSize
    std::vector<Complex> h_resultInit(totalSize, {0.0, 0.0});

    // Expected final result = exactly h_input
    std::vector<Complex> h_expected(h_input);

    // Copy data to device
    Complex* d_input  = nullptr;
    Complex* d_result = nullptr;
    CUDA_CHECK(hipMalloc(&d_input,  totalSize * sizeof(Complex)));
    CUDA_CHECK(hipMalloc(&d_result, totalSize * sizeof(Complex)));

    CUDA_CHECK(hipMemcpy(d_input,  h_input.data(),   totalSize * sizeof(Complex), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_result, h_resultInit.data(), totalSize * sizeof(Complex), hipMemcpyHostToDevice));

    // We'll put [start=0..end=6)
    int start = 0;
    int end   = totalSize;

    // Launch kernel
    int blockSize = 128;
    int gridSize  = (totalSize + blockSize - 1) / blockSize;
    MatrixIndexPutKernel<<<gridSize, blockSize>>>(d_input, start, end, rows, cols, d_result);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy result back to host
    std::vector<Complex> h_out(totalSize);
    CUDA_CHECK(hipMemcpy(h_out.data(), d_result, totalSize * sizeof(Complex), hipMemcpyDeviceToHost));

    // Check correctness
    double tol = 1e-9;
    for (int i = 0; i < totalSize; ++i) {
        EXPECT_NEAR(h_expected[i].real(), h_out[i].real(), tol)
            << "Mismatch at index " << i << " in real part";
        EXPECT_NEAR(h_expected[i].imag(), h_out[i].imag(), tol)
            << "Mismatch at index " << i << " in imaginary part";
    }

    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_result));
}

//------------------------------------------------------------------------------
// TEST 2: Put a partial slice of input into the middle of the result
//------------------------------------------------------------------------------
TEST(MatrixIndexPutTest, PartialRangePut)
{
    using Complex = thrust::complex<double>;

    // For a 2x3 matrix, totalSize = 6
    const int rows = 2;
    const int cols = 3;
    const int totalSize = rows * cols;

    // The input we want to copy: length = 3
    std::vector<Complex> h_input = {
        {10.0, -1.0}, {20.0, 1.0}, {30.0,  5.0}
    };

    // The result array initially (some placeholder values)
    // We'll define 6 elements so we can put data into [start=2..5)
    // Note: result array has totalSize = 6
    std::vector<Complex> h_resultInit = {
        {1.0, 1.0},  {2.0, 2.0},  // Indices 0,1
        {0.0, 0.0},  {0.0, 0.0},  // Indices 2,3 (to be overwritten with input[0..1])
        {0.0, 0.0},  {3.0, 4.0}   // Indices 4,5 (Index 4 will be overwritten with input[2], 5 remains)
    };

    // We'll put the input array of length 3 into the subrange [2..5) of the result
    int start = 2;
    int end   = 5;  // This covers indices 2,3,4 in 'result'

    // So the expected final result:
    //   Index: 0: {1.0,1.0}    (unchanged)
    //          1: {2.0,2.0}    (unchanged)
    //          2: {10.0,-1.0}  (from input[0])
    //          3: {20.0, 1.0}  (from input[1])
    //          4: {30.0, 5.0}  (from input[2])
    //          5: {3.0,4.0}    (unchanged)
    std::vector<Complex> h_expected = {
        {1.0, 1.0}, {2.0, 2.0}, 
        {10.0, -1.0}, {20.0, 1.0}, 
        {30.0, 5.0}, {3.0, 4.0}
    };

    // Allocate device memory
    Complex* d_input  = nullptr;
    Complex* d_result = nullptr;
    CUDA_CHECK(hipMalloc(&d_input,  h_input.size() * sizeof(Complex)));
    CUDA_CHECK(hipMalloc(&d_result, totalSize * sizeof(Complex)));

    CUDA_CHECK(hipMemcpy(d_input,  h_input.data(),   h_input.size() * sizeof(Complex), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_result, h_resultInit.data(), totalSize * sizeof(Complex), hipMemcpyHostToDevice));

    // Launch kernel
    int blockSize = 128;
    int gridSize  = (totalSize + blockSize - 1) / blockSize;
    MatrixIndexPutKernel<<<gridSize, blockSize>>>(d_input, start, end, rows, cols, d_result);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy result back to host
    std::vector<Complex> h_out(totalSize);
    CUDA_CHECK(hipMemcpy(h_out.data(), d_result, totalSize * sizeof(Complex), hipMemcpyDeviceToHost));

    // Check correctness
    double tol = 1e-9;
    ASSERT_EQ(h_out.size(), h_expected.size());
    for (int i = 0; i < totalSize; ++i) {
        EXPECT_NEAR(h_expected[i].real(), h_out[i].real(), tol)
            << "Mismatch at index " << i << " in real part";
        EXPECT_NEAR(h_expected[i].imag(), h_out[i].imag(), tol)
            << "Mismatch at index " << i << " in imaginary part";
    }

    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_result));
}

//------------------------------------------------------------------------------
// TEST 3: Put an empty range => no changes
//------------------------------------------------------------------------------
TEST(MatrixIndexPutTest, EmptyRangePut)
{
    using Complex = thrust::complex<double>;

    // For a 2x3 matrix, totalSize=6
    const int rows = 2;
    const int cols = 3;
    const int totalSize = rows * cols;

    // The input array (3 elements) but we won't actually use them
    std::vector<Complex> h_input = {
        {1.0,1.0}, {2.0,2.0}, {3.0,3.0}
    };

    // The result array
    std::vector<Complex> h_resultInit = {
        {1.0, 1.0}, {2.0, 2.0}, {3.0, 3.0},
        {4.0, 4.0}, {5.0, 5.0}, {6.0, 6.0}
    };

    // start = end => empty slice
    int start = 3;
    int end   = 3;

    // We expect no changes to h_resultInit
    std::vector<Complex> h_expected = h_resultInit;

    // Allocate device memory
    Complex* d_input  = nullptr;
    Complex* d_result = nullptr;
    CUDA_CHECK(hipMalloc(&d_input,  h_input.size() * sizeof(Complex)));
    CUDA_CHECK(hipMalloc(&d_result, totalSize * sizeof(Complex)));

    CUDA_CHECK(hipMemcpy(d_input,  h_input.data(),   h_input.size() * sizeof(Complex), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_result, h_resultInit.data(), totalSize * sizeof(Complex), hipMemcpyHostToDevice));

    // Launch kernel
    int blockSize = 128;
    int gridSize  = (totalSize + blockSize - 1) / blockSize;
    MatrixIndexPutKernel<<<gridSize, blockSize>>>(d_input, start, end, rows, cols, d_result);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy result back
    std::vector<Complex> h_out(totalSize);
    CUDA_CHECK(hipMemcpy(h_out.data(), d_result, totalSize * sizeof(Complex), hipMemcpyDeviceToHost));

    // Validate
    double tol = 1e-9;
    for (int i = 0; i < totalSize; ++i) {
        EXPECT_NEAR(h_expected[i].real(), h_out[i].real(), tol)
            << "Mismatch at index " << i << " in real part";
        EXPECT_NEAR(h_expected[i].imag(), h_out[i].imag(), tol)
            << "Mismatch at index " << i << " in imaginary part";
    }

    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_result));
}


//------------------------------------------------------------------------------
// Test 1: (1 x N), dim=0 => should copy the row [0..N-1] into a 1D array of length N
//------------------------------------------------------------------------------
TEST(MatrixSqueezeTest, OneByN_Dim0)
{
    using Complex = thrust::complex<double>;

    // 1 x 4 example
    int rows = 1;
    int cols = 4;

    // Flattened matrix: (1 row, 4 columns)
    // Indices: a[0], a[1], a[2], a[3]
    std::vector<Complex> h_a = {
        {1.0, 1.0}, {2.0, 2.0}, {3.0, 3.0}, {4.0, 4.0}
    };

    // We expect the output to be exactly these 4 elements
    std::vector<Complex> h_expected = h_a;

    // Allocate device memory
    Complex* d_a = nullptr;
    Complex* d_result = nullptr;
    int totalSize = rows * cols; // = 4
    CUDA_CHECK(hipMalloc(&d_a,      totalSize * sizeof(Complex)));
    CUDA_CHECK(hipMalloc(&d_result, cols * sizeof(Complex))); // length=4

    // Copy host -> device
    CUDA_CHECK(hipMemcpy(d_a, h_a.data(), totalSize * sizeof(Complex), hipMemcpyHostToDevice));

    // Launch the kernel
    // We only need up to 'cols' threads if (rows=1, dim=0).
    int blockSize = 128;
    int gridSize  = (cols + blockSize - 1) / blockSize;
    MatrixSqueezeKernel<<<gridSize, blockSize>>>(d_a, rows, cols, /*dim=*/0, d_result);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy back the result
    std::vector<Complex> h_result(cols);
    CUDA_CHECK(hipMemcpy(h_result.data(), d_result, cols*sizeof(Complex), hipMemcpyDeviceToHost));

    // Verify
    double tol = 1e-9;
    for (int i = 0; i < cols; ++i) {
        EXPECT_NEAR(h_expected[i].real(), h_result[i].real(), tol)
            << "Mismatch in real part at index " << i;
        EXPECT_NEAR(h_expected[i].imag(), h_result[i].imag(), tol)
            << "Mismatch in imag part at index " << i;
    }

    // Cleanup
    CUDA_CHECK(hipFree(d_a));
    CUDA_CHECK(hipFree(d_result));
}

//------------------------------------------------------------------------------
// Test 2: (N x 1), dim=1 => should copy the column [0..N-1] into a 1D array of length N
//------------------------------------------------------------------------------
TEST(MatrixSqueezeTest, NxOne_Dim1)
{
    using Complex = thrust::complex<double>;

    // 3 x 1
    int rows = 3;
    int cols = 1;

    // Flattened matrix: 3 elements, all in column 0
    // a[0] => (1, 1), a[1] => (2, 2), a[2] => (3, 3)
    std::vector<Complex> h_a = {
        {1.0,1.0}, {2.0,2.0}, {3.0,3.0}
    };
    int totalSize = rows * cols;  // = 3
    // We expect the output to be these 3 elements in a 1D array
    std::vector<Complex> h_expected = h_a;

    // Allocate
    Complex* d_a = nullptr;
    Complex* d_result = nullptr;
    CUDA_CHECK(hipMalloc(&d_a,      totalSize*sizeof(Complex)));
    CUDA_CHECK(hipMalloc(&d_result, rows*sizeof(Complex))); // =3

    CUDA_CHECK(hipMemcpy(d_a, h_a.data(), totalSize*sizeof(Complex), hipMemcpyHostToDevice));

    // Launch the kernel (rows threads if dim=1 and cols=1)
    int blockSize = 128;
    int gridSize  = (rows + blockSize - 1) / blockSize;
    MatrixSqueezeKernel<<<gridSize, blockSize>>>(d_a, rows, cols, /*dim=*/1, d_result);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy result
    std::vector<Complex> h_result(rows);
    CUDA_CHECK(hipMemcpy(h_result.data(), d_result, rows*sizeof(Complex), hipMemcpyDeviceToHost));

    // Check
    double tol = 1e-9;
    for (int i = 0; i < rows; ++i) {
        EXPECT_NEAR(h_expected[i].real(), h_result[i].real(), tol)
            << "Mismatch in real part at index " << i;
        EXPECT_NEAR(h_expected[i].imag(), h_result[i].imag(), tol)
            << "Mismatch in imag part at index " << i;
    }

    // Cleanup
    CUDA_CHECK(hipFree(d_a));
    CUDA_CHECK(hipFree(d_result));
}

//------------------------------------------------------------------------------
// Test 3: If dimension isn't actually size=1, it does nothing
//         -> result remains unchanged
//------------------------------------------------------------------------------
TEST(MatrixSqueezeTest, NoOpIfNotSizeOne)
{
    using Complex = thrust::complex<double>;

    // 2 x 3 => neither dim=0 nor dim=1 is size=1
    int rows = 2;
    int cols = 3;
    int totalSize = rows * cols;

    // Input data
    std::vector<Complex> h_a = {
        {1.0,1.0}, {2.0,2.0}, {3.0,3.0},
        {4.0,4.0}, {5.0,5.0}, {6.0,6.0}
    };

    // The result array (initialized with some distinct values to see if overwritten)
    // We'll just make it the same size to simplify; but the code won't fill it anyway.
    std::vector<Complex> h_resultInit = {
        {10.0,10.0}, {20.0,20.0}, {30.0,30.0},
        {40.0,40.0}, {50.0,50.0}, {60.0,60.0}
    };

    // Expected is the same as the init, because we do "nothing"
    std::vector<Complex> h_expected = h_resultInit;

    // Device memory
    Complex* d_a      = nullptr;
    Complex* d_result = nullptr;
    CUDA_CHECK(hipMalloc(&d_a,      totalSize*sizeof(Complex)));
    CUDA_CHECK(hipMalloc(&d_result, totalSize*sizeof(Complex)));

    // Copy
    CUDA_CHECK(hipMemcpy(d_a,      h_a.data(),      totalSize*sizeof(Complex), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_result, h_resultInit.data(), totalSize*sizeof(Complex), hipMemcpyHostToDevice));

    // Launch kernel with dim=0 but rows=2 => do nothing
    int blockSize = 128;
    int gridSize  = (totalSize + blockSize - 1) / blockSize;
    MatrixSqueezeKernel<<<gridSize, blockSize>>>(d_a, rows, cols, /*dim=*/0, d_result);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy result
    std::vector<Complex> h_out(totalSize);
    CUDA_CHECK(hipMemcpy(h_out.data(), d_result, totalSize*sizeof(Complex), hipMemcpyDeviceToHost));

    // Check => no changes
    double tol = 1e-9;
    for (int i = 0; i < totalSize; ++i) {
        EXPECT_NEAR(h_expected[i].real(), h_out[i].real(), tol)
            << "Mismatch in real part at index " << i;
        EXPECT_NEAR(h_expected[i].imag(), h_out[i].imag(), tol)
            << "Mismatch in imag part at index " << i;
    }

    // Cleanup
    CUDA_CHECK(hipFree(d_a));
    CUDA_CHECK(hipFree(d_result));
}

//------------------------------------------------------------------------------
// Test 4: (1 x 1) matrix for both dim=0 and dim=1
//         Both rows==1 and cols==1 => either dimension is size=1
//         Usually you'd pick dim=0 or dim=1. In typical libraries, you'd only
//         remove each dimension once, but let's see the code behavior
//------------------------------------------------------------------------------
TEST(MatrixSqueezeTest, OneByOne)
{
    using Complex = thrust::complex<double>;

    int rows = 1, cols = 1;
    // single element
    std::vector<Complex> h_a = { {42.0, -3.5} };

    // We'll try dim=0
    {
        // result should have length=cols=1
        // which is just the same single element
        std::vector<Complex> h_expected = h_a;

        Complex *d_a = nullptr, *d_result = nullptr;
        CUDA_CHECK(hipMalloc(&d_a,      1*sizeof(Complex)));
        CUDA_CHECK(hipMalloc(&d_result, 1*sizeof(Complex)));

        CUDA_CHECK(hipMemcpy(d_a, h_a.data(), 1*sizeof(Complex), hipMemcpyHostToDevice));

        // We only need 1 thread
        MatrixSqueezeKernel<<<1,1>>>(d_a, rows, cols, /*dim=*/0, d_result);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());

        // Check
        std::vector<Complex> h_out(1);
        CUDA_CHECK(hipMemcpy(h_out.data(), d_result, 1*sizeof(Complex), hipMemcpyDeviceToHost));

        double tol = 1e-9;
        EXPECT_NEAR(h_expected[0].real(), h_out[0].real(), tol) << "dim=0, single (1x1)";
        EXPECT_NEAR(h_expected[0].imag(), h_out[0].imag(), tol) << "dim=0, single (1x1)";

        CUDA_CHECK(hipFree(d_a));
        CUDA_CHECK(hipFree(d_result));
    }

    // Now dim=1
    {
        // Similarly, the result should have length=rows=1, the same single element
        std::vector<Complex> h_expected = h_a;

        Complex *d_a = nullptr, *d_result = nullptr;
        CUDA_CHECK(hipMalloc(&d_a,      1*sizeof(Complex)));
        CUDA_CHECK(hipMalloc(&d_result, 1*sizeof(Complex)));

        CUDA_CHECK(hipMemcpy(d_a, h_a.data(), 1*sizeof(Complex), hipMemcpyHostToDevice));

        MatrixSqueezeKernel<<<1,1>>>(d_a, rows, cols, /*dim=*/1, d_result);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());

        // Check
        std::vector<Complex> h_out(1);
        CUDA_CHECK(hipMemcpy(h_out.data(), d_result, 1*sizeof(Complex), hipMemcpyDeviceToHost));

        double tol = 1e-9;
        EXPECT_NEAR(h_expected[0].real(), h_out[0].real(), tol) << "dim=1, single (1x1)";
        EXPECT_NEAR(h_expected[0].imag(), h_out[0].imag(), tol) << "dim=1, single (1x1)";

        CUDA_CHECK(hipFree(d_a));
        CUDA_CHECK(hipFree(d_result));
    }
}


//------------------------------------------------------------------------
// Google Test for MatrixDualElementwiseAdd
//------------------------------------------------------------------------
TEST(MatrixDualElementwiseAddTest, BasicAddition)
{
    using Complex = thrust::complex<double>;

    // Example shape
    int rows = 2;
    int cols = 3;
    int dual_size = 2; // let's say we store the real part in dual[0], derivative in dual[1]

    int realSize = rows * cols;                 // size for real_ arrays
    int dualSize = rows * cols * dual_size;     // size for dual_ arrays

    // 1) Prepare host arrays for A and B (real + dual)
    // Let's create A and B with small test data

    // A: real part = [ (1,1), (2,2), (3,3),  (4,4), (5,5), (6,6) ]
    // B: real part = [ (0.5,0.5), (1.0,1.0), (2.0,2.0),  ... ]

    std::vector<Complex> h_a_real = {
        {1.0,1.0}, {2.0,2.0}, {3.0,3.0},
        {4.0,4.0}, {5.0,5.0}, {6.0,6.0}
    };
    std::vector<Complex> h_b_real = {
        {0.5, 0.5}, {1.0,1.0}, {2.0,2.0},
        {3.0,3.0},  {4.0,4.0}, {5.0,5.0}
    };

    // For dual parts, each has shape [rows * cols * dual_size].
    // We'll store some small example values. We'll treat each (i,j,k) as:
    //   A_dual[idx] = (real= i+ j/10. + k, imag=0), just to have distinct values.
    // We do something similar for B_dual.

    std::vector<Complex> h_a_dual(dualSize);
    std::vector<Complex> h_b_dual(dualSize);

    for (int i = 0; i < dualSize; ++i) {
        // Let’s do something arbitrary but consistent
        double reA = 10.0 + i;     // each index i
        double reB = 100.0 + i; 
        h_a_dual[i] = Complex(reA, -1.0 * i);  // imaginary part depends on i
        h_b_dual[i] = Complex(reB,  2.0 * i);
    }

    // 2) Compute the expected result on the host
    //   - result_real[i*cols + j] = a_real + b_real (for the row/col)
    //   - result_dual[idx]       = a_dual[idx] + b_dual[idx]

    std::vector<Complex> h_expected_real(realSize);
    for (int off = 0; off < realSize; ++off) {
        h_expected_real[off] = h_a_real[off] + h_b_real[off];
    }

    std::vector<Complex> h_expected_dual(dualSize);
    for (int idx = 0; idx < dualSize; ++idx) {
        h_expected_dual[idx] = h_a_dual[idx] + h_b_dual[idx];
    }

    // 3) Allocate device memory
    Complex *d_a_real = nullptr, *d_b_real = nullptr;
    Complex *d_a_dual = nullptr, *d_b_dual = nullptr;
    Complex *d_result_real = nullptr, *d_result_dual = nullptr;

    CUDA_CHECK(hipMalloc(&d_a_real,    realSize*sizeof(Complex)));
    CUDA_CHECK(hipMalloc(&d_b_real,    realSize*sizeof(Complex)));
    CUDA_CHECK(hipMalloc(&d_a_dual,    dualSize*sizeof(Complex)));
    CUDA_CHECK(hipMalloc(&d_b_dual,    dualSize*sizeof(Complex)));
    CUDA_CHECK(hipMalloc(&d_result_real, realSize*sizeof(Complex)));
    CUDA_CHECK(hipMalloc(&d_result_dual, dualSize*sizeof(Complex)));

    // 4) Copy host -> device
    CUDA_CHECK(hipMemcpy(d_a_real,  h_a_real.data(),  realSize*sizeof(Complex), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b_real,  h_b_real.data(),  realSize*sizeof(Complex), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_a_dual,  h_a_dual.data(),  dualSize*sizeof(Complex), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b_dual,  h_b_dual.data(),  dualSize*sizeof(Complex), hipMemcpyHostToDevice));

    // 5) Launch the kernel
    // total threads = rows * cols * dual_size
    int totalThreads = rows * cols * dual_size;
    int blockSize = 128;
    int gridSize  = (totalThreads + blockSize - 1) / blockSize;

    MatrixDualElementwiseAddKernel<<<gridSize, blockSize>>>(
        d_a_real, d_a_dual,
        d_b_real, d_b_dual,
        rows, cols, dual_size,
        d_result_real, d_result_dual
    );

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // 6) Copy results back to host
    std::vector<Complex> h_result_real(realSize);
    std::vector<Complex> h_result_dual(dualSize);

    CUDA_CHECK(hipMemcpy(h_result_real.data(), d_result_real, realSize*sizeof(Complex), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_result_dual.data(), d_result_dual, dualSize*sizeof(Complex), hipMemcpyDeviceToHost));

    // 7) Compare with expected
    double tol = 1e-9;

    // Compare real part
    for (int i = 0; i < realSize; ++i) {
        EXPECT_NEAR(h_expected_real[i].real(), h_result_real[i].real(), tol)
            << "Mismatch in real-part (REAL array) at index " << i << " (real component)";
        EXPECT_NEAR(h_expected_real[i].imag(), h_result_real[i].imag(), tol)
            << "Mismatch in real-part (REAL array) at index " << i << " (imag component)";
    }

    // Compare dual part
    for (int i = 0; i < dualSize; ++i) {
        EXPECT_NEAR(h_expected_dual[i].real(), h_result_dual[i].real(), tol)
            << "Mismatch in DUAL array at index " << i << " (real component)";
        EXPECT_NEAR(h_expected_dual[i].imag(), h_result_dual[i].imag(), tol)
            << "Mismatch in DUAL array at index " << i << " (imag component)";
    }

    // 8) Free device memory
    CUDA_CHECK(hipFree(d_a_real));
    CUDA_CHECK(hipFree(d_b_real));
    CUDA_CHECK(hipFree(d_a_dual));
    CUDA_CHECK(hipFree(d_b_dual));
    CUDA_CHECK(hipFree(d_result_real));
    CUDA_CHECK(hipFree(d_result_dual));
}


//--------------------------------------------------
// Google Test: Elementwise Multiply for Dual Tensors
//--------------------------------------------------
TEST(MatrixDualElementwiseMultiplyTest, BasicMultiply)
{
    using Complex = thrust::complex<double>;

    // Define small matrix dimensions
    int rows = 2;
    int cols = 3;
    int dual_size = 2;  // For example, storing 1 real + 1 derivative dimension

    // Size for real parts
    int realSize = rows * cols;       // 6
    // Size for dual parts
    int dualSize = rows * cols * dual_size;  // 12

    // 1) Prepare host data for matrix A and B
    //    We'll fill them with some distinct values.

    // A_real: e.g. [(1,1), (2,2), (3,3), (4,4), (5,5), (6,6)]
    std::vector<Complex> h_a_real = {
        {1.0,1.0}, {2.0,2.0}, {3.0,3.0},
        {4.0,4.0}, {5.0,5.0}, {6.0,6.0}
    };

    // B_real: e.g. [(0.5, 0.5), (1.0,1.0), (2.0, 2.0), ...]
    std::vector<Complex> h_b_real = {
        {0.5,0.5}, {1.0,1.0}, {2.0,2.0},
        {3.0,3.0}, {4.0,4.0}, {5.0,5.0}
    };

    // A_dual and B_dual: each of size dualSize=12
    // We'll create a small pattern so we can verify cross terms
    std::vector<Complex> h_a_dual(dualSize), h_b_dual(dualSize);
    for (int idx = 0; idx < dualSize; ++idx) {
        // Something like real= 10+idx, imag= -idx, etc.
        h_a_dual[idx] = Complex(10.0 + idx, -1.0 * idx);
        h_b_dual[idx] = Complex(20.0 + idx,  2.0 * idx);
    }

    // 2) Compute expected result on the host
    // real part: rA * rB
    // dual part: rA * dB + rB * dA
    std::vector<Complex> h_expected_real(realSize);
    std::vector<Complex> h_expected_dual(dualSize);

    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            int off = i * cols + j;
            Complex rA = h_a_real[off];
            Complex rB = h_b_real[off];

            // Real part
            Complex realProd = rA * rB; 
            h_expected_real[off] = realProd;

            // For each partial dimension k
            for (int k = 0; k < dual_size; ++k) {
                int idx = (i*(cols*dual_size)) + (j*dual_size) + k;
                Complex dA = h_a_dual[idx];
                Complex dB = h_b_dual[idx];

                Complex dualVal = rA * dB + rB * dA;
                h_expected_dual[idx] = dualVal;
            }
        }
    }

    // 3) Allocate device memory
    Complex *d_a_real = nullptr, *d_b_real = nullptr;
    Complex *d_a_dual = nullptr, *d_b_dual = nullptr;
    Complex *d_result_real = nullptr, *d_result_dual = nullptr;

    CUDA_CHECK(hipMalloc(&d_a_real,    realSize * sizeof(Complex)));
    CUDA_CHECK(hipMalloc(&d_b_real,    realSize * sizeof(Complex)));
    CUDA_CHECK(hipMalloc(&d_a_dual,    dualSize * sizeof(Complex)));
    CUDA_CHECK(hipMalloc(&d_b_dual,    dualSize * sizeof(Complex)));
    CUDA_CHECK(hipMalloc(&d_result_real, realSize * sizeof(Complex)));
    CUDA_CHECK(hipMalloc(&d_result_dual, dualSize * sizeof(Complex)));

    // 4) Copy data to device
    CUDA_CHECK(hipMemcpy(d_a_real, h_a_real.data(), 
                          realSize*sizeof(Complex), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b_real, h_b_real.data(), 
                          realSize*sizeof(Complex), hipMemcpyHostToDevice));

    CUDA_CHECK(hipMemcpy(d_a_dual, h_a_dual.data(), 
                          dualSize*sizeof(Complex), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b_dual, h_b_dual.data(), 
                          dualSize*sizeof(Complex), hipMemcpyHostToDevice));

    // 5) Launch the kernel
    int totalThreads = realSize * dual_size;  // = rows*cols*dual_size
    int blockSize = 128;
    int gridSize  = (totalThreads + blockSize - 1) / blockSize;

    MatrixDualElementwiseMultiplyKernel<<<gridSize, blockSize>>>(
        d_a_real, d_a_dual, d_b_real, d_b_dual, 
        rows, cols, dual_size, 
        d_result_real, d_result_dual
    );

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // 6) Copy results back
    std::vector<Complex> h_result_real(realSize);
    std::vector<Complex> h_result_dual(dualSize);

    CUDA_CHECK(hipMemcpy(h_result_real.data(), d_result_real, 
                          realSize*sizeof(Complex), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_result_dual.data(), d_result_dual, 
                          dualSize*sizeof(Complex), hipMemcpyDeviceToHost));

    // 7) Compare with expected
    double tol = 1e-9;

    // Check real part
    for (int i = 0; i < realSize; ++i) {
        EXPECT_NEAR(h_expected_real[i].real(), h_result_real[i].real(), tol)
            << "Mismatch in real-part (REAL array) at index " << i << " (Re)";
        EXPECT_NEAR(h_expected_real[i].imag(), h_result_real[i].imag(), tol)
            << "Mismatch in real-part (REAL array) at index " << i << " (Im)";
    }

    // Check dual part
    for (int i = 0; i < dualSize; ++i) {
        EXPECT_NEAR(h_expected_dual[i].real(), h_result_dual[i].real(), tol)
            << "Mismatch in (DUAL array) at index " << i << " (Re)";
        EXPECT_NEAR(h_expected_dual[i].imag(), h_result_dual[i].imag(), tol)
            << "Mismatch in (DUAL array) at index " << i << " (Im)";
    }

    // 8) Cleanup
    CUDA_CHECK(hipFree(d_a_real));
    CUDA_CHECK(hipFree(d_b_real));
    CUDA_CHECK(hipFree(d_a_dual));
    CUDA_CHECK(hipFree(d_b_dual));
    CUDA_CHECK(hipFree(d_result_real));
    CUDA_CHECK(hipFree(d_result_dual));
}



//--------------------------------------------------
// Google Test: MatrixDualSquare
//--------------------------------------------------
TEST(MatrixDualSquareTest, BasicSquare)
{
    using Complex = thrust::complex<double>;

    // Dimensions
    int rows = 2;
    int cols = 3;
    int dual_size = 2; // e.g., storing real + 1 partial derivative

    int realSize = rows * cols;          // 6
    int dualSize = rows * cols * dual_size; // 12

    // 1) Prepare a small matrix A on the host
    // For example, A_real = [ (1,1), (2,2), (3,3), (4,4), (5,5), (6,6) ]
    std::vector<Complex> h_a_real = {
        {1.0,1.0}, {2.0,2.0}, {3.0,3.0},
        {4.0,4.0}, {5.0,5.0}, {6.0,6.0}
    };

    // A_dual: size=12
    // We'll fill it with a distinct pattern so we can see how the derivative is handled.
    // For instance, let a_dual[i] = (10+i, -i).
    std::vector<Complex> h_a_dual(dualSize);
    for (int i = 0; i < dualSize; ++i) {
        h_a_dual[i] = Complex(10.0 + i, -1.0 * i);
    }

    // 2) Compute the expected "square" on the host
    //    For each element (r + eps*d): (r^2 + eps(2*r*d)).
    std::vector<Complex> h_expected_real(realSize);
    std::vector<Complex> h_expected_dual(dualSize);

    // We'll loop over (rows,cols, dual_size)
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            int off = i * cols + j;
            Complex r = h_a_real[off];

            // r^2 => real part
            h_expected_real[off] = r * r;

            // dual part => 2 * r * d
            for (int k = 0; k < dual_size; ++k) {
                int idx = (i*(cols*dual_size)) + (j*dual_size) + k;
                Complex d = h_a_dual[idx];
                // (r + eps*d)^2 => r^2 + eps(2*r*d)
                h_expected_dual[idx] = (Complex)(2.0) * r * d;
            }
        }
    }

    // 3) Allocate device memory
    Complex* d_a_real = nullptr;
    Complex* d_a_dual = nullptr;
    Complex* d_result_real = nullptr;
    Complex* d_result_dual = nullptr;

    CUDA_CHECK(hipMalloc(&d_a_real,      realSize * sizeof(Complex)));
    CUDA_CHECK(hipMalloc(&d_a_dual,      dualSize * sizeof(Complex)));
    CUDA_CHECK(hipMalloc(&d_result_real, realSize * sizeof(Complex)));
    CUDA_CHECK(hipMalloc(&d_result_dual, dualSize * sizeof(Complex)));

    // 4) Copy to device
    CUDA_CHECK(hipMemcpy(d_a_real, h_a_real.data(), realSize*sizeof(Complex), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_a_dual, h_a_dual.data(), dualSize*sizeof(Complex), hipMemcpyHostToDevice));

    // 5) Launch the kernel
    // total threads = rows*cols*dual_size
    int totalThreads = realSize * dual_size; 
    int blockSize = 128;
    int gridSize  = (totalThreads + blockSize - 1) / blockSize;
    MatrixDualSquareKernel<<<gridSize, blockSize>>>(d_a_real, d_a_dual, rows, cols, dual_size, 
                                                    d_result_real, d_result_dual);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // 6) Copy results back
    std::vector<Complex> h_result_real(realSize);
    std::vector<Complex> h_result_dual(dualSize);

    CUDA_CHECK(hipMemcpy(h_result_real.data(), d_result_real, realSize*sizeof(Complex), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_result_dual.data(), d_result_dual, dualSize*sizeof(Complex), hipMemcpyDeviceToHost));

    // 7) Validate
    double tol = 1e-9;
    // Real part
    for (int i = 0; i < realSize; ++i) {
        EXPECT_NEAR(h_expected_real[i].real(), h_result_real[i].real(), tol)
            << "Mismatch in real-part (REAL array) at index " << i << " (Re)";
        EXPECT_NEAR(h_expected_real[i].imag(), h_result_real[i].imag(), tol)
            << "Mismatch in real-part (REAL array) at index " << i << " (Im)";
    }
    // Dual part
    for (int i = 0; i < dualSize; ++i) {
        EXPECT_NEAR(h_expected_dual[i].real(), h_result_dual[i].real(), tol)
            << "Mismatch in (DUAL array) at index " << i << " (Re)";
        EXPECT_NEAR(h_expected_dual[i].imag(), h_result_dual[i].imag(), tol)
            << "Mismatch in (DUAL array) at index " << i << " (Im)";
    }

    // 8) Cleanup
    CUDA_CHECK(hipFree(d_a_real));
    CUDA_CHECK(hipFree(d_a_dual));
    CUDA_CHECK(hipFree(d_result_real));
    CUDA_CHECK(hipFree(d_result_dual));
}


//----------------------------------------------------------------------
// Google Test for MatrixDualPowKernel
//----------------------------------------------------------------------
TEST(MatrixDualPowTest, BasicPow)
{
    using Complex = thrust::complex<double>;

    // Example dimensions
    int rows = 2;
    int cols = 3;
    int dual_size = 2;  // e.g., store real + 1 partial derivative per element

    int realSize = rows * cols;            // 6
    int totalSize = realSize * dual_size;  // 12

    // Chosen exponent
    double power = 2.5;  // for example

    // 1) Prepare host data: a_real, a_dual
    // Let's do a_real with 6 distinct values
    std::vector<Complex> h_a_real = {
        {1.0,  0.0}, {2.0,  -1.0}, {3.0,  0.5},
        {0.5,  1.5}, {2.0,   2.0}, {4.0,  -2.0}
    };

    // a_dual = 12 values
    // We'll fill them with a simple pattern
    std::vector<Complex> h_a_dual(totalSize);
    for (int i = 0; i < totalSize; ++i) {
        double re = 10.0 + i;
        double im = -i;
        h_a_dual[i] = Complex(re, im);
    }

    // 2) Compute the expected results on the host
    //    result_real[i] = pow(a_real[i], power)
    //    result_dual[idx] = power * pow(a_real[off], power - 1) * a_dual[idx]
    std::vector<Complex> h_expected_real(realSize);
    std::vector<Complex> h_expected_dual(totalSize);

    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            int off = i * cols + j;
            // base for this element
            Complex base = h_a_real[off];
            // real part
            h_expected_real[off] = thrust::pow(base, power);

            // fill derivative parts
            for (int k = 0; k < dual_size; ++k) {
                int idx = off * dual_size + k;
                Complex d = h_a_dual[idx];
                // chain rule derivative
                h_expected_dual[idx] = power * thrust::pow(base, power - 1) * d;
            }
        }
    }

    // 3) Device allocations
    Complex* d_a_real      = nullptr;
    Complex* d_a_dual      = nullptr;
    Complex* d_result_real = nullptr;
    Complex* d_result_dual = nullptr;

    CUDA_CHECK(hipMalloc(&d_a_real,      realSize  * sizeof(Complex)));
    CUDA_CHECK(hipMalloc(&d_a_dual,      totalSize * sizeof(Complex)));
    CUDA_CHECK(hipMalloc(&d_result_real, realSize  * sizeof(Complex)));
    CUDA_CHECK(hipMalloc(&d_result_dual, totalSize * sizeof(Complex)));

    // 4) Copy host -> device
    CUDA_CHECK(hipMemcpy(d_a_real,  h_a_real.data(), 
                          realSize*sizeof(Complex), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_a_dual,  h_a_dual.data(), 
                          totalSize*sizeof(Complex), hipMemcpyHostToDevice));

    // 5) Launch kernel
    int blockSize = 128;
    int gridSize  = (totalSize + blockSize - 1) / blockSize;
    MatrixDualPowKernel<<<gridSize, blockSize>>>(
        d_a_real, d_a_dual, power, rows, cols, dual_size,
        d_result_real, d_result_dual
    );

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // 6) Copy results back
    std::vector<Complex> h_result_real(realSize);
    std::vector<Complex> h_result_dual(totalSize);

    CUDA_CHECK(hipMemcpy(h_result_real.data(), d_result_real, 
                          realSize*sizeof(Complex), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_result_dual.data(), d_result_dual, 
                          totalSize*sizeof(Complex), hipMemcpyDeviceToHost));

    // 7) Compare with expected
    double tol = 1e-9;

    // Compare the real part
    for (int i = 0; i < realSize; ++i) {
        EXPECT_NEAR(h_expected_real[i].real(), h_result_real[i].real(), tol)
            << "Mismatch in real-part (REAL array) at index " << i << " (Re)";
        EXPECT_NEAR(h_expected_real[i].imag(), h_result_real[i].imag(), tol)
            << "Mismatch in real-part (REAL array) at index " << i << " (Im)";
    }

    // Compare the dual part
    for (int i = 0; i < totalSize; ++i) {
        EXPECT_NEAR(h_expected_dual[i].real(), h_result_dual[i].real(), tol)
            << "Mismatch in (DUAL array) at index " << i << " (Re)";
        EXPECT_NEAR(h_expected_dual[i].imag(), h_result_dual[i].imag(), tol)
            << "Mismatch in (DUAL array) at index " << i << " (Im)";
    }

    // 8) Cleanup
    CUDA_CHECK(hipFree(d_a_real));
    CUDA_CHECK(hipFree(d_a_dual));
    CUDA_CHECK(hipFree(d_result_real));
    CUDA_CHECK(hipFree(d_result_dual));
}

// Test for slicing a dual matrix
TEST(MatrixDualIndexGet2DTest, Basic2DSlice)
{
    using Complex = thrust::complex<double>;

    // Full matrix dimension: rows=3, cols=4
    // So the real part has 3*4=12 elements
    int rows = 3;
    int cols = 4;
    // Let's say dual_size=2
    int dual_size = 2;
    int realSize  = rows * cols;           // 12
    int dualSize  = realSize * dual_size;  // 24

    // 1) Create host data for the real part
    // We'll store something easily trackable
    // row=0 => indices:0..3, row=1 => 4..7, row=2 => 8..11
    // for example:
    //   h_a_real[i] = ( (double)i, -(double)i ), to see them clearly
    std::vector<Complex> h_a_real(realSize);
    for (int i = 0; i < realSize; ++i) {
        double re = (double)(100 + i);
        double im = -(double)i;
        h_a_real[i] = Complex(re, im);
    }

    // 2) Create host data for the dual part (24 elements)
    // We'll make each partial distinct as well
    std::vector<Complex> h_a_dual(dualSize);
    for (int i = 0; i < dualSize; ++i) {
        double re = 200.0 + i;
        double im = 10.0 + i;
        h_a_dual[i] = Complex(re, im);
    }

    // 3) Define the slice parameters:
    //   rowStart=1, rowEnd=3 => that means row indices {1,2}
    //   colStart=1, colEnd=3 => that means col indices {1,2}
    // So the submatrix is shape (2 x 2)
    int rowStart = 1, rowEnd = 3;
    int colStart = 1, colEnd = 3;
    int outRows = rowEnd - rowStart; // 2
    int outCols = colEnd - colStart; // 2
    int outRealSize = outRows * outCols;        // 4
    int outDualSize = outRealSize * dual_size;  // 8

    // 4) Compute the expected submatrix on the host
    // For each row in [1..2], col in [1..2]
    // we read the real array => originalOff = row*cols + col
    // we read the dual array => originalOff*dual_size + k
    std::vector<Complex> h_expectedReal(outRealSize);
    std::vector<Complex> h_expectedDual(outDualSize);

    int idxSub = 0;  // index in submatrix
    for (int r = rowStart; r < rowEnd; ++r) {
        for (int c = colStart; c < colEnd; ++c) {
            int originalOff = r * cols + c;
            // copy real
            h_expectedReal[idxSub] = h_a_real[originalOff];
            // copy dual for partial dimensions
            for (int k = 0; k < dual_size; ++k) {
                int origDualOff = originalOff * dual_size + k;
                int subDualOff  = idxSub * dual_size + k;
                h_expectedDual[subDualOff] = h_a_dual[origDualOff];
            }
            idxSub++;
        }
    }

    // 5) Device allocations for input and output
    Complex *d_a_real = nullptr, *d_a_dual = nullptr;
    Complex *d_out_real = nullptr, *d_out_dual = nullptr;

    CUDA_CHECK(hipMalloc(&d_a_real, realSize*sizeof(Complex)));
    CUDA_CHECK(hipMalloc(&d_a_dual, dualSize*sizeof(Complex)));
    CUDA_CHECK(hipMalloc(&d_out_real, outRealSize*sizeof(Complex)));
    CUDA_CHECK(hipMalloc(&d_out_dual, outDualSize*sizeof(Complex)));

    // Copy host -> device
    CUDA_CHECK(hipMemcpy(d_a_real, h_a_real.data(), 
                          realSize*sizeof(Complex), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_a_dual, h_a_dual.data(), 
                          dualSize*sizeof(Complex), hipMemcpyHostToDevice));

    // 6) Launch the kernel
    // We'll need outRealSize*dual_size threads => (2*2)*2=8 in this example
    int totalThreads = outRealSize * dual_size;  
    int blockSize = 128;
    int gridSize  = (totalThreads + blockSize - 1) / blockSize;

    MatrixDualIndexGet2DKernel<<<gridSize, blockSize>>>(
        d_a_real, d_a_dual,
        rows, cols, dual_size,
        rowStart, rowEnd, colStart, colEnd,
        d_out_real, d_out_dual
    );

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // 7) Copy results back
    std::vector<Complex> h_resultReal(outRealSize);
    std::vector<Complex> h_resultDual(outDualSize);

    CUDA_CHECK(hipMemcpy(h_resultReal.data(), d_out_real, 
                          outRealSize*sizeof(Complex), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_resultDual.data(), d_out_dual, 
                          outDualSize*sizeof(Complex), hipMemcpyDeviceToHost));

    // 8) Compare with expected
    double tol = 1e-9;

    // Check real
    for (int i = 0; i < outRealSize; ++i) {
        EXPECT_NEAR(h_expectedReal[i].real(), h_resultReal[i].real(), tol)
            << "Mismatch in real part of submatrix index=" << i << " (Re)";
        EXPECT_NEAR(h_expectedReal[i].imag(), h_resultReal[i].imag(), tol)
            << "Mismatch in real part of submatrix index=" << i << " (Im)";
    }

    // Check dual
    for (int i = 0; i < outDualSize; ++i) {
        EXPECT_NEAR(h_expectedDual[i].real(), h_resultDual[i].real(), tol)
            << "Mismatch in dual part of submatrix index=" << i << " (Re)";
        EXPECT_NEAR(h_expectedDual[i].imag(), h_resultDual[i].imag(), tol)
            << "Mismatch in dual part of submatrix index=" << i << " (Im)";
    }

    // 9) Cleanup
    CUDA_CHECK(hipFree(d_a_real));
    CUDA_CHECK(hipFree(d_a_dual));
    CUDA_CHECK(hipFree(d_out_real));
    CUDA_CHECK(hipFree(d_out_dual));
}


//--------------------------------------------------
// Test
//--------------------------------------------------
TEST(MatrixDualIndexPut2DTest, BasicSubSlicePut)
{
    using Complex = thrust::complex<double>;

    // Source dual matrix: shape = (4 x 5), dual_size=2
    // => src_real has 20 elements, src_dual has 40
    int srcRows=4, srcCols=5, dual_size=2;
    int srcRealSize= srcRows * srcCols;      //=20
    int srcDualSize= srcRealSize * dual_size;//=40

    // We'll define a sub-slice:
    //   rowStartSrc=1, rowEndSrc=3 => rows in [1..2], 2 total
    //   colStartSrc=2, colEndSrc=4 => cols in [2..3], 2 total
    // So the sub-slice is shape (2 x 2).
    int rowStartSrc=1, rowEndSrc=3;
    int colStartSrc=2, colEndSrc=4;

    // Destination dual matrix: shape = (5 x 6), dual_size=2
    // => dst_real has 30 elements, dst_dual has 60
    int dstRows=5, dstCols=6;
    int dstRealSize= dstRows * dstCols;      //=30
    int dstDualSize= dstRealSize * dual_size;//=60

    // We'll place the sub-slice at (rowStartDst=2, colStartDst=1) in the destination
    int rowStartDst=2, colStartDst=1;

    // 1) Build host data for src_real, src_dual
    //    Fill them with recognizable patterns
    std::vector<Complex> h_srcReal(srcRealSize);
    for (int i=0; i< srcRealSize; ++i) {
        double re= 10.0 + i;
        double im= -1.0 * i;
        h_srcReal[i] = Complex(re, im);
    }
    std::vector<Complex> h_srcDual(srcDualSize);
    for (int i=0; i< srcDualSize; ++i) {
        double re= 100.0 + i;
        double im= 5.0 + i;
        h_srcDual[i] = Complex(re, im);
    }

    // 2) Build host data for dst_real, dst_dual
    //    We'll fill with placeholders
    std::vector<Complex> h_dstReal(dstRealSize);
    for (int i=0; i< dstRealSize; ++i) {
        double re= 1000.0 + i;
        double im= 10.0 + i;
        h_dstReal[i] = Complex(re, im);
    }
    std::vector<Complex> h_dstDual(dstDualSize);
    for (int i=0; i< dstDualSize; ++i) {
        double re= 2000.0 + i;
        double im= -(50.0 + i);
        h_dstDual[i] = Complex(re, im);
    }

    // 3) Build the expected final destination on the host
    //    We'll copy the sub-slice from src into this subregion of the original dst
    std::vector<Complex> h_expectedReal = h_dstReal; // start from original
    std::vector<Complex> h_expectedDual = h_dstDual;

    int subRows = rowEndSrc - rowStartSrc;  //=2
    int subCols = colEndSrc - colStartSrc;  //=2
    for(int rr=0; rr< subRows; ++rr) {
        for(int cc=0; cc< subCols; ++cc) {
            // global row/col in src
            int srcRow= rowStartSrc + rr;
            int srcCol= colStartSrc + cc;
            int srcOff = srcRow* srcCols + srcCol;

            // global row/col in dst
            int dstRow= rowStartDst + rr;
            int dstCol= colStartDst + cc;
            int dstOff= dstRow* dstCols + dstCol;

            // real part
            h_expectedReal[dstOff] = h_srcReal[srcOff];

            // dual part for each partial index
            for(int k=0; k< dual_size; ++k) {
                int srcDualOff= srcOff* dual_size + k;
                int dstDualOff= dstOff* dual_size + k;
                h_expectedDual[dstDualOff] = h_srcDual[srcDualOff];
            }
        }
    }

    // 4) Device allocations
    Complex *d_srcReal=nullptr, *d_srcDual=nullptr;
    Complex *d_dstReal=nullptr, *d_dstDual=nullptr;

    CUDA_CHECK(hipMalloc(&d_srcReal, srcRealSize*sizeof(Complex)));
    CUDA_CHECK(hipMalloc(&d_srcDual, srcDualSize*sizeof(Complex)));
    CUDA_CHECK(hipMalloc(&d_dstReal, dstRealSize*sizeof(Complex)));
    CUDA_CHECK(hipMalloc(&d_dstDual, dstDualSize*sizeof(Complex)));

    // Copy host -> device
    CUDA_CHECK(hipMemcpy(d_srcReal, h_srcReal.data(),
                          srcRealSize*sizeof(Complex), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_srcDual, h_srcDual.data(),
                          srcDualSize*sizeof(Complex), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_dstReal, h_dstReal.data(),
                          dstRealSize*sizeof(Complex), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_dstDual, h_dstDual.data(),
                          dstDualSize*sizeof(Complex), hipMemcpyHostToDevice));

    // 5) Launch kernel for the sub-slice
    int subSize = subRows * subCols * dual_size; // 2*2*2=8
    int blockSize=128;
    int gridSize= (subSize + blockSize - 1)/ blockSize;

    MatrixDualIndexPut2DKernel<<<gridSize, blockSize>>>(
        d_srcReal, d_srcDual,
        srcRows, srcCols, dual_size,
        rowStartSrc, rowEndSrc,
        colStartSrc, colEndSrc,
        d_dstReal, d_dstDual,
        dstRows, dstCols,
        rowStartDst, colStartDst
    );

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // 6) Copy results back
    std::vector<Complex> h_resultReal(dstRealSize);
    std::vector<Complex> h_resultDual(dstDualSize);

    CUDA_CHECK(hipMemcpy(h_resultReal.data(), d_dstReal,
                          dstRealSize*sizeof(Complex), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_resultDual.data(), d_dstDual,
                          dstDualSize*sizeof(Complex), hipMemcpyDeviceToHost));

    // 7) Compare
    double tol = 1e-9;
    // Real
    for(int i=0; i< dstRealSize; ++i) {
        EXPECT_NEAR(h_expectedReal[i].real(), h_resultReal[i].real(), tol)
            << "Mismatch in real part, index=" << i;
        EXPECT_NEAR(h_expectedReal[i].imag(), h_resultReal[i].imag(), tol)
            << "Mismatch in real part, index=" << i;
    }
    // Dual
    for(int i=0; i< dstDualSize; ++i) {
        EXPECT_NEAR(h_expectedDual[i].real(), h_resultDual[i].real(), tol)
            << "Mismatch in dual part, index=" << i << " (Re)";
        EXPECT_NEAR(h_expectedDual[i].imag(), h_resultDual[i].imag(), tol)
            << "Mismatch in dual part, index=" << i << " (Im)";
    }

    // 8) Cleanup
    CUDA_CHECK(hipFree(d_srcReal));
    CUDA_CHECK(hipFree(d_srcDual));
    CUDA_CHECK(hipFree(d_dstReal));
    CUDA_CHECK(hipFree(d_dstDual));
}


TEST(MatrixDualSigncondTest, BasicTest) {
    const int rows = 2;
    const int cols = 2;
    const int dual_size = 2;

    // Input matrices (real and dual parts)
    std::vector<thrust::complex<double>> a_real = {
        {1.0, 0.0}, {-2.0, 0.0},
        {3.0, 0.0}, {-4.0, 0.0}
    };
    std::vector<thrust::complex<double>> a_dual = {
        {0.1, 0.0}, {0.2, 0.0},
        {0.3, 0.0}, {0.4, 0.0},
        {0.5, 0.0}, {0.6, 0.0},
        {0.7, 0.0}, {0.8, 0.0}
    };

    std::vector<thrust::complex<double>> b_real = {
        {1.0, 0.0}, {1.0, 0.0},
        {-1.0, 0.0}, {-1.0, 0.0}
    };
    std::vector<thrust::complex<double>> b_dual = {
        {0.0, 0.0}, {0.0, 0.0},
        {0.0, 0.0}, {0.0, 0.0},
        {0.0, 0.0}, {0.0, 0.0},
        {0.0, 0.0}, {0.0, 0.0}
    };

    // Allocate device memory
    thrust::complex<double> *d_a_real, *d_a_dual, *d_b_real, *d_b_dual;
    thrust::complex<double> *d_result_real, *d_result_dual;

    size_t real_size = rows * cols * sizeof(thrust::complex<double>);
    size_t dual_size_total = rows * cols * dual_size * sizeof(thrust::complex<double>);

    hipMalloc(&d_a_real, real_size);
    hipMalloc(&d_a_dual, dual_size_total);
    hipMalloc(&d_b_real, real_size);
    hipMalloc(&d_b_dual, dual_size_total);
    hipMalloc(&d_result_real, real_size);
    hipMalloc(&d_result_dual, dual_size_total);

    // Copy data to device
    hipMemcpy(d_a_real, a_real.data(), real_size, hipMemcpyHostToDevice);
    hipMemcpy(d_a_dual, a_dual.data(), dual_size_total, hipMemcpyHostToDevice);
    hipMemcpy(d_b_real, b_real.data(), real_size, hipMemcpyHostToDevice);
    hipMemcpy(d_b_dual, b_dual.data(), dual_size_total, hipMemcpyHostToDevice);

    // Launch kernel
    dim3 blockDim(256);
    dim3 gridDim((rows * cols * dual_size + blockDim.x - 1) / blockDim.x);
    MatrixDualSigncondKernel<<<1, blockDim>>>(d_a_real, d_a_dual, d_b_real, d_b_dual,
                                                   rows, cols, dual_size,
                                                   d_result_real, d_result_dual, 1.0e-6);

    // Copy results back to host
    std::vector<thrust::complex<double>> result_real(rows * cols);
    std::vector<thrust::complex<double>> result_dual(rows * cols * dual_size);

    hipMemcpy(result_real.data(), d_result_real, real_size, hipMemcpyDeviceToHost);
    hipMemcpy(result_dual.data(), d_result_dual, dual_size_total, hipMemcpyDeviceToHost);

    // Expected results
    std::vector<thrust::complex<double>> expected_real = {
        {1.0, 0.0}, {2.0, 0.0},
        {-3.0, 0.0}, {-4.0, 0.0}
    };
    std::vector<thrust::complex<double>> expected_dual = {
        {0.1, 0.0}, {0.2, 0.0},
        {-0.3, 0.0}, {-0.4, 0.0},
        {-0.5, 0.0}, {-0.6, 0.0},
        {0.7, 0.0}, {0.8, 0.0}
    };

    // Validate results
    for (int i = 0; i < rows * cols; ++i) {
        EXPECT_EQ(result_real[i], expected_real[i]) << "Mismatch in real part at index " << i;
    }

    for (int i = 0; i < rows * cols * dual_size; ++i) {
        EXPECT_EQ(result_dual[i], expected_dual[i]) << "Mismatch in dual part at index " << i;
    }

    // Free device memory
    hipFree(d_a_real);
    hipFree(d_a_dual);
    hipFree(d_b_real);
    hipFree(d_b_dual);
    hipFree(d_result_real);
    hipFree(d_result_dual);
}


// ---------------------------------------------------------------------------
// Google Test: MatrixHyperDualTest
// ---------------------------------------------------------------------------
TEST(MatrixHyperDualTest, ElementwiseAdd_2x2_dualsize2)
{
    using T = double;

    // Matrix dimensions
    int rows = 2;
    int cols = 2;
    int dual_size = 2;

    // total_real = number of matrix entries
    int total_real  = rows * cols;             // = 4
    // total_dual = number of first-order partials
    int total_dual  = rows * cols * dual_size; // = 8
    // total_hyper = number of second-order partials
    int total_hyper = rows * cols * dual_size * dual_size; // = 16

    // ---------------------------------------------------------------------
    // 1) Prepare host data
    // ---------------------------------------------------------------------
    std::vector<thrust::complex<T>> a_real(total_real), b_real(total_real);
    for (int i = 0; i < total_real; ++i) {
        a_real[i] = thrust::complex<T>(static_cast<T>(i+1), static_cast<T>(i+1));
        b_real[i] = thrust::complex<T>(static_cast<T>(i+5), static_cast<T>(i+5));
    }

    std::vector<thrust::complex<T>> a_dual(total_dual), b_dual(total_dual);
    for (int i = 0; i < total_dual; ++i) {
        a_dual[i] = thrust::complex<T>(i + 0.1, i + 0.2);
        b_dual[i] = thrust::complex<T>(10*(i+1), 10*(i+1));
    }

    std::vector<thrust::complex<T>> a_hyper(total_hyper), b_hyper(total_hyper);
    for (int i = 0; i < total_hyper; ++i) {
        a_hyper[i] = thrust::complex<T>(2.0*i, 2.0*i);
        b_hyper[i] = thrust::complex<T>(3.0*i, -1.0*i);
    }

    // ---------------------------------------------------------------------
    // 2) Allocate device arrays & copy
    // ---------------------------------------------------------------------
    thrust::complex<T> *d_a_real, *d_b_real,
                       *d_a_dual, *d_b_dual,
                       *d_a_hyper, *d_b_hyper;
    thrust::complex<T> *d_result_real, *d_result_dual, *d_result_hyper;

    AllocateAndCopy(a_real,   &d_a_real);
    AllocateAndCopy(b_real,   &d_b_real);
    AllocateAndCopy(a_dual,   &d_a_dual);
    AllocateAndCopy(b_dual,   &d_b_dual);
    AllocateAndCopy(a_hyper,  &d_a_hyper);
    AllocateAndCopy(b_hyper,  &d_b_hyper);

    hipMalloc(&d_result_real,  total_real  * sizeof(thrust::complex<T>));
    // We allocate "total_hyper" for result_dual to avoid any boundary issues,
    // though only the first "total_dual" entries matter.
    hipMalloc(&d_result_dual,  total_hyper * sizeof(thrust::complex<T>));
    hipMalloc(&d_result_hyper, total_hyper * sizeof(thrust::complex<T>));

    // ---------------------------------------------------------------------
    // 3) Launch the kernel
    // ---------------------------------------------------------------------
    int blockSize = 128;
    int gridSize  = (total_hyper + blockSize - 1) / blockSize;
    MatrixHyperDualElementwiseAddKernel<T><<<gridSize, blockSize>>>(
        d_a_real, d_a_dual, d_a_hyper,
        d_b_real, d_b_dual, d_b_hyper,
        rows, cols, dual_size,
        d_result_real, d_result_dual, d_result_hyper
    );
    hipDeviceSynchronize();

    // ---------------------------------------------------------------------
    // 4) Copy results back to host
    // ---------------------------------------------------------------------
    auto res_real  = CopyToHost(d_result_real,  total_real);
    auto res_dual  = CopyToHost(d_result_dual,  total_hyper);  // read back 16
    auto res_hyper = CopyToHost(d_result_hyper, total_hyper);

    // Clean up device memory
    hipFree(d_a_real);
    hipFree(d_b_real);
    hipFree(d_a_dual);
    hipFree(d_b_dual);
    hipFree(d_a_hyper);
    hipFree(d_b_hyper);
    hipFree(d_result_real);
    hipFree(d_result_dual);
    hipFree(d_result_hyper);

    // ---------------------------------------------------------------------
    // 5) Verify correctness
    // ---------------------------------------------------------------------
    // The kernel logic does:
    //   if (k=0 && l=0) => result_real[off] = a_real[off] + b_real[off]
    //   if (l=0) => result_dual[idx/dual_size] = a_dual[idx/dual_size] + b_dual[idx/dual_size]
    //   result_hyper[idx] = a_hyper[idx] + b_hyper[idx]
    //
    // So the result_dual array is assigned for exactly "dual_size * rows*cols" threads 
    // (the ones where l=0). We'll check only the first total_dual entries.

    // -- Real part (4 elements)
    for (int i = 0; i < total_real; ++i) {
        thrust::complex<T> expected = a_real[i] + b_real[i];
        EXPECT_EQ(res_real[i], expected) << "Real mismatch at i=" << i;
    }

    // -- Dual part (8 elements) 
    //   The kernel uses half as many threads (l=0) to store these. 
    //   We only verify the first 8 entries in res_dual, which should match a_dual + b_dual.
    for (int i = 0; i < total_dual; ++i) {
        thrust::complex<T> expected = a_dual[i] + b_dual[i];
        EXPECT_EQ(res_dual[i], expected) << "Dual mismatch at i=" << i;
    }

    // -- Hyper part (16 elements)
    //   Always a_hyper[idx] + b_hyper[idx], for idx in [0..15].
    for (int i = 0; i < total_hyper; ++i) {
        thrust::complex<T> expected = a_hyper[i] + b_hyper[i];
        EXPECT_EQ(res_hyper[i], expected) << "Hyper mismatch at i=" << i;
    }
}


// ---------------------------------------------------------------------------
// Test: Multiply two 2x2 hyper-dual matrices (dual_size=2) elementwise
// ---------------------------------------------------------------------------
TEST(MatrixHyperDualTest, ElementwiseMul_2x2_dualsize2)
{
    using T = double;

    // Dimensions
    int rows = 2;
    int cols = 2;
    int dual_size = 2;

    // Sizes
    int total_real  = rows * cols;             // 4
    int total_dual  = rows * cols * dual_size; // 8
    int total_hyper = rows * cols * dual_size * dual_size; // 16

    // 1) Prepare host data for A and B
    //    a_real, a_dual, a_hyper; b_real, b_dual, b_hyper
    std::vector<thrust::complex<T>> a_real(total_real), b_real(total_real);
    std::vector<thrust::complex<T>> a_dual(total_dual), b_dual(total_dual);
    std::vector<thrust::complex<T>> a_hyper(total_hyper), b_hyper(total_hyper);

    // Fill with some patterns
    // real: (1,1), (2,2), (3,3), (4,4) for "A"
    //       (5,5), (6,6), (7,7), (8,8) for "B"
    for (int i = 0; i < total_real; ++i) {
        a_real[i] = thrust::complex<T>(i+1, i+1);
        b_real[i] = thrust::complex<T>(i+5, i+5);
    }

    // dual: for A => a_dual[i] = (i+0.1, i+0.2)
    //       for B => b_dual[i] = (10(i+1), 10(i+1))
    for (int i = 0; i < total_dual; ++i) {
        a_dual[i] = thrust::complex<T>(i + 0.1, i + 0.2);
        b_dual[i] = thrust::complex<T>(10.0 * (i+1), 10.0 * (i+1));
    }

    // hyper: for A => (2i, 2i)
    //         for B => (3i, -i)
    for (int i = 0; i < total_hyper; ++i) {
        a_hyper[i] = thrust::complex<T>(2.0*i, 2.0*i);
        b_hyper[i] = thrust::complex<T>(3.0*i, -1.0*i);
    }

    // 2) Allocate device memory & copy
    thrust::complex<T>* d_a_real;    thrust::complex<T>* d_b_real;
    thrust::complex<T>* d_a_dual;    thrust::complex<T>* d_b_dual;
    thrust::complex<T>* d_a_hyper;   thrust::complex<T>* d_b_hyper;
    thrust::complex<T>* d_c_real;    thrust::complex<T>* d_c_dual; 
    thrust::complex<T>* d_c_hyper;

    AllocateAndCopy(a_real,   &d_a_real);
    AllocateAndCopy(b_real,   &d_b_real);
    AllocateAndCopy(a_dual,   &d_a_dual);
    AllocateAndCopy(b_dual,   &d_b_dual);
    AllocateAndCopy(a_hyper,  &d_a_hyper);
    AllocateAndCopy(b_hyper,  &d_b_hyper);

    hipMalloc(&d_c_real,  total_real * sizeof(thrust::complex<T>));
    hipMalloc(&d_c_dual,  total_hyper * sizeof(thrust::complex<T>));
    hipMalloc(&d_c_hyper, total_hyper * sizeof(thrust::complex<T>));

    // 3) Launch the kernel
    int blockSize = 128;
    int gridSize  = (total_hyper + blockSize - 1) / blockSize;
    MatrixHyperDualElementwiseMulKernel<T><<<gridSize, blockSize>>>(
        d_a_real, d_a_dual, d_a_hyper,
        d_b_real, d_b_dual, d_b_hyper,
        rows, cols, dual_size,
        d_c_real, d_c_dual, d_c_hyper
    );
    hipDeviceSynchronize();

    // 4) Copy results back to host
    auto c_real  = CopyToHost(d_c_real,  total_real);
    auto c_dual  = CopyToHost(d_c_dual,  total_hyper);
    auto c_hyper = CopyToHost(d_c_hyper, total_hyper);

    // Free device memory
    hipFree(d_a_real);
    hipFree(d_b_real);
    hipFree(d_a_dual);
    hipFree(d_b_dual);
    hipFree(d_a_hyper);
    hipFree(d_b_hyper);
    hipFree(d_c_real);
    hipFree(d_c_dual);
    hipFree(d_c_hyper);

    // 5) Compute reference on host & compare
    //    We'll do a small nested loop:
    //    c_real(off) = a_real(off)*b_real(off)
    //    c_dual(off,k) = a_real(off)*b_dual(off,k) + b_real(off)*a_dual(off,k)
    //    c_hyper(off,k,l) = ...
    // 
    //    Where:
    //      off = i*cols + j
    //      dual_off(i,j,k)  = i*cols*dual_size + j*dual_size + k
    //      hyper_off(i,j,k,l)= i*cols*dual_size*dual_size + j*dual_size*dual_size + k*dual_size + l

    std::vector<thrust::complex<T>> ref_real(total_real);
    std::vector<thrust::complex<T>> ref_dual(total_dual);
    std::vector<thrust::complex<T>> ref_hyper(total_hyper);

    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            int off = i*cols + j;
            // real
            auto ar = a_real[off];
            auto br = b_real[off];
            ref_real[off] = ar * br;

            // dual
            for (int k = 0; k < dual_size; ++k) {
                int dual_off = i*cols*dual_size + j*dual_size + k;
                auto ad = a_dual[dual_off];
                auto bd = b_dual[dual_off];
                ref_dual[dual_off] = ar * bd + br * ad;
            }

            // hyper
            for (int k = 0; k < dual_size; ++k) {
                for (int l = 0; l < dual_size; ++l) {
                    int hyper_off = i*cols*dual_size*dual_size + j*dual_size*dual_size + k*dual_size + l;
                    auto ah = a_hyper[hyper_off];
                    auto bh = b_hyper[hyper_off];
                    auto adk = a_dual[i*cols*dual_size + j*dual_size + k];
                    auto bdl = b_dual[i*cols*dual_size + j*dual_size + l];

                    // c_hyper = a_real*b_hyper + b_real*a_hyper + a_dual[k]*b_dual[l]
                    ref_hyper[hyper_off] = (ar * bh) + (br * ah) + (adk * bdl);
                }
            }
        }
    }

    // 6) Compare c_* vs. ref_*
    // real
    for (int i = 0; i < total_real; ++i) {
        EXPECT_NEAR(c_real[i].real(), ref_real[i].real(), 1.0e-6) << "Mismatch in real part at i=" << i;
        EXPECT_NEAR(c_real[i].imag(), ref_real[i].imag(), 1.0e-6) << "Mismatch in real part at i=" << i;
    }
    // dual
    // We only wrote "dual_size" threads per matrix entry, but we stored them in c_dual 
    // across "total_hyper" allocated space. In the kernel code, we used 
    //   result_dual[off_dual_k], 
    // so only the first "total_dual" elements matter.
    for (int i = 0; i < total_dual; ++i) {
        EXPECT_NEAR(c_dual[i].real(), ref_dual[i].real(), 1.0e-6) << "Mismatch in dual part at i=" << i;
        EXPECT_NEAR(c_dual[i].imag(), ref_dual[i].imag(), 1.0e-6) << "Mismatch in dual part at i=" << i;
    }
    // hyper
    for (int i = 0; i < total_hyper; ++i) {
        EXPECT_NEAR(c_hyper[i].real(), ref_hyper[i].real(), 1.0e-6) << "Mismatch in hyper part at i=" << i;
        EXPECT_NEAR(c_hyper[i].imag(), ref_hyper[i].imag(), 1.0e-6) << "Mismatch in hyper part at i=" << i;
    }
}

// Add more tests for IndexGet, IndexPut, ElementwiseMultiply, Square, Pow, and Sqrt similarly.
// Main entry point for Google Test
int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}