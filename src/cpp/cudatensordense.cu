#include "hip/hip_runtime.h"
#ifndef _CU_DUAL_TENSOR_HPP
#define _CU_DUAL_TENSOR_HPP
#include <hip/hip_runtime.h>
#include <iostream>
#include <complex>
//Utility class to implement dual tensor operations only necessary for QR decomposition
//This is a simplified version of the more extensive Dual class in the original codebase
//and it is implemented using cuBLAS and cuSPARSE for matrix operations
#include <hipblas.h>
#include <hipsparse.h>
#include <thrust/device_vector.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/iterator_traits.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <memory>
#include <vector>
#include <complex>
#include <thrust/device_ptr.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/complex.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/device_vector.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdexcept>
#include <iostream>
#include <cmath> // for fabs


namespace janus {

    class VectorBool {
    private:

    public:
        int size_;       // Vector length
        bool* data_;     // Pointer to device memory


        __device__ VectorBool() : size_(0), data_(nullptr) {}

        // Constructor: Initializes the wrapper with device memory
        __device__ VectorBool(bool* data, int size)
            : size_(size), data_(data) {}

        // Destructor: No action needed since memory is managed externally
        __device__ ~VectorBool() = default;

        // Disable copy constructor to avoid unintended copying
        __device__ VectorBool(const VectorBool&) = delete;

        __device__ VectorBool(VectorBool&& other) noexcept 
            : size_(other.size_), data_(other.data_) {}

        __device__ VectorBool& operator=(VectorBool&& other) noexcept {
            if (this != &other) {
                size_ = other.size_;
                data_ = other.data_;
            }
            return *this;
        }


    };
    
    // Set values in range
    __device__ void boolIndexPut(bool* input, int start, int end, const bool* subvector) {
          int idx = threadIdx.x + blockIdx.x * blockDim.x;
          int range = end - start;

          if (idx < range) {
            input[start + idx] = subvector[idx];
          }
    }


    //Wrapper functions
    __device__ void boolIndexGet(bool* input, int start, int end, bool* result) {
        int idx = threadIdx.x + blockIdx.x * blockDim.x;
        if (idx >= start && idx < end) {
                result[idx - start] = input[idx];
        }    
    }

    __global__ void boolIndexGetKernel(bool* input, int start, int end, bool* result) {
        boolIndexGet(input, start, end, result);
    }

    __global__ void boolIndexPutKernel(bool* input, int start, int end, bool* subvector) {
        boolIndexPut(input, start, end, subvector);
    }





    template <typename T>
    class Vector {
    public:
        int size_;                    // Vector length
        thrust::complex<T>* data_;  // Data pointer
    };
    
    template <typename T>
    __device__ void VectorElementwiseAdd(thrust::complex<T>* a, 
                                         thrust::complex<T>* b, 
                                         int size, 
                                         thrust::complex<T>* result)  {
        // Calculate global thread index
        int idx = threadIdx.x + blockIdx.x * blockDim.x;
        // Boundary check to prevent out-of-bounds access
        if (idx >= size) return;

        // Perform elementwise addition
        result[idx] = a[idx] + b[idx];

    }
    
    template <typename T>
    __global__ void VectorElementwiseAddKernel(thrust::complex<T>* a, 
                                               thrust::complex<T>* b,
                                               int size, 
                                               thrust::complex<T>* result) {
        VectorElementwiseAdd(a, b, size, result);
    }


    /**
     * @brief Computes a tensor result based on the signs of two input tensors.
     *
     * The function processes two input tensors, `a` and `b`, and modifies `a` based
     * on the sign of its real part (`a_sign`) and the sign of the real part of `b` (`b_sign`).
     * Small values are treated as positive to ensure compatibility with MATLAB.
     *
     * @param a A tensor representing the input tensor `a`.
     * @param b A tensor representing the input tensor `b`.
     * @return A tensor modified based on the conditional logic defined by the signs of `a` and `b`.
     *
     * @details 
     * The function follows these steps:
     * - Computes the signs of the real parts of `a` and `b` using `custom_sign`.
     * - Handles small numbers by assuming their sign is positive.
     * - Applies the following conditional logic to compute the result tensor:
     *   \f[
     *   \text{result} = 
     *   \begin{cases} 
     *   a, & \text{if } b_{\text{sign}} \geq 0 \text{ and } a_{\text{sign}} \geq 0 \\
     *   -a, & \text{if } b_{\text{sign}} \geq 0 \text{ and } a_{\text{sign}} < 0 \\
     *   -a, & \text{if } b_{\text{sign}} < 0 \text{ and } a_{\text{sign}} \geq 0 \\
     *   a, & \text{if } b_{\text{sign}} < 0 \text{ and } a_{\text{sign}} < 0
     *   \end{cases}
     *   \f]
     * - Returns the computed tensor `result`.
     *
     * @note Debug information for `a_sign` and `b_sign` is printed to `std::cerr`.
     * @note The function ensures compatibility with MATLAB-like behavior for very small numbers.
     */
    template <typename T>
    __device__ void VectorSigncond(thrust::complex<T>* a, 
                                   thrust::complex<T>* b,
                                   int size,  
                                   thrust::complex<T>* result, double tol = 1.0e-6) {
        int idx = threadIdx.x + blockIdx.x * blockDim.x;

        // Ensure we are within bounds
        if (idx >= size) return;
        auto aidx = a[idx];
        auto bidx = b[idx];
        // Retrieve the real parts of `a` and `b`
        T a_real = aidx.real();
        T b_real = bidx.real();

        // Compute the sign of the real parts with tolerance
        int a_sign = (fabs(static_cast<double>(a_real)) >= tol) ? (a_real >= 0 ? 1 : -1) : 1;
        int b_sign = (fabs(static_cast<double>(b_real)) >= tol) ? (b_real >= 0 ? 1 : -1) : 1;

        // Apply the conditional logic for result computation
        if (b_sign >= 0) {
            result[idx] = (a_sign >= 0) ? aidx : -aidx;
        } else {
            result[idx] = (a_sign >= 0) ? -aidx : aidx;
        }
    }

    template <typename T>
    __global__ void VectorSigncondKernel(thrust::complex<T>* a, 
                                         thrust::complex<T>* b, 
                                         int size, 
                                         thrust::complex<T>* result, double tol) {
        VectorSigncond(a, b, size, result, tol);
    }



    /**
     * Add an square() method that squares the complex numbers in the vector. 
     */
    template <typename T>
    __device__ void VectorSquare(const thrust::complex<T>* input, 
                                 int size, 
                                 thrust::complex<T>* result) {
        int idx = threadIdx.x + blockIdx.x * blockDim.x;

        // Ensure the thread is within bounds
        if (idx >= size) return;

        // Compute the square of the complex number
        result[idx] = input[idx] * input[idx];
    }

    //Create a global kernel for the square method
    template <typename T>
    __global__ void VectorSquareKernel(const thrust::complex<T>* input, 
                                       int size, 
                                       thrust::complex<T>* result) {
        VectorSquare(input, size, result);
    }

    /**
     * Multiply each element in the vector by a scalar.
     */
    template <typename T>
    __device__ void VectorScalarMultiply(const thrust::complex<T>*input, 
                                         T scalar,
                                         int size, 
                                         thrust::complex<T>*result) {
        int idx = threadIdx.x + blockIdx.x * blockDim.x;

        // Ensure the thread is within bounds
        if (idx >= size) return;

        // Compute the product of the complex number and the scalar
        result[idx] = input[idx] * scalar;
    }

    /**
     * Create a global kernel for the scalar multiply method
     */
    template <typename T>
    __global__ void VectorScalarMultiplyKernel(const thrust::complex<T>* input, 
                                               T scalar, 
                                               int size, 
                                               thrust::complex<T>* result) {
        VectorScalarMultiply(input, scalar, size, result);
    }

    /**
     * Take the reciprocal of each element in the vector.
     */
    template <typename T>
    __device__ void VectorReciprocal(const thrust::complex<T>*input, 
                                     int size,
                                     thrust::complex<T>* result) {
        int idx = threadIdx.x + blockIdx.x * blockDim.x;

        // Ensure the thread is within bounds
        if (idx >= size) return;

        // Compute the reciprocal of the complex number
        result[idx] = 1.0 / input[idx];
    }

    /**
     * Create a global kernel for the reciprocal method
     */
    template <typename T>
    __global__ void VectorReciprocalKernel(const thrust::complex<T>* input, 
                                           int size, 
                                           thrust::complex<T>* result) {
        VectorReciprocal(input, size, result);
    }


    /**
     * Elementise multiplication of two vectors.
     */
    template <typename T>
    __device__ void VectorElementwiseMultiply(const thrust::complex<T>* a, 
                                              const thrust::complex<T>* b, 
                                              int size, 
                                              thrust::complex<T>* result) {
        int idx = threadIdx.x + blockIdx.x * blockDim.x;

        // Ensure the thread is within bounds
        if (idx >= size) return;

        // Perform elementwise multiplication
        result[idx] = a[idx] * b[idx];
    }

    /**
     * Create a global kernel for the elementwise multiplication method
     */
    template <typename T>
    __global__ void VectorElementwiseMultiplyKernel(const thrust::complex<T>* a, 
                                                    const thrust::complex<T>* b, 
                                                    int size, 
                                                    thrust::complex<T>* result) {
        VectorElementwiseMultiply(a, b, size, result);
    }


    /**
     * Elementwise sqrt
     */
    template <typename T>
    __device__ void VectorSqrt(const thrust::complex<T>*a, 
                               int size, 
                               thrust::complex<T>*result) {
        int idx = threadIdx.x + blockIdx.x * blockDim.x;

        // Ensure the thread is within bounds
        if (idx >= size) return;

        // Perform elementwise sqrt
        result[idx] = thrust::sqrt(a[idx]);
    }

    /**
     * Create a global kernel for the sqrt method
     */
    template <typename T>
    __global__ void VectorSqrtKernel(const thrust::complex<T>* a, 
                                     int size, 
                                     thrust::complex<T>* result) {
        VectorSqrt(a, size, result);
    }

    /**
     * Implement the pow method
     */
    template <typename T>
    __device__ void VectorPow(const thrust::complex<T>* a, 
                              T power, 
                              int size,
                              thrust::complex<T>* result) {
        int idx = threadIdx.x + blockIdx.x * blockDim.x;

        // Ensure the thread is within bounds
        if (idx >= size) return;

        // Perform elementwise power using CUDA's pow function
        result[idx] = pow(a[idx], power);
    }

    /**
     * Create a global kernel for the pow method
     */
    template <typename T>
    __global__ void VectorPowKernel(const thrust::complex<T>* a, 
                                    T power, 
                                    int size, 
                                    thrust::complex<T>* result) {
        VectorPow(a, power, size, result);
    }

    /**
     * Sum all the elements of the vector into a single complex number.
     */
    template <typename T>
    __device__ void VectorReduce(const thrust::complex<T>* a, 
                                 int size, 
                                 thrust::complex<T>* result) {
        extern __shared__ thrust::complex<T> shared_data[];

        int tid = threadIdx.x;
        int idx = threadIdx.x + blockIdx.x * blockDim.x;

        shared_data[tid] = (idx < size) ? a[idx] : thrust::complex<T>(0, 0);
        __syncthreads();

        for (int s = blockDim.x / 2; s > 0; s >>= 1) {
            if (tid < s) {
                shared_data[tid] += shared_data[tid + s];
            }
            __syncthreads();
        }

        if (tid == 0) {
            result[blockIdx.x] = shared_data[0];
        }
    }

    /**
     * Create a global kernel for the reduce method
     */
    template <typename T>
    __global__ void VectorReduceKernel(const thrust::complex<T>* a, 
                                       int size, 
                                       thrust::complex<T>* result) {
        VectorReduce(a, size, result);
    }

    /** Retrieve elements from start to end into a new Vector 
     * This is a device function
     */
    template <typename T>
    __device__ void VectorIndexGet(const thrust::complex<T>* a, 
                                   int start, 
                                   int end, 
                                   int size, 
                                   thrust::complex<T>* result) {
        int idx = threadIdx.x + blockIdx.x * blockDim.x;

        // Ensure the thread is within bounds
        if (idx >= size) return;

        // Copy the elements from start to end
        if (idx >= start && idx < end) {
            result[idx - start] = a[idx];
        }
    }

    //Create a global kernel for the index get method
    template <typename T>
    __global__ void VectorIndexGetKernel(const thrust::complex<T>* a, 
                                         int start, 
                                         int end, 
                                         int size, 
                                         thrust::complex<T>* result) {
        VectorIndexGet(a, start, end, size, result);
    }


    /**
     * IndexPut for Vector
     * Put the elements from start to end from input Vector into the result vector
     */
    template<typename T>
    __device__ void VectorIndexPut(const thrust::complex<T>* input, 
                                   int start, 
                                   int end, 
                                   int size, 
                                   thrust::complex<T>* result) {
        int idx = threadIdx.x + blockIdx.x * blockDim.x;

        // Ensure the thread is within bounds
        if (idx >= size) return;

        // Copy the elements from start to end
        if (idx >= start && idx < end) {
            result[idx] = input[idx - start];
        }
    }


    //Create a global kernel for the index put method
    template <typename T>
    __global__ void VectorIndexPutKernel(const thrust::complex<T>* input, 
                                         int start, 
                                         int end, 
                                         int size, 
                                         thrust::complex<T>* result) {
        VectorIndexPut(input, start, end, size, result);
    }
    


    /**
     * Dual tensor class
     */
    template <typename T>
    class VectorDual {
    public:
        int real_size_;                    // Vector length
        int dual_size_;               // Dual dimension
        thrust::complex<T>* real_;   // Real part
        thrust::complex<T>* dual_;   // Dual part
    };
     
    template <typename T>
    __device__ void VectorRealDualProduct(const VectorDual<T>& a, 
                                 const VectorDual<T>& b, 
                                 VectorDual<T>& result) {
        int idx = threadIdx.x + blockIdx.x * blockDim.x;

        // Ensure the thread is within bounds
        if (idx >= a.real_size_*a.dual_size_) return;

        // Perform outer multiplication
        int i = idx / a.dual_size_;
        if (i ==0) {  
            result.real_[i] = a.real_[i] * b.real_[i];
        }
        int j = idx % a.real_size_;
        result.dual_[j] = a.real_[i] * b.dual_[j] + b.real_[i] * a.dual_[j];
    }



    /**
     * IndexGet for VectorDual
     * Given a VectorDual and a range, return a new VectorDual with the elements in the range 
     */
    template <typename T>
    __device__ void VectorDualIndexGet(const VectorDual<T>& input,   
                                       int start, 
                                       int end,
                                       VectorDual<T>& result) {
        // Get the real part
        VectorIndexGet(input.real, start, end, input.real_size_, result.real);

        // Get the dual part
        VectorIndexGet(input.dual, start*input.dual_size_, end*input.dual_size_, 
                       (end-start-1)*input.dual_size_, result.dual_);
    }

    /**
     * IndexPut for VectorDual
     */
    template <typename T>
    __device__ void VectorDualIndexPut(const VectorDual<T>& input, 
                                       int start, 
                                       int end, 
                                       VectorDual<T>& result) {
        int real_size = input.real_size_;
        int dual_size = input.dual_size_;
        // Put the real part
        VectorIndexPut(input.real_, start, end, real_size, result.real_);

        // Put the dual part
        VectorIndexPut(input.dual_, start*dual_size, end*dual_size, real_size*dual_size, result.dual_);
    }

    /**
     * Elementwise addition of two VectorDual tensors
     */
    template <typename T>
    __device__ void VectorDualElementwiseAdd(const VectorDual<T>& a, 
                                             const VectorDual<T>& b, 
                                             VectorDual<T>& result) {
        // Perform elementwise addition of the real part
        VectorElementwiseAdd(a.real_, b.real_, a.real_size_, result.real_);

        // Perform elementwise addition of the dual part
        VectorElementwiseAdd(a.dual_, b.dual_, a.size_*a.dual_size, result.dual_);
    }

    /**
     * Elementwise multiplication of two VectorDual tensors
     */
    template <typename T>
    __device__ void VectorDualElementwiseMultiply(const VectorDual<T>& a, 
                                                  const VectorDual<T>& b,
                                                  VectorDual<T>& result) {
        // Perform elementwise multiplication of the real part
        VectorElementwiseMultiply(a.real_, b.real_, a.size_, result.real_);

        // Perform elementwise multiplication for the dual part
        VectorRealDualProduct(a.real_, b.dual_, a.real_size_, a.dual_size_, result.dual_);
        VectorRealDualProduct(b.real_, a.dual_, b.real_size_, b.dual_size_, result.dual_);
    }

    /**
     * Square each element in the VectorDual tensor
     */
    template <typename T>
    __device__ void VectorDualSquare(VectorDual<T>& input, 
                                     VectorDual<T>& result) {
        VectorDualElementwiseMultiply(input, input, result);
    }

    /**
     * Sqrt each element in the VectorDual tensor
     */
    template <typename T>
    __device__ void VectorDualSqrt(VectorDual<T>& input,
                                   VectorDual<T>& work, //Intermediate storage 
                                   VectorDual<T>& result) {
        // Perform elementwise sqrt for the real part
        VectorSqrt(input.real_, input.real_size_, result.real_);
        //The dual part is 0.5*input.real^(-0.5)*input.dual
        VectorPow(input.real_, -0.5, work.real_);
        VectorScalarMultiply(work.real_, 0.5, work.real_);
        VectorRealDualProduct(work.real_, input.dual_, result.dual_);
    }
} // namespace Janus
#endif // _CU_DUAL_TENSOR_HPP