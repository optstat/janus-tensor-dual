#include "hip/hip_runtime.h"
#ifndef _CU_DUAL_TENSOR_HPP
#define _CU_DUAL_TENSOR_HPP
#include <hip/hip_runtime.h>
#include <iostream>
#include <complex>
//Utility class to implement dual tensor operations only necessary for QR decomposition
//This is a simplified version of the more extensive Dual class in the original codebase
//and it is implemented using cuBLAS and cuSPARSE for matrix operations
#include <hipblas.h>
#include <hipsparse.h>
#include <thrust/device_vector.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/iterator_traits.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <memory>
#include <vector>


namespace janus {

template <typename T>
class VectorDense {
private:
    int batch_size_;  // Batch dimension (M)
    int size_;        // Vector length (N)

    T* data_;         // Real data [M, N]
    bool owns_memory_; // Indicates if memory is managed internally

    // cuBLAS handle
    hipblasHandle_t handle_;
    hipStream_t stream_;

public:
    // Constructor with external memory
    VectorDense(int batch_size, int size, T* data)
        : batch_size_(batch_size), size_(size), data_(data), owns_memory_(false) {
        if (!data_) {
            throw std::invalid_argument("Data pointer is null");
        }
        initializeHandles();
    }

    // Constructor with internal memory allocation
    VectorDense(int batch_size, int size)
        : batch_size_(batch_size), size_(size), owns_memory_(true) {
        if (batch_size <= 0 || size <= 0) {
            throw std::invalid_argument("Batch size and vector size must be positive.");
        }

        size_t data_size = batch_size * size * sizeof(T);
        if (hipMalloc(&data_, data_size) != hipSuccess) {
            throw std::runtime_error("Failed to allocate GPU memory for data.");
        }

        initializeHandles();
    }

    // Destructor
    ~VectorDense() {
        if (owns_memory_ && data_) {
            hipFree(data_);
        }
        if (handle_) {
            hipblasDestroy(handle_);
        }
        if (stream_) {
            hipStreamDestroy(stream_);
        }
    }

    // Initialize data from host
    void initialize(const T* host_data, size_t data_size) {
        if (data_size != batch_size_ * size_) {
            throw std::invalid_argument("Input size does not match vector dimensions.");
        }

        hipMemcpyAsync(data_, host_data, data_size * sizeof(T), hipMemcpyHostToDevice, stream_);
        hipStreamSynchronize(stream_);
    }

    // Elementwise addition
    VectorDense elementwiseAdd(const VectorDense& other) const {
        if (batch_size_ != other.batch_size_ || size_ != other.size_) {
            throw std::invalid_argument("Vector dimensions do not match for elementwise addition.");
        }

        VectorDense result(batch_size_, size_);

        int total_elements = batch_size_ * size_;
        thrust::device_ptr<T> d_ptr1(data_);
        thrust::device_ptr<T> d_ptr2(other.data_);
        thrust::device_ptr<T> d_ptr_result(result.data_);

        thrust::transform(
            d_ptr1,
            d_ptr1 + total_elements,
            d_ptr2,
            d_ptr_result,
            thrust::plus<T>());

        return result;
    }

    // Elementwise multiplication
    VectorDense elementwiseMultiply(const VectorDense& other) const {
        if (batch_size_ != other.batch_size_ || size_ != other.size_) {
            throw std::invalid_argument("Vector dimensions do not match for elementwise multiplication.");
        }

        VectorDense result(batch_size_, size_);

        int total_elements = batch_size_ * size_;

        // Perform elementwise multiplication
        thrust::transform(
            thrust::device_pointer_cast(data_),
            thrust::device_pointer_cast(data_ + total_elements),
            thrust::device_pointer_cast(other.data_),
            thrust::device_pointer_cast(result.data_),
            thrust::multiplies<T>());

        return result;
    }


    // Compute the sum of the vector along the size dimension
    VectorDense sum() const {
        VectorDense result(batch_size_, 1);

        // Allocate a vector of ones on the GPU
        T* ones;
        hipMalloc(&ones, size_ * sizeof(T));
        thrust::device_vector<T> host_ones(size_, static_cast<T>(1.0));
        hipMemcpy(ones, host_ones.data(), size_ * sizeof(T), hipMemcpyHostToDevice);

        // Use cuBLAS to compute the sum for each batch
        for (int b = 0; b < batch_size_; ++b) {
            hipblasSdot(handle_,             // cuBLAS handle
                    size_,               // Number of elements to sum
                    data_ + b * size_,   // Pointer to the batch data
                    1,                   // Stride for data_
                    ones,                // Pointer to vector of ones
                    1,                   // Stride for ones vector
                    result.data_ + b);   // Pointer to store the result
        }

        // Free the vector of ones
        hipFree(ones);

        return result;
    }



    VectorDense sign() const {
        VectorDense result(batch_size_, size_); // Create an output vector
        int total_elements = batch_size_ * size_; // Total number of elements

        // Use Thrust to apply sign element-wise
        thrust::transform(
            thrust::device_pointer_cast(data_),
            thrust::device_pointer_cast(data_ + total_elements),
            thrust::device_pointer_cast(result.data_),
            [] __device__ (T x) {
                return (x > 0) - (x < 0); // Sign computation
            });

        return result; // Return the result
    }


    static VectorDense where(const VectorDense& condition, 
                             const VectorDense& if_true, 
                             const VectorDense& if_false) {
        if (condition.batchSize() != if_true.batchSize() || condition.batchSize() != if_false.batchSize() ||
            condition.size() != if_true.size() || condition.size() != if_false.size()) {
            throw std::invalid_argument("Input dimensions must match.");
        }

        VectorDense result(condition.batchSize(), condition.size());
        int total_elements = condition.batchSize() * condition.size();

        // Use Thrust's zip iterator to combine condition, if_true, and if_false
        auto condition_begin = thrust::device_pointer_cast(condition.data());
        auto if_true_begin = thrust::device_pointer_cast(if_true.data());
        auto if_false_begin = thrust::device_pointer_cast(if_false.data());
        auto result_begin = thrust::device_pointer_cast(result.data());

        thrust::transform(
            thrust::make_zip_iterator(thrust::make_tuple(condition_begin, if_true_begin, if_false_begin)),
            thrust::make_zip_iterator(thrust::make_tuple(condition_begin + total_elements,
                                                        if_true_begin + total_elements,
                                                        if_false_begin + total_elements)),
            result_begin,
            [] __device__ (thrust::tuple<T, T, T> tup) {
                T cond = thrust::get<0>(tup);
                T a = thrust::get<1>(tup);
                T b = thrust::get<2>(tup);
                return cond ? a : b;
            });

        return result;
    }

    static VectorDense zeros_like(const VectorDense& other) {
        VectorDense result(other.batchSize(), other.size());
        hipMemset(result.data(), 0, other.batchSize() * other.size() * sizeof(T));
        return result;
    }

    VectorDense custom_sign(const VectorDense& input, double threshold = 1e-6) 
    {
        auto mask = input.abs() > threshold;
        auto sign_vector = input.sign();
        auto output = VectorDense::where(mask, sign_vector, VectorDense::zeros_like(input));
        return output;
    }
    

    // Accessors
    T* data() { return data_; }
    const T* data() const { return data_; }
    int batchSize() const { return batch_size_; }
    int size() const { return size_; }

private:
    // Initialize cuBLAS handle and CUDA stream
    void initializeHandles() {
        if (hipblasCreate(&handle_) != HIPBLAS_STATUS_SUCCESS) {
            throw std::runtime_error("Failed to create cuBLAS handle.");
        }
        if (hipStreamCreate(&stream_) != hipSuccess) {
            hipblasDestroy(handle_);
            throw std::runtime_error("Failed to create CUDA stream.");
        }
        hipblasSetStream(handle_, stream_);
    }
};


template <typename T>
class VectorDualDense {
private:
    using ComplexT = std::complex<T>;

    // Dimensions
    int batch_size_;  // M
    int size_;        // N (length of each vector)
    int dual_dim_;    // D (number of dual components)

    // Primal and Dual Data
    ComplexT* primal_data_;  // [M, N]
    ComplexT* dual_data_;    // [M, N, D]
    bool owns_memory_;       // Indicates if memory is managed internally

    // cuBLAS handle
    hipblasHandle_t handle_;
    hipStream_t stream_;

public:
    // Constructor with external memory
    VectorDualDense(int batch_size, int size, int dual_dim, ComplexT* primal_data, ComplexT* dual_data)
        : batch_size_(batch_size), size_(size), dual_dim_(dual_dim),
          primal_data_(primal_data), dual_data_(dual_data), owns_memory_(false) {
        if (!primal_data_ || !dual_data_) {
            throw std::invalid_argument("Primal or dual data pointer is null");
        }
        initializeHandles();
    }

    // Constructor with internal memory allocation
    VectorDualDense(int batch_size, int size, int dual_dim)
        : batch_size_(batch_size), size_(size), dual_dim_(dual_dim), owns_memory_(true) {
        if (batch_size <= 0 || size <= 0 || dual_dim <= 0) {
            throw std::invalid_argument("All dimensions must be positive.");
        }

        size_t primal_size = batch_size * size * sizeof(ComplexT);
        size_t dual_size = batch_size * size * dual_dim * sizeof(ComplexT);

        if (hipMalloc(&primal_data_, primal_size) != hipSuccess) {
            throw std::runtime_error("Failed to allocate GPU memory for primal data.");
        }

        if (hipMalloc(&dual_data_, dual_size) != hipSuccess) {
            hipFree(primal_data_);
            throw std::runtime_error("Failed to allocate GPU memory for dual data.");
        }

        initializeHandles();
    }

    ~VectorDualDense() {
        if (owns_memory_) {
            if (primal_data_) hipFree(primal_data_);
            if (dual_data_) hipFree(dual_data_);
        }
        hipblasDestroy(handle_);
        hipStreamDestroy(stream_);
    }

    void initialize(const ComplexT* primal, const ComplexT* dual, size_t primal_size, size_t dual_size) {
        if (primal_size != batch_size_ * size_ ||
            dual_size != batch_size_ * size_ * dual_dim_) {
            throw std::invalid_argument("Input sizes do not match tensor dimensions.");
        }

        if (primal) {
            hipMemcpyAsync(primal_data_, primal, primal_size * sizeof(ComplexT), hipMemcpyHostToDevice, stream_);
        }
        if (dual) {
            hipMemcpyAsync(dual_data_, dual, dual_size * sizeof(ComplexT), hipMemcpyHostToDevice, stream_);
        }
        hipStreamSynchronize(stream_);
    }

    template <typename U>
    VectorDualDense<U> indexGet(int start_row, int end_row) const {
        // Validate row range
        if (start_row < 0 || end_row > batch_size_ || start_row >= end_row) {
            throw std::invalid_argument("Invalid row range for selection.");
        }

        // Calculate the number of rows in the selected range
        int selected_batch_size = end_row - start_row;

        // Calculate offsets in the primal and dual data
        ComplexT* selected_primal_data = primal_data_ + start_row * size_;
        ComplexT* selected_dual_data = dual_data_ + start_row * size_ * dual_dim_;

        // Create a new VectorDualDense instance sharing the data with the original
        return VectorDualDense<T>(selected_batch_size, size_, dual_dim_, selected_primal_data, selected_dual_data);
    }

    template <typename U>
    void indexPut(int start_row, int end_row, const VectorDualDense<U>& data) {
        // Validate row range
        if (start_row < 0 || end_row > batch_size_ || start_row >= end_row) {
            throw std::invalid_argument("Invalid row range for index_put.");
        }

        // Validate dimensions of the input data
        if (data.batch_size() != (end_row - start_row) || data.size() != size_ || data.dual_dim() != dual_dim_) {
            throw std::invalid_argument("Input data dimensions do not match the target range.");
        }

        // Compute offsets in the primal and dual tensors
        ComplexT* target_primal_data = primal_data_ + start_row * size_;
        ComplexT* target_dual_data = dual_data_ + start_row * size_ * dual_dim_;

        // Compute data sizes
        size_t primal_size = data.batch_size() * size_ * sizeof(ComplexT);
        size_t dual_size = data.batch_size() * size_ * dual_dim_ * sizeof(ComplexT);

        // Copy data for the primal part
        hipMemcpyAsync(target_primal_data, data.primal_data(), primal_size, hipMemcpyDeviceToDevice, stream_);

        // Copy data for the dual part
        hipMemcpyAsync(target_dual_data, data.dual_data(), dual_size, hipMemcpyDeviceToDevice, stream_);

        // Synchronize to ensure data transfer is complete
        hipStreamSynchronize(stream_);
    }

    VectorDualDense<T> elementwiseAdd(const VectorDualDense<T>& other) const {
        if (batch_size_ != other.batch_size_ || size_ != other.size_ || dual_dim_ != other.dual_dim_) {
            throw std::invalid_argument("Tensor dimensions do not match for elementwise addition.");
        }

        VectorDualDense<T> result(batch_size_, size_, dual_dim_);

        int total_primal_elements = batch_size_ * size_;
        int total_dual_elements = total_primal_elements * dual_dim_;

        // Perform elementwise addition for the primal part
        thrust::transform(
            thrust::device_pointer_cast(primal_data_),
            thrust::device_pointer_cast(primal_data_ + total_primal_elements),
            thrust::device_pointer_cast(other.primal_data_),
            thrust::device_pointer_cast(result.primal_data_),
            thrust::plus<ComplexT>());

        // Perform elementwise addition for the dual part
        thrust::transform(
            thrust::device_pointer_cast(dual_data_),
            thrust::device_pointer_cast(dual_data_ + total_dual_elements),
            thrust::device_pointer_cast(other.dual_data_),
            thrust::device_pointer_cast(result.dual_data_),
            thrust::plus<ComplexT>());

        return result;
    }

    VectorDualDense<T> elementwiseMultiply(const VectorDualDense<T>& other) const {
        if (batch_size_ != other.batch_size_ || size_ != other.size_ || dual_dim_ != other.dual_dim_) {
            throw std::invalid_argument("Tensor dimensions do not match for elementwise multiplication.");
        }

        VectorDualDense<T> result(batch_size_, size_, dual_dim_);

        int total_primal_elements = batch_size_ * size_;
        int total_dual_elements = total_primal_elements * dual_dim_;

        // Perform elementwise multiplication for primal part
        thrust::transform(
            thrust::device_pointer_cast(primal_data_),
            thrust::device_pointer_cast(primal_data_ + total_primal_elements),
            thrust::device_pointer_cast(other.primal_data_),
            thrust::device_pointer_cast(result.primal_data_),
            thrust::multiplies<ComplexT>());

        // Perform elementwise multiplication for dual part using the product rule
        thrust::for_each(
            thrust::counting_iterator<int>(0),
            thrust::counting_iterator<int>(total_dual_elements),
            [=] __device__(int idx) {
                int primal_idx = idx / dual_dim_;
                int dual_idx = idx % dual_dim_;

                result.dual_data_[idx] =
                    dual_data_[idx] * other.primal_data_[primal_idx] +
                    primal_data_[primal_idx] * other.dual_data_[idx];
            });

        return result;
    }

    VectorDualDense<T> elementwiseSquare() const {
        // Create result vector
        VectorDualDense<T> result(batch_size_, size_, dual_dim_);

        int total_primal_elements = batch_size_ * size_;
        int total_dual_elements = total_primal_elements * dual_dim_;

        // Elementwise square for primal part
        thrust::transform(
            thrust::device_pointer_cast(primal_data_),
            thrust::device_pointer_cast(primal_data_ + total_primal_elements),
            thrust::device_pointer_cast(result.primal_data_),
            [] __device__(ComplexT x) {
                return x * x;
            });

        // Elementwise square for dual part (product rule)
        thrust::for_each(
            thrust::counting_iterator<int>(0),
            thrust::counting_iterator<int>(total_dual_elements),
            [=] __device__(int idx) {
                int primal_idx = idx / dual_dim_;
                result.dual_data_[idx] = 2.0 * primal_data_[primal_idx] * dual_data_[idx];
            });

        return result;
    }


    VectorDualDense<T> sum() const {
        // Create result vector
        VectorDualDense<T> result(batch_size_, 1, dual_dim_);

        // Scalars for cuBLAS
        ComplexT alpha = ComplexT(1.0, 0.0);
        ComplexT beta = ComplexT(0.0, 0.0);

        // Primal part: Sum over real dimension (N)
        for (int b = 0; b < batch_size_; ++b) {
            hipblasZgemv(handle_,
                        HIPBLAS_OP_T,  // Transpose to sum over columns (rows in memory layout)
                        size_, 1,     // Rows = N, Columns = 1
                        &alpha,
                        primal_data_ + b * size_, size_,
                        thrust::raw_pointer_cast(thrust::make_constant_iterator(ComplexT(1.0, 0.0))), 1,
                        &beta,
                        result.primal_data_ + b, 1);  // Store in result
        }

        // Dual part: Sum over real dimension (N) for each dual dimension (D)
        for (int b = 0; b < batch_size_; ++b) {
            for (int d = 0; d < dual_dim_; ++d) {
                hipblasZgemv(handle_,
                            HIPBLAS_OP_T,  // Transpose to sum over columns (rows in memory layout)
                            size_, 1,     // Rows = N, Columns = 1
                            &alpha,
                            dual_data_ + b * size_ * dual_dim_ + d * size_, size_,
                            thrust::raw_pointer_cast(thrust::make_constant_iterator(ComplexT(1.0, 0.0))), 1,
                            &beta,
                            result.dual_data_ + b * dual_dim_ + d, 1);  // Store in result
            }
        }

        // Return the summed vector
        return result;
    }

    

private:
    void initializeHandles() {
        if (hipblasCreate(&handle_) != HIPBLAS_STATUS_SUCCESS) {
            throw std::runtime_error("Failed to create cuBLAS handle.");
        }
        if (hipStreamCreate(&stream_) != hipSuccess) {
            hipblasDestroy(handle_);
            throw std::runtime_error("Failed to create CUDA stream.");
        }
        hipblasSetStream(handle_, stream_);
    }

public:
    // Disable copy constructor and copy assignment
    VectorDualDense(const VectorDualDense&) = delete;
    VectorDualDense& operator=(const VectorDualDense&) = delete;

    // Enable move constructor and move assignment
    VectorDualDense(VectorDualDense&&) noexcept = default;
    VectorDualDense& operator=(VectorDualDense&&) noexcept = default;

    // Getters for dimensions
    int batch_size() const { return batch_size_; }
    int size() const { return size_; }
    int dual_dim() const { return dual_dim_; }

    // Getters for data pointers
    ComplexT* primal_data() { return primal_data_; }
    ComplexT* dual_data() { return dual_data_; }

    const ComplexT* primal_data() const { return primal_data_; }
    const ComplexT* dual_data() const { return dual_data_; }
};

template <typename T>
class VectorHyperDualDense {
private:
    using ComplexT = std::complex<T>;

    // Dimensions
    int batch_size_;   // M
    int size_;         // N (length of each vector)
    int dual_dim_;     // D (number of dual components)

    // Primal, Dual, and Hyper-Dual Data
    ComplexT* primal_data_;      // [M, N]
    ComplexT* dual_data_;        // [M, N, D]
    ComplexT* hyper_dual_data_;  // [M, N, D, D]
    bool owns_memory_;           // Indicates if memory is managed internally

    // cuBLAS handle
    hipblasHandle_t handle_;
    hipStream_t stream_;

public:
    // Constructor with external memory
    VectorHyperDualDense(int batch_size, int size, int dual_dim, ComplexT* primal_data, ComplexT* dual_data, ComplexT* hyper_dual_data)
        : batch_size_(batch_size), size_(size), dual_dim_(dual_dim),
          primal_data_(primal_data), dual_data_(dual_data), hyper_dual_data_(hyper_dual_data),
          owns_memory_(false) {
        if (!primal_data_ || !dual_data_ || !hyper_dual_data_) {
            throw std::invalid_argument("Primal, dual, or hyper-dual data pointer is null.");
        }
        initializeHandles();
    }

    // Constructor with internal memory allocation
    VectorHyperDualDense(int batch_size, int size, int dual_dim)
        : batch_size_(batch_size), size_(size), dual_dim_(dual_dim), owns_memory_(true) {
        if (batch_size <= 0 || size <= 0 || dual_dim <= 0) {
            throw std::invalid_argument("All dimensions must be positive.");
        }

        size_t primal_size = batch_size * size * sizeof(ComplexT);
        size_t dual_size = batch_size * size * dual_dim * sizeof(ComplexT);
        size_t hyper_dual_size = batch_size * size * dual_dim * dual_dim * sizeof(ComplexT);

        if (hipMalloc(&primal_data_, primal_size) != hipSuccess) {
            throw std::runtime_error("Failed to allocate GPU memory for primal data.");
        }

        if (hipMalloc(&dual_data_, dual_size) != hipSuccess) {
            hipFree(primal_data_);
            throw std::runtime_error("Failed to allocate GPU memory for dual data.");
        }

        if (hipMalloc(&hyper_dual_data_, hyper_dual_size) != hipSuccess) {
            hipFree(primal_data_);
            hipFree(dual_data_);
            throw std::runtime_error("Failed to allocate GPU memory for hyper-dual data.");
        }

        initializeHandles();
    }

    ~VectorHyperDualDense() {
        if (owns_memory_) {
            if (primal_data_) hipFree(primal_data_);
            if (dual_data_) hipFree(dual_data_);
            if (hyper_dual_data_) hipFree(hyper_dual_data_);
        }
        hipblasDestroy(handle_);
        hipStreamDestroy(stream_);
    }

    void initialize(const ComplexT* primal, const ComplexT* dual, const ComplexT* hyper_dual, size_t primal_size, size_t dual_size, size_t hyper_dual_size) {
        if (primal_size != batch_size_ * size_ ||
            dual_size != batch_size_ * size_ * dual_dim_ ||
            hyper_dual_size != batch_size_ * size_ * dual_dim_ * dual_dim_) {
            throw std::invalid_argument("Input sizes do not match tensor dimensions.");
        }

        if (primal) {
            hipMemcpyAsync(primal_data_, primal, primal_size * sizeof(ComplexT), hipMemcpyHostToDevice, stream_);
        }
        if (dual) {
            hipMemcpyAsync(dual_data_, dual, dual_size * sizeof(ComplexT), hipMemcpyHostToDevice, stream_);
        }
        if (hyper_dual) {
            hipMemcpyAsync(hyper_dual_data_, hyper_dual, hyper_dual_size * sizeof(ComplexT), hipMemcpyHostToDevice, stream_);
        }
        hipStreamSynchronize(stream_);
    }

    // Example elementwise addition
    VectorHyperDualDense<T> elementwiseAdd(const VectorHyperDualDense<T>& other) const {
        if (batch_size_ != other.batch_size_ || size_ != other.size_ || dual_dim_ != other.dual_dim_) {
            throw std::invalid_argument("Tensor dimensions do not match for elementwise addition.");
        }

        VectorHyperDualDense<T> result(batch_size_, size_, dual_dim_);

        int total_primal_elements = batch_size_ * size_;
        int total_dual_elements = total_primal_elements * dual_dim_;
        int total_hyper_dual_elements = total_dual_elements * dual_dim_;

        // Perform elementwise addition for the primal part
        thrust::transform(
            thrust::device_pointer_cast(primal_data_),
            thrust::device_pointer_cast(primal_data_ + total_primal_elements),
            thrust::device_pointer_cast(other.primal_data_),
            thrust::device_pointer_cast(result.primal_data_),
            thrust::plus<ComplexT>());

        // Perform elementwise addition for the dual part
        thrust::transform(
            thrust::device_pointer_cast(dual_data_),
            thrust::device_pointer_cast(dual_data_ + total_dual_elements),
            thrust::device_pointer_cast(other.dual_data_),
            thrust::device_pointer_cast(result.dual_data_),
            thrust::plus<ComplexT>());

        // Perform elementwise addition for the hyper-dual part
        thrust::transform(
            thrust::device_pointer_cast(hyper_dual_data_),
            thrust::device_pointer_cast(hyper_dual_data_ + total_hyper_dual_elements),
            thrust::device_pointer_cast(other.hyper_dual_data_),
            thrust::device_pointer_cast(result.hyper_dual_data_),
            thrust::plus<ComplexT>());

        return result;
    }

private:
    void initializeHandles() {
        if (hipblasCreate(&handle_) != HIPBLAS_STATUS_SUCCESS) {
            throw std::runtime_error("Failed to create cuBLAS handle.");
        }
        if (hipStreamCreate(&stream_) != hipSuccess) {
            hipblasDestroy(handle_);
            throw std::runtime_error("Failed to create CUDA stream.");
        }
        hipblasSetStream(handle_, stream_);
    }

public:
    // Disable copy constructor and copy assignment
    VectorHyperDualDense(const VectorHyperDualDense&) = delete;
    VectorHyperDualDense& operator=(const VectorHyperDualDense&) = delete;

    // Enable move constructor and move assignment
    VectorHyperDualDense(VectorHyperDualDense&&) noexcept = default;
    VectorHyperDualDense& operator=(VectorHyperDualDense&&) noexcept = default;

    // Getters for dimensions
    int batch_size() const { return batch_size_; }
    int size() const { return size_; }
    int dual_dim() const { return dual_dim_; }

    // Getters for data pointers
    ComplexT* primal_data() { return primal_data_; }
    ComplexT* dual_data() { return dual_data_; }
    ComplexT* hyper_dual_data() { return hyper_dual_data_; }

    const ComplexT* primal_data() const { return primal_data_; }
    const ComplexT* dual_data() const { return dual_data_; }
    const ComplexT* hyper_dual_data() const { return hyper_dual_data_; }
};


template <typename T>
class MatrixDense {
private:
    using ComplexT = std::complex<T>;

    // Dimensions
    int batch_size_;
    int rows_;
    int cols_;

    // Primal Data
    ComplexT* primal_data_; // Device-side primal part
    bool owns_memory_;      // Indicates if memory is managed internally

    // cuBLAS handle
    hipblasHandle_t handle_;
    hipStream_t stream_;

public:
    // Constructor with external memory
    MatrixDense(int batch_size, int rows, int cols, ComplexT* primal_data)
        : batch_size_(batch_size), rows_(rows), cols_(cols),
          primal_data_(primal_data), owns_memory_(false) {
        if (!primal_data_) {
            throw std::invalid_argument("Primal data pointer is null");
        }

        initializeHandles();
    }

    // Constructor with internal memory allocation
    MatrixDense(int batch_size, int rows, int cols)
        : batch_size_(batch_size), rows_(rows), cols_(cols), owns_memory_(true) {
        if (batch_size <= 0 || rows <= 0 || cols <= 0) {
            throw std::invalid_argument("All dimensions must be positive.");
        }

        size_t primal_size = batch_size * rows * cols * sizeof(ComplexT);

        if (hipMalloc(&primal_data_, primal_size) != hipSuccess) {
            throw std::runtime_error("Failed to allocate GPU memory for primal data.");
        }

        initializeHandles();
    }

    ~MatrixDense() {
        if (owns_memory_ && primal_data_) {
            hipFree(primal_data_);
        }
        hipblasDestroy(handle_);
        hipStreamDestroy(stream_);
    }

    void initialize(const ComplexT* primal, size_t primal_size) {
        if (primal_size != batch_size_ * rows_ * cols_) {
            throw std::invalid_argument("Input size does not match tensor dimensions.");
        }

        if (primal) {
            hipMemcpyAsync(primal_data_, primal, primal_size * sizeof(ComplexT), hipMemcpyHostToDevice, stream_);
        }
        hipStreamSynchronize(stream_);
    }

    template <typename U>
    MatrixDense<U> indexGet(int start_row, int end_row, int start_col, int end_col) const {
        // Validate row and column ranges
        if (start_row < 0 || end_row > rows_ || start_row >= end_row ||
            start_col < 0 || end_col > cols_ || start_col >= end_col) {
            throw std::invalid_argument("Invalid row or column range for indexGet.");
        }

        // Dimensions of the submatrix
        int sub_rows = end_row - start_row;
        int sub_cols = end_col - start_col;

        // Calculate the offset for the primal data
        ComplexT* sub_primal_data = primal_data_ + start_row * cols_ + start_col;

        // Create a new MatrixDense instance sharing the data with the original
        return MatrixDense<T>(batch_size_, sub_rows, sub_cols, sub_primal_data);
    }


    template <typename U>
    void indexPut(int start_row, int end_row, int start_col, int end_col, const MatrixDense<U>& data) {
        // Validate row and column ranges
        if (start_row < 0 || end_row > rows_ || start_row >= end_row ||
            start_col < 0 || end_col > cols_ || start_col >= end_col) {
            throw std::invalid_argument("Invalid row or column range for indexPut.");
        }

        // Validate dimensions of the input data
        if (data.rows_ != (end_row - start_row) || data.cols_ != (end_col - start_col) || 
            data.batch_size_ != batch_size_) {
            throw std::invalid_argument("Input data dimensions do not match the target range.");
        }

        // Calculate the offset for the primal data
        ComplexT* target_primal_data = primal_data_ + start_row * cols_ + start_col;

        // Update primal data
        for (int b = 0; b < batch_size_; ++b) {
            hipMemcpy2DAsync(target_primal_data + b * rows_ * cols_,
                            cols_ * sizeof(ComplexT),
                            data.primal_data_ + b * data.rows_ * data.cols_,
                            data.cols_ * sizeof(ComplexT),
                            data.cols_ * sizeof(ComplexT),
                            data.rows_,
                            hipMemcpyDeviceToDevice,
                            stream_);
        }

        // Synchronize to ensure data transfer is complete
        hipStreamSynchronize(stream_);
    }


    MatrixDense<T> multiply(const MatrixDense<T>& other) const {
        // Validate dimensions
        if (cols_ != other.rows_ || batch_size_ != other.batch_size_) {
            throw std::invalid_argument("Matrix dimensions do not match for multiplication.");
        }

        // Create the result matrix
        MatrixDense<T> result(batch_size_, rows_, other.cols_);

        // Scaling factors for cuBLAS
        ComplexT alpha = ComplexT(1.0, 0.0);
        ComplexT beta = ComplexT(0.0, 0.0);

        // Perform batched matrix multiplication
        hipblasStatus_t status = hipblasZgemmStridedBatched(
            handle_,
            HIPBLAS_OP_N,           // No transpose for this matrix
            HIPBLAS_OP_N,           // No transpose for the other matrix
            rows_,                 // Number of rows of the output matrix
            other.cols_,           // Number of columns of the output matrix
            cols_,                 // Shared dimension (this.cols_ == other.rows_)
            &alpha,                // Scaling factor for the multiplication
            primal_data_,          // Pointer to this matrix data
            rows_,                 // Leading dimension of this matrix
            rows_ * cols_,         // Stride between consecutive matrices in the batch
            other.primal_data_,    // Pointer to other matrix data
            other.rows_,           // Leading dimension of the other matrix
            other.rows_ * other.cols_, // Stride between consecutive matrices in the batch
            &beta,                 // Scaling factor for the result matrix
            result.primal_data_,   // Pointer to result matrix data
            result.rows_,          // Leading dimension of the result matrix
            result.rows_ * result.cols_, // Stride between consecutive matrices in the batch
            batch_size_            // Number of matrices in the batch
        );

        if (status != HIPBLAS_STATUS_SUCCESS) {
            throw std::runtime_error("cuBLAS matrix multiplication failed.");
        }

        return result;
    }

    VectorDense<T> matrixVectorProduct(const VectorDense<T>& vector) const {
        // Validate dimensions
        if (cols_ != vector.size() || batch_size_ != vector.batch_size()) {
            throw std::invalid_argument("Matrix and vector dimensions do not match for multiplication.");
        }

        // Create the result vector
        VectorDense<T> result(batch_size_, rows_);

        // Scaling factors for cuBLAS
        ComplexT alpha = ComplexT(1.0, 0.0);
        ComplexT beta = ComplexT(0.0, 0.0);

        // Perform batched matrix-vector multiplication
        for (int b = 0; b < batch_size_; ++b) {
            hipblasStatus_t status = hipblasZgemv(
                handle_,
                HIPBLAS_OP_N,                            // No transpose for this matrix
                rows_, cols_,                           // Dimensions of the matrix
                &alpha,                                 // Scaling factor for multiplication
                primal_data_ + b * rows_ * cols_,       // Pointer to the matrix for this batch
                rows_,                                  // Leading dimension of the matrix
                vector.primal_data() + b * vector.size(), // Pointer to the vector for this batch
                1,                                      // Stride for the vector
                &beta,                                  // Scaling factor for the result
                result.primal_data() + b * rows_,       // Pointer to the result vector for this batch
                1                                       // Stride for the result vector
            );

            if (status != HIPBLAS_STATUS_SUCCESS) {
                throw std::runtime_error("cuBLAS matrix-vector multiplication failed.");
            }
        }

        return result;
    }

    MatrixDense<T> transpose() const {
        MatrixDense<T> result(batch_size_, cols_, rows_);
        ComplexT alpha = ComplexT(1.0, 0.0);
        ComplexT beta = ComplexT(0.0, 0.0);

        for (int b = 0; b < batch_size_; ++b) {
            hipblasZgeam(handle_,
                        HIPBLAS_OP_T, HIPBLAS_OP_T,
                        cols_, rows_,
                        &alpha,
                        primal_data_ + b * rows_ * cols_, rows_,
                        &beta,
                        nullptr, cols_,
                        result.primal_data_ + b * cols_ * rows_, cols_);
        }

        return result;
    }

    MatrixDense<T> elementwiseAdd(const MatrixDense<T>& other) const {
        if (batch_size_ != other.batch_size_ || rows_ != other.rows_ || cols_ != other.cols_) {
            throw std::invalid_argument("Tensor dimensions do not match for addition.");
        }

        MatrixDense<T> result(batch_size_, rows_, cols_);
            int total_elements = batch_size_ * rows_ * cols_;

            thrust::transform(thrust::device_pointer_cast(primal_data_),
                            thrust::device_pointer_cast(primal_data_ + total_elements),
                            thrust::device_pointer_cast(other.primal_data_),
                            thrust::device_pointer_cast(result.primal_data_),
                            thrust::plus<ComplexT>());

            return result;
    }

    MatrixDense<T> square() const {
        // Create a new MatrixDense object to store the result
        MatrixDense<T> result(batch_size_, rows_, cols_);

        // Calculate the total number of elements in the tensor
        int total_elements = batch_size_ * rows_ * cols_;

        // Use thrust to perform element-wise squaring of the tensor
        thrust::transform(
            thrust::device_pointer_cast(primal_data_), 
            thrust::device_pointer_cast(primal_data_ + total_elements),
            thrust::device_pointer_cast(result.primal_data()),
            [] __device__(ComplexT x) { return x * x; });

        return result;
    }

    MatrixDense<T> upperTriangular() const {
        // Create a result matrix with the same dimensions
        MatrixDense<T> result(batch_size_, rows_, cols_);

        size_t total_elements = rows_ * cols_;

        for (int b = 0; b < batch_size_; ++b) {
            // Primal data pointers for this batch
            ComplexT* batch_primal_src = primal_data_ + b * total_elements;
            ComplexT* batch_primal_dst = result.primal_data_ + b * total_elements;

            // Apply the upper triangular operation
            thrust::for_each(
                thrust::counting_iterator<int>(0),
                thrust::counting_iterator<int>(rows_ * cols_),
                [=] __device__(int idx) {
                    int row = idx / cols_;
                    int col = idx % cols_;
                    batch_primal_dst[idx] = (row <= col) ? batch_primal_src[idx] : ComplexT(0.0, 0.0);
                });
        }

        return result;
    }

    MatrixDense<T> lowerTriangular() const {
        // Create a result matrix with the same dimensions
        MatrixDense<T> result(batch_size_, rows_, cols_);

        size_t total_elements = rows_ * cols_;

        for (int b = 0; b < batch_size_; ++b) {
            // Primal data pointers for this batch
            ComplexT* batch_primal_src = primal_data_ + b * total_elements;
            ComplexT* batch_primal_dst = result.primal_data_ + b * total_elements;

            // Apply the lower triangular operation
            thrust::for_each(
                thrust::counting_iterator<int>(0),
                thrust::counting_iterator<int>(rows_ * cols_),
                [=] __device__(int idx) {
                    int row = idx / cols_;
                    int col = idx % cols_;
                    batch_primal_dst[idx] = (row >= col) ? batch_primal_src[idx] : ComplexT(0.0, 0.0);
                });
        }

        return result;
    }




private:
    void initializeHandles() {
        if (hipblasCreate(&handle_) != HIPBLAS_STATUS_SUCCESS) {
            throw std::runtime_error("Failed to create cuBLAS handle.");
        }
        if (hipStreamCreate(&stream_) != hipSuccess) {
            hipblasDestroy(handle_);
            throw std::runtime_error("Failed to create CUDA stream.");
        }
        hipblasSetStream(handle_, stream_);
    }

public:
    // Disable copy constructor and copy assignment
    MatrixDense(const MatrixDense&) = delete;
    MatrixDense& operator=(const MatrixDense&) = delete;

    // Enable move constructor and move assignment
    MatrixDense(MatrixDense&&) noexcept = default;
    MatrixDense& operator=(MatrixDense&&) noexcept = default;

    // Getters for dimensions
    int batch_size() const { return batch_size_; }
    int rows() const { return rows_; }
    int cols() const { return cols_; }

    // Getters for data pointers
    ComplexT* primal_data() { return primal_data_; }
    const ComplexT* primal_data() const { return primal_data_; }
};




template <typename T>
class MatrixDualDense {
private:
    using ComplexT = std::complex<T>;

    // Dimensions
    int batch_size_;
    int rows_;
    int cols_;
    int dual_dim_;

    // Primal and Dual Data
    ComplexT* primal_data_; // Device-side primal part
    ComplexT* dual_data_;   // Device-side dual part
    bool owns_memory_;      // Indicates if memory is managed internally

    // cuBLAS handle
    hipblasHandle_t handle_;
    hipStream_t stream_;

public:
    // Constructor with external memory
    MatrixDualDense(int batch_size, int rows, int cols, int dual_dim, ComplexT* primal_data, ComplexT* dual_data)
        : batch_size_(batch_size), rows_(rows), cols_(cols), dual_dim_(dual_dim),
          primal_data_(primal_data), dual_data_(dual_data), owns_memory_(false) {
        if (!primal_data_ || !dual_data_) {
            throw std::invalid_argument("Primal or dual data pointer is null");
        }

        initializeHandles();
    }

    // Constructor with internal memory allocation
    MatrixDualDense(int batch_size, int rows, int cols, int dual_dim)
        : batch_size_(batch_size), rows_(rows), cols_(cols), dual_dim_(dual_dim), owns_memory_(true) {
        if (batch_size <= 0 || rows <= 0 || cols <= 0 || dual_dim <= 0) {
            throw std::invalid_argument("All dimensions must be positive.");
        }

        size_t primal_size = batch_size * rows * cols * sizeof(ComplexT);
        size_t dual_size = batch_size * rows * cols * dual_dim * sizeof(ComplexT);

        if (hipMalloc(&primal_data_, primal_size) != hipSuccess) {
            throw std::runtime_error("Failed to allocate GPU memory for primal data.");
        }

        if (hipMalloc(&dual_data_, dual_size) != hipSuccess) {
            hipFree(primal_data_);
            throw std::runtime_error("Failed to allocate GPU memory for dual data.");
        }

        initializeHandles();
    }

    ~MatrixDualDense() {
        if (owns_memory_) {
            if (primal_data_) hipFree(primal_data_);
            if (dual_data_) hipFree(dual_data_);
        }
        hipblasDestroy(handle_);
        hipStreamDestroy(stream_);
    }

    void initialize(const ComplexT* primal, const ComplexT* dual, size_t primal_size, size_t dual_size) {
        if (primal_size != batch_size_ * rows_ * cols_ || dual_size != batch_size_ * rows_ * cols_ * dual_dim_) {
            throw std::invalid_argument("Input sizes do not match tensor dimensions.");
        }

        if (primal) {
            hipMemcpyAsync(primal_data_, primal, primal_size * sizeof(ComplexT), hipMemcpyHostToDevice, stream_);
        }
        if (dual) {
            hipMemcpyAsync(dual_data_, dual, dual_size * sizeof(ComplexT), hipMemcpyHostToDevice, stream_);
        }
        hipStreamSynchronize(stream_);
    }

    template <typename U>
    MatrixDualDense<U> indexGet(int start_row, int end_row, int start_col, int end_col) const {
        // Validate row and column ranges
        if (start_row < 0 || end_row > rows_ || start_row >= end_row ||
            start_col < 0 || end_col > cols_ || start_col >= end_col) {
            throw std::invalid_argument("Invalid row or column range for indexGet.");
        }

        // Dimensions of the submatrix
        int sub_rows = end_row - start_row;
        int sub_cols = end_col - start_col;

        // Calculate offsets for the primal and dual data
        ComplexT* sub_primal_data = primal_data_ + start_row * cols_ + start_col;
        ComplexT* sub_dual_data = dual_data_ + start_row * cols_ * dual_dim_ + start_col * dual_dim_;

        // Create a new MatrixDualDense instance sharing the data with the original
        return MatrixDualDense<T>(batch_size_, sub_rows, sub_cols, dual_dim_, sub_primal_data, sub_dual_data);
    }

    template <typename U>
    void indexPut(int start_row, int end_row, int start_col, int end_col, const MatrixDualDense<U>& data) {
        // Validate row and column ranges
        if (start_row < 0 || end_row > rows_ || start_row >= end_row ||
            start_col < 0 || end_col > cols_ || start_col >= end_col) {
            throw std::invalid_argument("Invalid row or column range for indexPut.");
        }

        // Validate dimensions of the input data
        if (data.rows_ != (end_row - start_row) || data.cols_ != (end_col - start_col) || 
            data.dual_dim_ != dual_dim_ || data.batch_size_ != batch_size_) {
            throw std::invalid_argument("Input data dimensions do not match the target range.");
        }

        // Calculate offsets for the primal and dual data
        ComplexT* target_primal_data = primal_data_ + start_row * cols_ + start_col;
        ComplexT* target_dual_data = dual_data_ + start_row * cols_ * dual_dim_ + start_col * dual_dim_;

        // Calculate data sizes
        size_t primal_size = data.rows_ * data.cols_ * sizeof(ComplexT);
        size_t dual_size = data.rows_ * data.cols_ * dual_dim_ * sizeof(ComplexT);

        // Update primal data
        for (int b = 0; b < batch_size_; ++b) {
            hipMemcpy2DAsync(target_primal_data + b * rows_ * cols_,
                            cols_ * sizeof(ComplexT),
                            data.primal_data_ + b * data.rows_ * data.cols_,
                            data.cols_ * sizeof(ComplexT),
                            data.cols_ * sizeof(ComplexT),
                            data.rows_,
                            hipMemcpyDeviceToDevice,
                            stream_);
        }

        // Update dual data
        for (int b = 0; b < batch_size_; ++b) {
            hipMemcpy2DAsync(target_dual_data + b * rows_ * cols_ * dual_dim_,
                            cols_ * dual_dim_ * sizeof(ComplexT),
                            data.dual_data_ + b * data.rows_ * data.cols_ * dual_dim_,
                            data.cols_ * dual_dim_ * sizeof(ComplexT),
                            data.cols_ * dual_dim_ * sizeof(ComplexT),
                            data.rows_,
                            hipMemcpyDeviceToDevice,
                            stream_);
        }

        // Synchronize to ensure data transfer is complete
        hipStreamSynchronize(stream_);
    }    

    void square() {
        // Element-wise square for the primal part
        size_t total_primal_elements = batch_size_ * rows_ * cols_;
        size_t total_dual_elements = batch_size_ * rows_ * cols_ * dual_dim_;

        // Kernel to compute element-wise square
        auto squareKernel = [] __device__(ComplexT x) -> ComplexT {
            return x * x;
        };

        // Launch a CUDA kernel to square the primal part
        thrust::device_ptr<ComplexT> primal_ptr(primal_data_);
        thrust::transform(thrust::device, primal_ptr, primal_ptr + total_primal_elements, primal_ptr, squareKernel);

        // Update the dual part according to the product rule:
        // If u = f(x) and v = f'(x), then square(u) has derivative: 2 * u * v.
        auto dualKernel = [] __device__(ComplexT u, ComplexT v) -> ComplexT {
            return ComplexT(2.0, 0.0) * u * v;
        };

        // Process the dual part
        for (int d = 0; d < dual_dim_; ++d) {
            ComplexT* dual_ptr = dual_data_ + d * batch_size_ * rows_ * cols_;
            thrust::device_ptr<ComplexT> dual_thrust_ptr(dual_ptr);
            thrust::transform(thrust::device, primal_ptr, primal_ptr + total_primal_elements,
                            dual_thrust_ptr, dual_thrust_ptr, dualKernel);
        }

        hipStreamSynchronize(stream_);
    }




    template <typename U>
    MatrixDualDense<U> sum(int dimension) const {
        if (dimension != 1 && dimension != 2) {
            throw std::invalid_argument("Dimension must be 1 (rows) or 2 (columns).");
        }

        // Determine the new dimensions after summing along the specified axis
        int new_rows = (dimension == 1) ? 1 : rows_;
        int new_cols = (dimension == 2) ? 1 : cols_;

        // Create the resulting MatrixDualDense object
        MatrixDualDense<T> result(batch_size_, new_rows, new_cols, dual_dim_);

        // Allocate memory for temporary host buffers
        size_t primal_size = rows_ * cols_;
        size_t dual_size = rows_ * cols_ * dual_dim_;

        // Perform the summation along the specified dimension
        for (int b = 0; b < batch_size_; ++b) {
            if (dimension == 1) {
                // Summing along rows
                for (int c = 0; c < cols_; ++c) {
                    ComplexT sum_primal = ComplexT(0.0, 0.0);
                    std::vector<ComplexT> sum_dual(dual_dim_, ComplexT(0.0, 0.0));

                    for (int r = 0; r < rows_; ++r) {
                        int idx = b * rows_ * cols_ + r * cols_ + c;
                        sum_primal += primal_data_[idx];

                        for (int d = 0; d < dual_dim_; ++d) {
                            int dual_idx = b * rows_ * cols_ * dual_dim_ + r * cols_ * dual_dim_ + c * dual_dim_ + d;
                            sum_dual[d] += dual_data_[dual_idx];
                        }
                    }

                    // Store the result in the output matrix
                    int result_idx = b * new_rows * new_cols + c;
                    result.primal_data_[result_idx] = sum_primal;

                    for (int d = 0; d < dual_dim_; ++d) {
                        int result_dual_idx = b * new_rows * new_cols * dual_dim_ + c * dual_dim_ + d;
                        result.dual_data_[result_dual_idx] = sum_dual[d];
                    }
                }
            } else if (dimension == 2) {
                // Summing along columns
                for (int r = 0; r < rows_; ++r) {
                    ComplexT sum_primal = ComplexT(0.0, 0.0);
                    std::vector<ComplexT> sum_dual(dual_dim_, ComplexT(0.0, 0.0));

                    for (int c = 0; c < cols_; ++c) {
                        int idx = b * rows_ * cols_ + r * cols_ + c;
                        sum_primal += primal_data_[idx];

                        for (int d = 0; d < dual_dim_; ++d) {
                            int dual_idx = b * rows_ * cols_ * dual_dim_ + r * cols_ * dual_dim_ + c * dual_dim_ + d;
                            sum_dual[d] += dual_data_[dual_idx];
                        }
                    }

                    // Store the result in the output matrix
                    int result_idx = b * new_rows * new_cols + r;
                    result.primal_data_[result_idx] = sum_primal;

                    for (int d = 0; d < dual_dim_; ++d) {
                        int result_dual_idx = b * new_rows * new_cols * dual_dim_ + r * dual_dim_ + d;
                        result.dual_data_[result_dual_idx] = sum_dual[d];
                    }
                }
            }
        }

        hipStreamSynchronize(stream_);
        return result;
    }

    VectorDualDense<T> squeeze(int dim) const {
        // Check the validity of the dimension
        if (dim < 1 || dim > 2) {
            throw std::invalid_argument("Dimension to squeeze must be 1 (rows) or 2 (columns).");
        }

        // Ensure the specified dimension has size 1
        if ((dim == 1 && rows_ != 1) || (dim == 2 && cols_ != 1)) {
            throw std::invalid_argument("Cannot squeeze a dimension with size greater than 1.");
        }

        // Determine the size of the resulting vector
        int vector_size = (dim == 1) ? cols_ : rows_;

        // Calculate the pointer to primal and dual data
        ComplexT* squeezed_primal_data = primal_data_;
        ComplexT* squeezed_dual_data = dual_data_;

        // Create and return a VectorDual object
        return VectorDualDense<T>(vector_size, dual_dim_, squeezed_primal_data, squeezed_dual_data);
    }

    template <typename U>
    VectorDualDense<U> matrixVectorProduct(const VectorDualDense<U>& vector) const {
        // Validate dimensions
        if (cols_ != vector.size_ || batch_size_ != vector.batch_size_ || dual_dim_ != vector.dual_dim_) {
            throw std::invalid_argument("Matrix and vector dimensions do not match for multiplication.");
        }

        // Result vector
        VectorDualDense<T> result(batch_size_, rows_, dual_dim_);

        int matrix_primal_size = rows_ * cols_;
        int vector_primal_size = vector.size_;
        int result_primal_size = rows_;

        int matrix_dual_size = matrix_primal_size * dual_dim_;
        int vector_dual_size = vector_primal_size * dual_dim_;
        int result_dual_size = result_primal_size * dual_dim_;

        ComplexT alpha = ComplexT(1.0, 0.0);
        ComplexT beta = ComplexT(0.0, 0.0);

        // Perform matrix-vector multiplication for primal part
        for (int b = 0; b < batch_size_; ++b) {
            hipblasZgemv(handle_,
                        HIPBLAS_OP_N,
                        rows_, cols_,
                        &alpha,
                        primal_data_ + b * matrix_primal_size, rows_,
                        vector.primal_data_ + b * vector_primal_size, 1,
                        &beta,
                        result.primal_data_ + b * result_primal_size, 1);
        }

        // Perform matrix-vector multiplication for dual part
        for (int b = 0; b < batch_size_; ++b) {
            for (int d = 0; d < dual_dim_; ++d) {
                // Matrix * Dual(Vector)
                hipblasZgemv(handle_,
                            HIPBLAS_OP_N,
                            rows_, cols_,
                            &alpha,
                            primal_data_ + b * matrix_primal_size, rows_,
                            vector.dual_data_ + b * vector_primal_size * dual_dim_ + d * vector_primal_size, 1,
                            &beta,
                            result.dual_data_ + b * result_primal_size * dual_dim_ + d * result_primal_size, 1);

                // Dual(Matrix) * Vector
                hipblasZgemv(handle_,
                            HIPBLAS_OP_N,
                            rows_, cols_,
                            &alpha,
                            dual_data_ + b * matrix_dual_size + d * matrix_primal_size, rows_,
                            vector.primal_data_ + b * vector_primal_size, 1,
                            &alpha, // Accumulate
                            result.dual_data_ + b * result_primal_size * dual_dim_ + d * result_primal_size, 1);
            }
        }

        return result;
    }


    MatrixDualDense<T> transpose() const {
        MatrixDualDense<T> result(batch_size_, cols_, rows_, dual_dim_);
        ComplexT alpha = ComplexT(1.0, 0.0);
        ComplexT beta = ComplexT(0.0, 0.0);

        for (int b = 0; b < batch_size_; ++b) {
            hipblasZgeam(handle_,
                        HIPBLAS_OP_T, HIPBLAS_OP_T,
                        cols_, rows_,
                        &alpha,
                        primal_data_ + b * rows_ * cols_, rows_,
                        &beta,
                        nullptr, cols_,
                        result.primal_data_ + b * cols_ * rows_, cols_);
        }

        for (int b = 0; b < batch_size_; ++b) {
            for (int d = 0; d < dual_dim_; ++d) {
                hipblasZgeam(handle_,
                            HIPBLAS_OP_T, HIPBLAS_OP_T,
                            cols_, rows_,
                            &alpha,
                            dual_data_ + b * rows_ * cols_ * dual_dim_ + d * rows_ * cols_, rows_,
                            &beta,
                            nullptr, cols_,
                            result.dual_data_ + b * cols_ * rows_ * dual_dim_ + d * cols_ * rows_, cols_);
            }
        }

        return result;
    }

    // Method to generate an upper triangular matrix
    MatrixDualDense<T> upperTriangular() const {
        // Create a new matrix for the result
        MatrixDualDense<T> result(batch_size_, rows_, cols_, dual_dim_);

        size_t total_elements = rows_ * cols_;

        for (int b = 0; b < batch_size_; ++b) {
            // Initialize primal part to upper triangular
            ComplexT* batch_primal_src = primal_data_ + b * total_elements;
            ComplexT* batch_primal_dst = result.primal_data_ + b * total_elements;

            thrust::for_each(
                thrust::counting_iterator<int>(0),
                thrust::counting_iterator<int>(rows_ * cols_),
                [=] __device__(int idx) {
                    int row = idx / cols_;
                    int col = idx % cols_;
                    batch_primal_dst[idx] = (row <= col) ? batch_primal_src[idx] : ComplexT(0.0, 0.0);
                });

            // Initialize dual part to zero
            ComplexT* batch_dual_dst = result.dual_data_ + b * total_elements * dual_dim_;
            thrust::fill(thrust::device_pointer_cast(batch_dual_dst),
                         thrust::device_pointer_cast(batch_dual_dst + total_elements * dual_dim_),
                         ComplexT(0.0, 0.0));
        }

        return result;
    }


    // Method to generate a lower triangular matrix
    MatrixDualDense<T> lowerTriangular() const {
            // Create a new matrix for the result
            MatrixDualDense<T> result(batch_size_, rows_, cols_, dual_dim_);

            size_t total_elements = rows_ * cols_;

            for (int b = 0; b < batch_size_; ++b) {
                // Initialize primal part to lower triangular
                ComplexT* batch_primal_src = primal_data_ + b * total_elements;
                ComplexT* batch_primal_dst = result.primal_data_ + b * total_elements;

                thrust::for_each(
                    thrust::counting_iterator<int>(0),
                    thrust::counting_iterator<int>(rows_ * cols_),
                    [=] __device__(int idx) {
                        int row = idx / cols_;
                        int col = idx % cols_;
                        batch_primal_dst[idx] = (row >= col) ? batch_primal_src[idx] : ComplexT(0.0, 0.0);
                    });

                // Initialize dual part to zero
                ComplexT* batch_dual_dst = result.dual_data_ + b * total_elements * dual_dim_;
                thrust::fill(thrust::device_pointer_cast(batch_dual_dst),
                            thrust::device_pointer_cast(batch_dual_dst + total_elements * dual_dim_),
                            ComplexT(0.0, 0.0));
            }

            return result;
        }

    template <typename U>
    MatrixDualDense<U> matrixMultiply(const MatrixDualDense<U>& other) const {
        // Validate dimensions
        if (cols_ != other.rows_ || batch_size_ != other.batch_size_ || dual_dim_ != other.dual_dim_) {
            throw std::invalid_argument("Matrix dimensions do not match for multiplication.");
        }

        // Create the result matrix
        MatrixDualDense<T> result(batch_size_, rows_, other.cols_, dual_dim_);

        ComplexT alpha = ComplexT(1.0, 0.0);
        ComplexT beta = ComplexT(0.0, 0.0);

        // Perform batched matrix multiplication for primal and dual parts
        for (int b = 0; b < batch_size_; ++b) {
            // Primal part
            hipblasZgemm(handle_,
                        HIPBLAS_OP_N, HIPBLAS_OP_N,
                        rows_, other.cols_, cols_,
                        &alpha,
                        primal_data_ + b * rows_ * cols_, rows_,
                        other.primal_data_ + b * other.rows_ * other.cols_, other.rows_,
                        &beta,
                        result.primal_data_ + b * rows_ * other.cols_, rows_);

            // Dual part
            for (int d = 0; d < dual_dim_; ++d) {
                // Primal * Dual(other)
                hipblasZgemm(handle_,
                            HIPBLAS_OP_N, HIPBLAS_OP_N,
                            rows_, other.cols_, cols_,
                            &alpha,
                            primal_data_ + b * rows_ * cols_, rows_,
                            other.dual_data_ + b * other.rows_ * other.cols_ * dual_dim_ + d * other.rows_ * other.cols_, other.rows_,
                            &beta,
                            result.dual_data_ + b * rows_ * other.cols_ * dual_dim_ + d * rows_ * other.cols_, rows_);

                // Dual(Matrix) * Primal(other)
                hipblasZgemm(handle_,
                            HIPBLAS_OP_N, HIPBLAS_OP_N,
                            rows_, other.cols_, cols_,
                            &alpha,
                            dual_data_ + b * rows_ * cols_ * dual_dim_ + d * rows_ * cols_, rows_,
                            other.primal_data_ + b * other.rows_ * other.cols_, other.rows_,
                            &alpha, // Accumulate
                            result.dual_data_ + b * rows_ * other.cols_ * dual_dim_ + d * rows_ * other.cols_, rows_);
            }
        }

        hipStreamSynchronize(stream_);
        return result;
    }


    template <typename U>
    MatrixDualDense<U> elementwiseMultiply(const MatrixDualDense<U>& other) const {
        // Validate dimensions
        if (batch_size_ != other.batch_size_ || rows_ != other.rows_ || cols_ != other.cols_ || dual_dim_ != other.dual_dim_) {
            throw std::invalid_argument("Tensor dimensions do not match for elementwise multiplication.");
        }

        // Create result tensor
        MatrixDualDense<T> result(batch_size_, rows_, cols_, dual_dim_);

        int total_primal_elements = batch_size_ * rows_ * cols_;
        int total_dual_elements = total_primal_elements * dual_dim_;

        // Elementwise multiplication for primal part
        thrust::transform(
            thrust::device_pointer_cast(primal_data_),
            thrust::device_pointer_cast(primal_data_ + total_primal_elements),
            thrust::device_pointer_cast(other.primal_data_),
            thrust::device_pointer_cast(result.primal_data_),
            thrust::multiplies<ComplexT>());

        // Elementwise multiplication for dual part using product rule
        thrust::for_each(
            thrust::counting_iterator<int>(0),
            thrust::counting_iterator<int>(total_dual_elements),
            [=] __device__(int idx) {
                int primal_idx = idx / dual_dim_;
                int dual_idx = idx % dual_dim_;

                result.dual_data_[idx] =
                    dual_data_[idx] * other.primal_data_[primal_idx] +
                    primal_data_[primal_idx] * other.dual_data_[idx];
            });

        return result;
    };

    

private:
    void initializeHandles() {
        if (hipblasCreate(&handle_) != HIPBLAS_STATUS_SUCCESS) {
            throw std::runtime_error("Failed to create cuBLAS handle.");
        }
        if (hipStreamCreate(&stream_) != hipSuccess) {
            hipblasDestroy(handle_);
            throw std::runtime_error("Failed to create CUDA stream.");
        }
        hipblasSetStream(handle_, stream_);
    }
};


template <typename T>
class MatrixHyperDualDense {
private:
    using ComplexT = std::complex<T>;

    // Dimensions
    int batch_size_;
    int rows_;
    int cols_;
    int dual_dim_;

    // Primal, Dual, and Hyper-Dual Data
    ComplexT* primal_data_;     // [M, N, L]
    ComplexT* dual_data_;       // [M, N, L, D]
    ComplexT* hyper_dual_data_; // [M, N, L, D, D]
    bool owns_memory_;          // Indicates if memory is managed internally

    // cuBLAS handle
    hipblasHandle_t handle_;
    hipStream_t stream_;

public:
    // Constructor with external memory
    MatrixHyperDualDense(int batch_size, int rows, int cols, int dual_dim,
                           ComplexT* primal_data, ComplexT* dual_data, ComplexT* hyper_dual_data)
        : batch_size_(batch_size), rows_(rows), cols_(cols), dual_dim_(dual_dim),
          primal_data_(primal_data), dual_data_(dual_data), hyper_dual_data_(hyper_dual_data),
          owns_memory_(false) {
        if (!primal_data_ || !dual_data_ || !hyper_dual_data_) {
            throw std::invalid_argument("Primal, dual, or hyper-dual data pointer is null");
        }

        initializeHandles();
    }

    // Constructor with internal memory allocation
    MatrixHyperDualDense(int batch_size, int rows, int cols, int dual_dim)
        : batch_size_(batch_size), rows_(rows), cols_(cols), dual_dim_(dual_dim), owns_memory_(true) {
        if (batch_size <= 0 || rows <= 0 || cols <= 0 || dual_dim <= 0) {
            throw std::invalid_argument("All dimensions must be positive.");
        }

        size_t primal_size = batch_size * rows * cols * sizeof(ComplexT);
        size_t dual_size = batch_size * rows * cols * dual_dim * sizeof(ComplexT);
        size_t hyper_dual_size = batch_size * rows * cols * dual_dim * dual_dim * sizeof(ComplexT);

        if (hipMalloc(&primal_data_, primal_size) != hipSuccess) {
            throw std::runtime_error("Failed to allocate GPU memory for primal data.");
        }

        if (hipMalloc(&dual_data_, dual_size) != hipSuccess) {
            hipFree(primal_data_);
            throw std::runtime_error("Failed to allocate GPU memory for dual data.");
        }

        if (hipMalloc(&hyper_dual_data_, hyper_dual_size) != hipSuccess) {
            hipFree(primal_data_);
            hipFree(dual_data_);
            throw std::runtime_error("Failed to allocate GPU memory for hyper-dual data.");
        }

        initializeHandles();
    }

    ~MatrixHyperDualDense() {
        if (owns_memory_) {
            if (primal_data_) hipFree(primal_data_);
            if (dual_data_) hipFree(dual_data_);
            if (hyper_dual_data_) hipFree(hyper_dual_data_);
        }
        hipblasDestroy(handle_);
        hipStreamDestroy(stream_);
    }

    void initialize(const ComplexT* primal, const ComplexT* dual, const ComplexT* hyper_dual,
                    size_t primal_size, size_t dual_size, size_t hyper_dual_size) {
        if (primal_size != batch_size_ * rows_ * cols_ ||
            dual_size != batch_size_ * rows_ * cols_ * dual_dim_ ||
            hyper_dual_size != batch_size_ * rows_ * cols_ * dual_dim_ * dual_dim_) {
            throw std::invalid_argument("Input sizes do not match tensor dimensions.");
        }

        if (primal) {
            hipMemcpyAsync(primal_data_, primal, primal_size * sizeof(ComplexT), hipMemcpyHostToDevice, stream_);
        }
        if (dual) {
            hipMemcpyAsync(dual_data_, dual, dual_size * sizeof(ComplexT), hipMemcpyHostToDevice, stream_);
        }
        if (hyper_dual) {
            hipMemcpyAsync(hyper_dual_data_, hyper_dual, hyper_dual_size * sizeof(ComplexT), hipMemcpyHostToDevice, stream_);
        }
        hipStreamSynchronize(stream_);
    }

    template <typename U>
    MatrixHyperDualDense<U> indexGet(int start_row, int end_row, int start_col, int end_col) const {
        // Validate row and column ranges
        if (start_row < 0 || end_row > rows_ || start_row >= end_row ||
            start_col < 0 || end_col > cols_ || start_col >= end_col) {
            throw std::invalid_argument("Invalid row or column range for indexGet.");
        }

        // Dimensions of the submatrix
        int sub_rows = end_row - start_row;
        int sub_cols = end_col - start_col;

        // Calculate offsets for the primal, dual, and hyper-dual data
        ComplexT* sub_primal_data = primal_data_ + start_row * cols_ + start_col;
        ComplexT* sub_dual_data = dual_data_ + start_row * cols_ * dual_dim_ + start_col * dual_dim_;
        ComplexT* sub_hyper_dual_data = hyper_dual_data_ +
                                        start_row * cols_ * dual_dim_ * dual_dim_ +
                                        start_col * dual_dim_ * dual_dim_;

        // Create a new MatrixHyperDualDense instance sharing the data with the original
        return MatrixHyperDualDense<T>(batch_size_, sub_rows, sub_cols, dual_dim_,
                                    sub_primal_data, sub_dual_data, sub_hyper_dual_data);
    }


    template <typename U>
    void indexPut(int start_row, int end_row, int start_col, int end_col, const MatrixHyperDualDense<U>& data) {
        // Validate row and column ranges
        if (start_row < 0 || end_row > rows_ || start_row >= end_row ||
            start_col < 0 || end_col > cols_ || start_col >= end_col) {
            throw std::invalid_argument("Invalid row or column range for indexPut.");
        }

        // Validate dimensions of the input data
        if (data.rows_ != (end_row - start_row) || data.cols_ != (end_col - start_col) ||
            data.dual_dim_ != dual_dim_ || data.batch_size_ != batch_size_) {
            throw std::invalid_argument("Input data dimensions do not match the target range.");
        }

        // Calculate offsets for the primal, dual, and hyper-dual data
        ComplexT* target_primal_data = primal_data_ + start_row * cols_ + start_col;
        ComplexT* target_dual_data = dual_data_ + start_row * cols_ * dual_dim_ + start_col * dual_dim_;
        ComplexT* target_hyper_dual_data = hyper_dual_data_ +
                                        start_row * cols_ * dual_dim_ * dual_dim_ +
                                        start_col * dual_dim_ * dual_dim_;

        // Update primal data
        for (int b = 0; b < batch_size_; ++b) {
            hipMemcpy2DAsync(target_primal_data + b * rows_ * cols_,
                            cols_ * sizeof(ComplexT),
                            data.primal_data_ + b * data.rows_ * data.cols_,
                            data.cols_ * sizeof(ComplexT),
                            data.cols_ * sizeof(ComplexT),
                            data.rows_,
                            hipMemcpyDeviceToDevice,
                            stream_);
        }

        // Update dual data
        for (int b = 0; b < batch_size_; ++b) {
            hipMemcpy2DAsync(target_dual_data + b * rows_ * cols_ * dual_dim_,
                            cols_ * dual_dim_ * sizeof(ComplexT),
                            data.dual_data_ + b * data.rows_ * data.cols_ * dual_dim_,
                            data.cols_ * dual_dim_ * sizeof(ComplexT),
                            data.cols_ * dual_dim_ * sizeof(ComplexT),
                            data.rows_,
                            hipMemcpyDeviceToDevice,
                            stream_);
        }

        // Update hyper-dual data
        for (int b = 0; b < batch_size_; ++b) {
            hipMemcpy2DAsync(target_hyper_dual_data + b * rows_ * cols_ * dual_dim_ * dual_dim_,
                            cols_ * dual_dim_ * dual_dim_ * sizeof(ComplexT),
                            data.hyper_dual_data_ + b * data.rows_ * data.cols_ * dual_dim_ * dual_dim_,
                            data.cols_ * dual_dim_ * dual_dim_ * sizeof(ComplexT),
                            data.cols_ * dual_dim_ * dual_dim_ * sizeof(ComplexT),
                            data.rows_,
                            hipMemcpyDeviceToDevice,
                            stream_);
        }

        // Synchronize to ensure data transfer is complete
        hipStreamSynchronize(stream_);
    }

    MatrixHyperDualDense<T> transpose() const {
        MatrixHyperDualDense<T> result(batch_size_, cols_, rows_, dual_dim_);
        ComplexT alpha = ComplexT(1.0, 0.0);
        ComplexT beta = ComplexT(0.0, 0.0);

        // Transpose primal part
        for (int b = 0; b < batch_size_; ++b) {
            hipblasZgeam(handle_,
                        HIPBLAS_OP_T, HIPBLAS_OP_T,
                        cols_, rows_,
                        &alpha,
                        primal_data_ + b * rows_ * cols_, rows_,
                        &beta,
                        nullptr, cols_,
                        result.primal_data_ + b * cols_ * rows_, cols_);
        }

        // Transpose dual part
        for (int b = 0; b < batch_size_; ++b) {
            for (int d = 0; d < dual_dim_; ++d) {
                hipblasZgeam(handle_,
                            HIPBLAS_OP_T, HIPBLAS_OP_T,
                            cols_, rows_,
                            &alpha,
                            dual_data_ + b * rows_ * cols_ * dual_dim_ + d * rows_ * cols_, rows_,
                            &beta,
                            nullptr, cols_,
                            result.dual_data_ + b * cols_ * rows_ * dual_dim_ + d * cols_ * rows_, cols_);
            }
        }

        // Transpose hyper-dual part
        for (int b = 0; b < batch_size_; ++b) {
            for (int d1 = 0; d1 < dual_dim_; ++d1) {
                for (int d2 = 0; d2 < dual_dim_; ++d2) {
                    hipblasZgeam(handle_,
                                HIPBLAS_OP_T, HIPBLAS_OP_T,
                                cols_, rows_,
                                &alpha,
                                hyper_dual_data_ + b * rows_ * cols_ * dual_dim_ * dual_dim_ +
                                    d1 * rows_ * cols_ * dual_dim_ + d2 * rows_ * cols_,
                                rows_,
                                &beta,
                                nullptr, cols_,
                                result.hyper_dual_data_ + b * cols_ * rows_ * dual_dim_ * dual_dim_ +
                                    d1 * cols_ * rows_ * dual_dim_ + d2 * cols_ * rows_,
                                cols_);
                }
            }
        }

        return result;
    }

    MatrixHyperDualDense<T> upperTriangular() const {
        // Create a result matrix with the same dimensions
        MatrixHyperDualDense<T> result(batch_size_, rows_, cols_, dual_dim_);

        size_t total_elements = rows_ * cols_;

        for (int b = 0; b < batch_size_; ++b) {
            // Primal data pointers for this batch
            ComplexT* batch_primal_src = primal_data_ + b * total_elements;
            ComplexT* batch_primal_dst = result.primal_data_ + b * total_elements;

            // Apply the upper triangular operation to the primal part
            thrust::for_each(
                thrust::counting_iterator<int>(0),
                thrust::counting_iterator<int>(rows_ * cols_),
                [=] __device__(int idx) {
                    int row = idx / cols_;
                    int col = idx % cols_;
                    batch_primal_dst[idx] = (row <= col) ? batch_primal_src[idx] : ComplexT(0.0, 0.0);
                });

            // Dual part is zero-initialized
            ComplexT* batch_dual_dst = result.dual_data_ + b * total_elements * dual_dim_;
            thrust::fill(thrust::device_pointer_cast(batch_dual_dst),
                        thrust::device_pointer_cast(batch_dual_dst + total_elements * dual_dim_),
                        ComplexT(0.0, 0.0));

            // Hyper-dual part is zero-initialized
            ComplexT* batch_hyper_dual_dst = result.hyper_dual_data_ + b * total_elements * dual_dim_ * dual_dim_;
            thrust::fill(thrust::device_pointer_cast(batch_hyper_dual_dst),
                        thrust::device_pointer_cast(batch_hyper_dual_dst + total_elements * dual_dim_ * dual_dim_),
                        ComplexT(0.0, 0.0));
        }

        return result;
    }

    MatrixHyperDualDense<T> lowerTriangular() const {
        // Create a result matrix with the same dimensions
        MatrixHyperDualDense<T> result(batch_size_, rows_, cols_, dual_dim_);

        size_t total_elements = rows_ * cols_;

        for (int b = 0; b < batch_size_; ++b) {
            // Primal data pointers for this batch
            ComplexT* batch_primal_src = primal_data_ + b * total_elements;
            ComplexT* batch_primal_dst = result.primal_data_ + b * total_elements;

            // Apply the lower triangular operation to the primal part
            thrust::for_each(
                thrust::counting_iterator<int>(0),
                thrust::counting_iterator<int>(rows_ * cols_),
                [=] __device__(int idx) {
                    int row = idx / cols_;
                    int col = idx % cols_;
                    batch_primal_dst[idx] = (row >= col) ? batch_primal_src[idx] : ComplexT(0.0, 0.0);
                });

            // Dual part is zero-initialized
            ComplexT* batch_dual_dst = result.dual_data_ + b * total_elements * dual_dim_;
            thrust::fill(thrust::device_pointer_cast(batch_dual_dst),
                        thrust::device_pointer_cast(batch_dual_dst + total_elements * dual_dim_),
                        ComplexT(0.0, 0.0));

            // Hyper-dual part is zero-initialized
            ComplexT* batch_hyper_dual_dst = result.hyper_dual_data_ + b * total_elements * dual_dim_ * dual_dim_;
            thrust::fill(thrust::device_pointer_cast(batch_hyper_dual_dst),
                        thrust::device_pointer_cast(batch_hyper_dual_dst + total_elements * dual_dim_ * dual_dim_),
                        ComplexT(0.0, 0.0));
        }

        return result;
    }
    template <typename U>
    MatrixHyperDualDense<U> matrixMultiply(const MatrixHyperDualDense<U>& other) const {
        // Validate dimensions
        if (cols_ != other.rows_ || batch_size_ != other.batch_size_ || dual_dim_ != other.dual_dim_) {
            throw std::invalid_argument("Matrix dimensions do not match for multiplication.");
        }

        // Create the result matrix
        MatrixHyperDualDense<T> result(batch_size_, rows_, other.cols_, dual_dim_);

        ComplexT alpha = ComplexT(1.0, 0.0);
        ComplexT beta = ComplexT(0.0, 0.0);

        for (int b = 0; b < batch_size_; ++b) {
            // Real part: Primal * Primal
            hipblasZgemm(handle_,
                        HIPBLAS_OP_N, HIPBLAS_OP_N,
                        rows_, other.cols_, cols_,
                        &alpha,
                        primal_data_ + b * rows_ * cols_, rows_,
                        other.primal_data_ + b * other.rows_ * other.cols_, other.rows_,
                        &beta,
                        result.primal_data_ + b * rows_ * other.cols_, rows_);

            // Dual part
            for (int d = 0; d < dual_dim_; ++d) {
                // Primal * Dual(other) + Dual * Primal(other)
                hipblasZgemm(handle_,
                            HIPBLAS_OP_N, HIPBLAS_OP_N,
                            rows_, other.cols_, cols_,
                            &alpha,
                            primal_data_ + b * rows_ * cols_, rows_,
                            other.dual_data_ + b * other.rows_ * other.cols_ * dual_dim_ + d * other.rows_ * other.cols_, other.rows_,
                            &beta,
                            result.dual_data_ + b * rows_ * other.cols_ * dual_dim_ + d * rows_ * other.cols_, rows_);

                hipblasZgemm(handle_,
                            HIPBLAS_OP_N, HIPBLAS_OP_N,
                            rows_, other.cols_, cols_,
                            &alpha,
                            dual_data_ + b * rows_ * cols_ * dual_dim_ + d * rows_ * cols_, rows_,
                            other.primal_data_ + b * other.rows_ * other.cols_, other.rows_,
                            &alpha, // Accumulate
                            result.dual_data_ + b * rows_ * other.cols_ * dual_dim_ + d * rows_ * other.cols_, rows_);
            }

            // Hyper-dual part
            for (int d1 = 0; d1 < dual_dim_; ++d1) {
                for (int d2 = 0; d2 < dual_dim_; ++d2) {
                    // (Primal * Hyper-Dual(other)) + (Dual * Dual(other)) + (Hyper-Dual * Primal(other))
                    hipblasZgemm(handle_,
                                HIPBLAS_OP_N, HIPBLAS_OP_N,
                                rows_, other.cols_, cols_,
                                &alpha,
                                primal_data_ + b * rows_ * cols_, rows_,
                                other.hyper_dual_data_ + b * other.rows_ * other.cols_ * dual_dim_ * dual_dim_ +
                                    d1 * other.rows_ * other.cols_ * dual_dim_ + d2 * other.rows_ * other.cols_, other.rows_,
                                &beta,
                                result.hyper_dual_data_ + b * rows_ * other.cols_ * dual_dim_ * dual_dim_ +
                                    d1 * rows_ * other.cols_ * dual_dim_ + d2 * rows_ * other.cols_, rows_);

                    hipblasZgemm(handle_,
                                HIPBLAS_OP_N, HIPBLAS_OP_N,
                                rows_, other.cols_, cols_,
                                &alpha,
                                dual_data_ + b * rows_ * cols_ * dual_dim_ + d1 * rows_ * cols_, rows_,
                                other.dual_data_ + b * other.rows_ * other.cols_ * dual_dim_ + d2 * other.rows_ * other.cols_, other.rows_,
                                &alpha, // Accumulate
                                result.hyper_dual_data_ + b * rows_ * other.cols_ * dual_dim_ * dual_dim_ +
                                    d1 * rows_ * other.cols_ * dual_dim_ + d2 * rows_ * other.cols_, rows_);

                    hipblasZgemm(handle_,
                                HIPBLAS_OP_N, HIPBLAS_OP_N,
                                rows_, other.cols_, cols_,
                                &alpha,
                                hyper_dual_data_ + b * rows_ * cols_ * dual_dim_ * dual_dim_ +
                                    d1 * rows_ * cols_ * dual_dim_ + d2 * rows_ * cols_, rows_,
                                other.primal_data_ + b * other.rows_ * other.cols_, other.rows_,
                                &alpha, // Accumulate
                                result.hyper_dual_data_ + b * rows_ * other.cols_ * dual_dim_ * dual_dim_ +
                                    d1 * rows_ * other.cols_ * dual_dim_ + d2 * rows_ * other.cols_, rows_);
                }
            }
        }

        hipStreamSynchronize(stream_);
        return result;
    }

    template <typename U>
    VectorHyperDualDense<U> matrixVectorProduct(const VectorHyperDualDense<U>& vector) const {
        // Validate dimensions
        if (cols_ != vector.size() || batch_size_ != vector.batch_size() || dual_dim_ != vector.dual_dim()) {
            throw std::invalid_argument("Matrix and vector dimensions do not match for multiplication.");
        }

        // Create the result vector
        VectorHyperDualDense<T> result(batch_size_, rows_, dual_dim_);

        ComplexT alpha = ComplexT(1.0, 0.0);
        ComplexT beta = ComplexT(0.0, 0.0);

        for (int b = 0; b < batch_size_; ++b) {
            // Real part: Primal * Primal(vector)
            hipblasZgemv(handle_,
                        HIPBLAS_OP_N,
                        rows_, cols_,
                        &alpha,
                        primal_data_ + b * rows_ * cols_, rows_,
                        vector.primal_data() + b * vector.size(), 1,
                        &beta,
                        result.primal_data() + b * rows_, 1);

            // Dual part
            for (int d = 0; d < dual_dim_; ++d) {
                // Primal * Dual(vector) + Dual * Primal(vector)
                hipblasZgemv(handle_,
                            HIPBLAS_OP_N,
                            rows_, cols_,
                            &alpha,
                            primal_data_ + b * rows_ * cols_, rows_,
                            vector.dual_data() + b * vector.size() * dual_dim_ + d * vector.size(), 1,
                            &beta,
                            result.dual_data() + b * rows_ * dual_dim_ + d * rows_, 1);

                hipblasZgemv(handle_,
                            HIPBLAS_OP_N,
                            rows_, cols_,
                            &alpha,
                            dual_data_ + b * rows_ * cols_ * dual_dim_ + d * rows_ * cols_, rows_,
                            vector.primal_data() + b * vector.size(), 1,
                            &alpha, // Accumulate
                            result.dual_data() + b * rows_ * dual_dim_ + d * rows_, 1);
            }

            // Hyper-dual part
            for (int d1 = 0; d1 < dual_dim_; ++d1) {
                for (int d2 = 0; d2 < dual_dim_; ++d2) {
                    // (Primal * Hyper-Dual(vector)) + (Dual * Dual(vector)) + (Hyper-Dual * Primal(vector))
                    hipblasZgemv(handle_,
                                HIPBLAS_OP_N,
                                rows_, cols_,
                                &alpha,
                                primal_data_ + b * rows_ * cols_, rows_,
                                vector.hyper_dual_data() + b * vector.size() * dual_dim_ * dual_dim_ +
                                    d1 * vector.size() * dual_dim_ + d2 * vector.size(), 1,
                                &beta,
                                result.hyper_dual_data() + b * rows_ * dual_dim_ * dual_dim_ +
                                    d1 * rows_ * dual_dim_ + d2 * rows_, 1);

                    hipblasZgemv(handle_,
                                HIPBLAS_OP_N,
                                rows_, cols_,
                                &alpha,
                                dual_data_ + b * rows_ * cols_ * dual_dim_ + d1 * rows_ * cols_, rows_,
                                vector.dual_data() + b * vector.size() * dual_dim_ + d2 * vector.size(), 1,
                                &alpha, // Accumulate
                                result.hyper_dual_data() + b * rows_ * dual_dim_ * dual_dim_ +
                                    d1 * rows_ * dual_dim_ + d2 * rows_, 1);

                    hipblasZgemv(handle_,
                                HIPBLAS_OP_N,
                                rows_, cols_,
                                &alpha,
                                hyper_dual_data_ + b * rows_ * cols_ * dual_dim_ * dual_dim_ +
                                    d1 * rows_ * cols_ * dual_dim_ + d2 * rows_ * cols_, rows_,
                                vector.primal_data() + b * vector.size(), 1,
                                &alpha, // Accumulate
                                result.hyper_dual_data() + b * rows_ * dual_dim_ * dual_dim_ +
                                    d1 * rows_ * dual_dim_ + d2 * rows_, 1);
                }
            }
        }

        hipStreamSynchronize(stream_);
        return result;
    }


private:
    void initializeHandles() {
        if (hipblasCreate(&handle_) != HIPBLAS_STATUS_SUCCESS) {
            throw std::runtime_error("Failed to create cuBLAS handle.");
        }
        if (hipStreamCreate(&stream_) != hipSuccess) {
            hipblasDestroy(handle_);
            throw std::runtime_error("Failed to create CUDA stream.");
        }
        hipblasSetStream(handle_, stream_);
    }

public:
    // Disable copy constructor and copy assignment
    MatrixHyperDualDense(const MatrixHyperDualDense&) = delete;
    MatrixHyperDualDense& operator=(const MatrixHyperDualDense&) = delete;

    // Enable move constructor and move assignment
    MatrixHyperDualDense(MatrixHyperDualDense&&) noexcept = default;
    MatrixHyperDualDense& operator=(MatrixHyperDualDense&&) noexcept = default;

    // Getters for dimensions
    int batch_size() const { return batch_size_; }
    int rows() const { return rows_; }
    int cols() const { return cols_; }
    int dual_dim() const { return dual_dim_; }

    // Getters for data pointers
    ComplexT* primal_data() { return primal_data_; }
    ComplexT* dual_data() { return dual_data_; }
    ComplexT* hyper_dual_data() { return hyper_dual_data_; }

    const ComplexT* primal_data() const { return primal_data_; }
    const ComplexT* dual_data() const { return dual_data_; }
    const ComplexT* hyper_dual_data() const { return hyper_dual_data_; }
}; // class MatrixHyperDualDense






}  // namespace janus
#endif // _CU_DUAL_TENSOR_HPP