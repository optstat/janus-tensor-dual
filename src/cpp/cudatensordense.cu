#ifndef _CU_DUAL_TENSOR_HPP
#define _CU_DUAL_TENSOR_HPP
#include <hip/hip_runtime.h>
#include <iostream>
#include <complex>
//Utility class to implement dual tensor operations only necessary for QR decomposition
//This is a simplified version of the more extensive Dual class in the original codebase
//and it is implemented using cuBLAS and cuSPARSE for matrix operations
#include <hipblas.h>
#include <hipsparse.h>
#include <thrust/device_vector.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/iterator_traits.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <memory>
#include <vector>
#include <complex>
#include <thrust/device_ptr.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/complex.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/device_vector.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdexcept>
#include <iostream>


namespace janus {

#include <thrust/complex.h>
#include <thrust/device_ptr.h>
#include <thrust/transform.h>
#include <vector>
#include <stdexcept>
#include <hipblas.h>
#include <hip/hip_runtime.h>

template <typename T>
class VectorDenseCuda {
private:
    int batch_size_;  // Batch dimension (M)
    int size_;        // Vector length (N)

    thrust::complex<T>* data_;  // Data [M, N] (complex)
    bool owns_memory_;          // Indicates if memory is managed internally

    // cuBLAS handle
    hipblasHandle_t handle_;
    hipStream_t stream_;

public:
    // Constructor with external memory
    VectorDenseCuda(int batch_size, int size, thrust::complex<T>* data)
        : batch_size_(batch_size), size_(size), data_(data), owns_memory_(false) {
        if (!data_) {
            throw std::invalid_argument("Data pointer is null");
        }
        initializeHandles();
    }

    // Constructor with internal memory allocation
    VectorDenseCuda(int batch_size, int size)
        : batch_size_(batch_size), size_(size), owns_memory_(true) {
        if (batch_size <= 0 || size <= 0) {
            throw std::invalid_argument("Batch size and vector size must be positive.");
        }

        size_t data_size = batch_size * size * sizeof(thrust::complex<T>);
        if (hipMalloc(&data_, data_size) != hipSuccess) {
            throw std::runtime_error("Failed to allocate GPU memory for data.");
        }

        initializeHandles();
    }

    // Destructor
    ~VectorDenseCuda() {
        if (owns_memory_ && data_) {
            hipFree(data_);
        }
        if (handle_) {
            hipblasDestroy(handle_);
        }
        if (stream_) {
            hipStreamDestroy(stream_);
        }
    }

    // Initialize data from host
    void initialize(const thrust::complex<T>* host_data, size_t data_size) {
        if (data_size != batch_size_ * size_) {
            throw std::invalid_argument("Input size does not match vector dimensions.");
        }

        hipMemcpyAsync(data_, host_data, data_size * sizeof(thrust::complex<T>), hipMemcpyHostToDevice, stream_);
        hipStreamSynchronize(stream_);
    }

    // Elementwise addition
    VectorDenseCuda elementwiseAdd(const VectorDenseCuda& other) const {
        if (batch_size_ != other.batch_size_ || size_ != other.size_) {
            throw std::invalid_argument("Vector dimensions do not match for elementwise addition.");
        }

        VectorDenseCuda result(batch_size_, size_);
        int total_elements = batch_size_ * size_;

        thrust::device_ptr<thrust::complex<T>> d_ptr1(data_);
        thrust::device_ptr<thrust::complex<T>> d_ptr2(other.data_);
        thrust::device_ptr<thrust::complex<T>> d_ptr_result(result.data_);

        thrust::transform(
            d_ptr1, d_ptr1 + total_elements,
            d_ptr2,
            d_ptr_result,
            thrust::plus<thrust::complex<T>>());

        return result;
    }

    // Elementwise multiplication
    VectorDenseCuda elementwiseMultiply(const VectorDenseCuda& other) const {
        if (batch_size_ != other.batch_size_ || size_ != other.size_) {
            throw std::invalid_argument("Vector dimensions do not match for elementwise multiplication.");
        }

        VectorDenseCuda result(batch_size_, size_);
        int total_elements = batch_size_ * size_;

        thrust::transform(
            thrust::device_pointer_cast(data_),
            thrust::device_pointer_cast(data_ + total_elements),
            thrust::device_pointer_cast(other.data_),
            thrust::device_pointer_cast(result.data_),
            thrust::multiplies<thrust::complex<T>>());

        return result;
    }

    // Accessors
    thrust::complex<T>* data() { return data_; }
    const thrust::complex<T>* data() const { return data_; }
    int batchSize() const { return batch_size_; }
    int size() const { return size_; }

private:
    // Initialize cuBLAS handle and CUDA stream
    void initializeHandles() {
        if (hipblasCreate(&handle_) != HIPBLAS_STATUS_SUCCESS) {
            throw std::runtime_error("Failed to create cuBLAS handle.");
        }
        if (hipStreamCreate(&stream_) != hipSuccess) {
            hipblasDestroy(handle_);
            throw std::runtime_error("Failed to create CUDA stream.");
        }
        hipblasSetStream(handle_, stream_);
    }
};



#define CUDA_CHECK(call) do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        throw std::runtime_error(hipGetErrorString(err)); \
    } \
} while (0)

#define CUBLAS_CHECK(call) do { \
    hipblasStatus_t err = call; \
    if (err != HIPBLAS_STATUS_SUCCESS) { \
        throw std::runtime_error("cuBLAS error"); \
    } \
} while (0)

template <typename T>
struct BroadcastDualMultiply {
    const thrust::complex<T>* real;
    const thrust::complex<T>* dual;
    int real_size, dual_size;

    BroadcastDualMultiply(const thrust::complex<T>* real, 
                          const thrust::complex<T>* dual, int real_size, int dual_size)
        : real(real), dual(dual), real_size(real_size), dual_size(dual_size) {}

    __device__ thrust::complex<float> operator()(int idx) const {
        int batch_idx = idx / (real_size * dual_size);
        int real_idx = (idx / dual_size) % real_size;
        int dual_idx = idx % dual_size;

        int real_offset = batch_idx * real_size + real_idx;  // Corresponding real tensor index
        return real[real_offset] * dual[idx];
    }
};


template <typename T>
void multiplyDualTensor(const thrust::complex<T>* real,
                        const thrust::complex<T>* dual,
                        thrust::complex<T>* result,
                        int batch_size, int real_size, int dual_size) {
    int total_elements = batch_size * real_size * dual_size;

    thrust::transform(
        thrust::make_counting_iterator(0),
        thrust::make_counting_iterator(total_elements),
        thrust::device_pointer_cast(result),
        [=] __device__(int idx) {
            int batch_idx = idx / (real_size * dual_size);
            int real_idx = (idx / dual_size) % real_size;
            //int dual_idx = idx % dual_size;

            int real_offset = batch_idx * real_size + real_idx;  // Corresponding real tensor index
            return real[real_offset] * dual[idx];
        });
}


template <typename T>
class VectorDualDenseCuda {
private:
    int batch_size_;             // Batch dimension (M)
    int real_size_;              // Vector length (N)
    int dual_size_;              // Dual dimension (D)

    thrust::complex<T>* real_;   // Real part [M, N]
    thrust::complex<T>* dual_;   // Dual part [M, N, D]
    bool owns_memory_;           // Indicates if memory is managed internally

    hipStream_t stream_;        // CUDA stream for asynchronous operations

public:
    // Constructor with external memory
    VectorDualDenseCuda(int batch_size, int real_size, int dual_size,
         thrust::complex<T>* real, thrust::complex<T>* dual)
        : batch_size_(batch_size), real_size_(real_size), dual_size_(dual_size),
          real_(real), dual_(dual), owns_memory_(false) {
        if (!real_ || !dual_) {
            throw std::invalid_argument("Real or dual data pointer is null.");
        }
        initializeStream();
    }

    // Constructor with internal memory allocation
    VectorDualDenseCuda(int batch_size, int real_size, int dual_size)
        : batch_size_(batch_size), real_size_(real_size), dual_size_(dual_size), owns_memory_(true) {
        if (batch_size <= 0 || real_size <= 0 || dual_size <= 0) {
            throw std::invalid_argument("Batch size, vector size, and dual size must be positive.");
        }

        size_t real_data_size = batch_size * real_size * sizeof(thrust::complex<T>);
        size_t dual_data_size = batch_size * real_size * dual_size * sizeof(thrust::complex<T>);

        if (hipMalloc(&real_, real_data_size) != hipSuccess ||
            hipMalloc(&dual_, dual_data_size) != hipSuccess) {
            throw std::runtime_error("Failed to allocate GPU memory for dual number components.");
        }

        initializeStream();
    }

    // Destructor
    ~VectorDualDenseCuda() {
        if (owns_memory_) {
            if (real_) hipFree(real_);
            if (dual_) hipFree(dual_);
        }
        if (stream_) {
            hipStreamDestroy(stream_);
        }
    }

    void initialize(const thrust::complex<T>* host_real, const thrust::complex<T>* host_dual) {
        size_t real_size = batch_size_ * real_size_ * sizeof(thrust::complex<T>);
        size_t dual_size = batch_size_ * real_size_ * dual_size_ * sizeof(thrust::complex<T>);

        hipError_t real_status = hipMemcpyAsync(real_, host_real, real_size, hipMemcpyHostToDevice, stream_);
        hipError_t dual_status = hipMemcpyAsync(dual_, host_dual, dual_size, hipMemcpyHostToDevice, stream_);

        if (real_status != hipSuccess || dual_status != hipSuccess) {
            throw std::runtime_error("Failed to copy data to device.");
        }

        hipStreamSynchronize(stream_);
    }

    // Elementwise addition
    VectorDualDenseCuda elementwiseAdd(const VectorDualDenseCuda& other) const {
        if (batch_size_ != other.batch_size_ || real_size_ != other.real_size_ || dual_size_ != other.dual_size_) {
            throw std::invalid_argument("Dual dimensions do not match for elementwise addition.");
        }

        VectorDualDenseCuda result(batch_size_, real_size_, dual_size_);
        
        int real_total_elements = batch_size_ * real_size_;
        int dual_total_elements = batch_size_ * real_size_ * dual_size_;

        thrust::device_ptr<thrust::complex<T>> real1(real_);
        thrust::device_ptr<thrust::complex<T>> real2(other.real_);
        thrust::device_ptr<thrust::complex<T>> real_result(result.real_);

        thrust::device_ptr<thrust::complex<T>> dual1(dual_);
        thrust::device_ptr<thrust::complex<T>> dual2(other.dual_);
        thrust::device_ptr<thrust::complex<T>> dual_result(result.dual_);

        thrust::transform(real1, real1 + real_total_elements, real2, real_result, thrust::plus<thrust::complex<T>>());
        thrust::transform(dual1, dual1 + dual_total_elements, dual2, dual_result, thrust::plus<thrust::complex<T>>());

        return result;
    }



    VectorDualDenseCuda elementwiseMultiply(const VectorDualDenseCuda& other) const {
        if (batch_size_ != other.batch_size_ || real_size_ != other.real_size_ || dual_size_ != other.dual_size_) {
            throw std::invalid_argument("Dual dimensions do not match for elementwise multiplication.");
        }
        //The real part is multiplied elementwise
        VectorDualDenseCuda result(batch_size_, real_size_, dual_size_);
        int total_elements_real = batch_size_ * real_size_;

        thrust::transform(
            thrust::device_pointer_cast(real_),
            thrust::device_pointer_cast(real_ + total_elements_real),
            thrust::device_pointer_cast(other.real_),
            thrust::device_pointer_cast(result.real_),
            thrust::multiplies<thrust::complex<T>>());

        //Now get the dual part
        int total_elements_dual = batch_size_ * real_size_ * dual_size_;
        //Create a holder for the data
        thrust::device_vector<thrust::complex<T>> result_dual1(total_elements_dual);
        thrust::device_vector<thrust::complex<T>> result_dual2(total_elements_dual);
        multiplyDualTensor<T>(real_,
                              other.dual_,
                              thrust::raw_pointer_cast(result_dual1.data()), 
                              batch_size_, real_size_, dual_size_);
        multiplyDualTensor<T>(other.real_,
                              dual_,
                              thrust::raw_pointer_cast(result_dual2.data()), 
                              batch_size_, real_size_, dual_size_); 

        //Now add the two results
        thrust::transform(
            thrust::device_pointer_cast(result_dual1.data()),
            thrust::device_pointer_cast(result_dual1.data() + total_elements_dual),
            thrust::device_pointer_cast(result_dual2.data()),
            thrust::device_pointer_cast(result.dual_),
            thrust::plus<thrust::complex<T>>());


        return result;
    }

    // Accessors
    thrust::complex<T>* real() { return real_; }
    const thrust::complex<T>* real() const { return real_; }
    thrust::complex<T>* dual() { return dual_; }
    const thrust::complex<T>* dual() const { return dual_; }
    int batchSize() const { return batch_size_; }
    int size() const { return real_size_; }
    int dualSize() const { return dual_size_; }

private:
    // Initialize CUDA stream
    void initializeStream() {
        if (hipStreamCreate(&stream_) != hipSuccess) {
            throw std::runtime_error("Failed to create CUDA stream.");
        }
    }
};

template <typename T>
class VectorHyperDualDense {
private:
    using ComplexT = std::complex<T>;

    // Dimensions
    int batch_size_;   // M
    int size_;         // N (length of each vector)
    int dual_dim_;     // D (number of dual components)

    // Primal, Dual, and Hyper-Dual Data
    ComplexT* primal_data_;      // [M, N]
    ComplexT* dual_data_;        // [M, N, D]
    ComplexT* hyper_dual_data_;  // [M, N, D, D]
    bool owns_memory_;           // Indicates if memory is managed internally

    // cuBLAS handle
    hipblasHandle_t handle_;
    hipStream_t stream_;

public:
    // Constructor with external memory
    VectorHyperDualDense(int batch_size, int size, int dual_dim, ComplexT* primal_data, ComplexT* dual_data, ComplexT* hyper_dual_data)
        : batch_size_(batch_size), size_(size), dual_dim_(dual_dim),
          primal_data_(primal_data), dual_data_(dual_data), hyper_dual_data_(hyper_dual_data),
          owns_memory_(false) {
        if (!primal_data_ || !dual_data_ || !hyper_dual_data_) {
            throw std::invalid_argument("Primal, dual, or hyper-dual data pointer is null.");
        }
        initializeHandles();
    }

    // Constructor with internal memory allocation
    VectorHyperDualDense(int batch_size, int size, int dual_dim)
        : batch_size_(batch_size), size_(size), dual_dim_(dual_dim), owns_memory_(true) {
        if (batch_size <= 0 || size <= 0 || dual_dim <= 0) {
            throw std::invalid_argument("All dimensions must be positive.");
        }

        size_t primal_size = batch_size * size * sizeof(ComplexT);
        size_t dual_size = batch_size * size * dual_dim * sizeof(ComplexT);
        size_t hyper_dual_size = batch_size * size * dual_dim * dual_dim * sizeof(ComplexT);

        if (hipMalloc(&primal_data_, primal_size) != hipSuccess) {
            throw std::runtime_error("Failed to allocate GPU memory for primal data.");
        }

        if (hipMalloc(&dual_data_, dual_size) != hipSuccess) {
            hipFree(primal_data_);
            throw std::runtime_error("Failed to allocate GPU memory for dual data.");
        }

        if (hipMalloc(&hyper_dual_data_, hyper_dual_size) != hipSuccess) {
            hipFree(primal_data_);
            hipFree(dual_data_);
            throw std::runtime_error("Failed to allocate GPU memory for hyper-dual data.");
        }

        initializeHandles();
    }

    ~VectorHyperDualDense() {
        if (owns_memory_) {
            if (primal_data_) hipFree(primal_data_);
            if (dual_data_) hipFree(dual_data_);
            if (hyper_dual_data_) hipFree(hyper_dual_data_);
        }
        hipblasDestroy(handle_);
        hipStreamDestroy(stream_);
    }

    void initialize(const ComplexT* primal, const ComplexT* dual, const ComplexT* hyper_dual, size_t primal_size, size_t dual_size, size_t hyper_dual_size) {
        if (primal_size != batch_size_ * size_ ||
            dual_size != batch_size_ * size_ * dual_dim_ ||
            hyper_dual_size != batch_size_ * size_ * dual_dim_ * dual_dim_) {
            throw std::invalid_argument("Input sizes do not match tensor dimensions.");
        }

        if (primal) {
            hipMemcpyAsync(primal_data_, primal, primal_size * sizeof(ComplexT), hipMemcpyHostToDevice, stream_);
        }
        if (dual) {
            hipMemcpyAsync(dual_data_, dual, dual_size * sizeof(ComplexT), hipMemcpyHostToDevice, stream_);
        }
        if (hyper_dual) {
            hipMemcpyAsync(hyper_dual_data_, hyper_dual, hyper_dual_size * sizeof(ComplexT), hipMemcpyHostToDevice, stream_);
        }
        hipStreamSynchronize(stream_);
    }

    // Example elementwise addition
    VectorHyperDualDense<T> elementwiseAdd(const VectorHyperDualDense<T>& other) const {
        if (batch_size_ != other.batch_size_ || size_ != other.size_ || dual_dim_ != other.dual_dim_) {
            throw std::invalid_argument("Tensor dimensions do not match for elementwise addition.");
        }

        VectorHyperDualDense<T> result(batch_size_, size_, dual_dim_);

        int total_primal_elements = batch_size_ * size_;
        int total_dual_elements = total_primal_elements * dual_dim_;
        int total_hyper_dual_elements = total_dual_elements * dual_dim_;

        // Perform elementwise addition for the primal part
        thrust::transform(
            thrust::device_pointer_cast(primal_data_),
            thrust::device_pointer_cast(primal_data_ + total_primal_elements),
            thrust::device_pointer_cast(other.primal_data_),
            thrust::device_pointer_cast(result.primal_data_),
            thrust::plus<ComplexT>());

        // Perform elementwise addition for the dual part
        thrust::transform(
            thrust::device_pointer_cast(dual_data_),
            thrust::device_pointer_cast(dual_data_ + total_dual_elements),
            thrust::device_pointer_cast(other.dual_data_),
            thrust::device_pointer_cast(result.dual_data_),
            thrust::plus<ComplexT>());

        // Perform elementwise addition for the hyper-dual part
        thrust::transform(
            thrust::device_pointer_cast(hyper_dual_data_),
            thrust::device_pointer_cast(hyper_dual_data_ + total_hyper_dual_elements),
            thrust::device_pointer_cast(other.hyper_dual_data_),
            thrust::device_pointer_cast(result.hyper_dual_data_),
            thrust::plus<ComplexT>());

        return result;
    }

private:
    void initializeHandles() {
        if (hipblasCreate(&handle_) != HIPBLAS_STATUS_SUCCESS) {
            throw std::runtime_error("Failed to create cuBLAS handle.");
        }
        if (hipStreamCreate(&stream_) != hipSuccess) {
            hipblasDestroy(handle_);
            throw std::runtime_error("Failed to create CUDA stream.");
        }
        hipblasSetStream(handle_, stream_);
    }

public:
    // Disable copy constructor and copy assignment
    VectorHyperDualDense(const VectorHyperDualDense&) = delete;
    VectorHyperDualDense& operator=(const VectorHyperDualDense&) = delete;

    // Enable move constructor and move assignment
    VectorHyperDualDense(VectorHyperDualDense&&) noexcept = default;
    VectorHyperDualDense& operator=(VectorHyperDualDense&&) noexcept = default;

    // Getters for dimensions
    int batch_size() const { return batch_size_; }
    int size() const { return size_; }
    int dual_dim() const { return dual_dim_; }

    // Getters for data pointers
    ComplexT* primal_data() { return primal_data_; }
    ComplexT* dual_data() { return dual_data_; }
    ComplexT* hyper_dual_data() { return hyper_dual_data_; }

    const ComplexT* primal_data() const { return primal_data_; }
    const ComplexT* dual_data() const { return dual_data_; }
    const ComplexT* hyper_dual_data() const { return hyper_dual_data_; }
};


template <typename T>
class MatrixDense {
private:
    using ComplexT = std::complex<T>;

    // Dimensions
    int batch_size_;
    int rows_;
    int cols_;

    // Primal Data
    ComplexT* primal_data_; // Device-side primal part
    bool owns_memory_;      // Indicates if memory is managed internally

    // cuBLAS handle
    hipblasHandle_t handle_;
    hipStream_t stream_;

public:
    // Constructor with external memory
    MatrixDense(int batch_size, int rows, int cols, ComplexT* primal_data)
        : batch_size_(batch_size), rows_(rows), cols_(cols),
          primal_data_(primal_data), owns_memory_(false) {
        if (!primal_data_) {
            throw std::invalid_argument("Primal data pointer is null");
        }

        initializeHandles();
    }

    // Constructor with internal memory allocation
    MatrixDense(int batch_size, int rows, int cols)
        : batch_size_(batch_size), rows_(rows), cols_(cols), owns_memory_(true) {
        if (batch_size <= 0 || rows <= 0 || cols <= 0) {
            throw std::invalid_argument("All dimensions must be positive.");
        }

        size_t primal_size = batch_size * rows * cols * sizeof(ComplexT);

        if (hipMalloc(&primal_data_, primal_size) != hipSuccess) {
            throw std::runtime_error("Failed to allocate GPU memory for primal data.");
        }

        initializeHandles();
    }

    ~MatrixDense() {
        if (owns_memory_ && primal_data_) {
            hipFree(primal_data_);
        }
        hipblasDestroy(handle_);
        hipStreamDestroy(stream_);
    }

    void initialize(const ComplexT* primal, size_t primal_size) {
        if (primal_size != batch_size_ * rows_ * cols_) {
            throw std::invalid_argument("Input size does not match tensor dimensions.");
        }

        if (primal) {
            hipMemcpyAsync(primal_data_, primal, primal_size * sizeof(ComplexT), hipMemcpyHostToDevice, stream_);
        }
        hipStreamSynchronize(stream_);
    }

    template <typename U>
    MatrixDense<U> indexGet(int start_row, int end_row, int start_col, int end_col) const {
        // Validate row and column ranges
        if (start_row < 0 || end_row > rows_ || start_row >= end_row ||
            start_col < 0 || end_col > cols_ || start_col >= end_col) {
            throw std::invalid_argument("Invalid row or column range for indexGet.");
        }

        // Dimensions of the submatrix
        int sub_rows = end_row - start_row;
        int sub_cols = end_col - start_col;

        // Calculate the offset for the primal data
        ComplexT* sub_primal_data = primal_data_ + start_row * cols_ + start_col;

        // Create a new MatrixDense instance sharing the data with the original
        return MatrixDense<T>(batch_size_, sub_rows, sub_cols, sub_primal_data);
    }


    template <typename U>
    void indexPut(int start_row, int end_row, int start_col, int end_col, const MatrixDense<U>& data) {
        // Validate row and column ranges
        if (start_row < 0 || end_row > rows_ || start_row >= end_row ||
            start_col < 0 || end_col > cols_ || start_col >= end_col) {
            throw std::invalid_argument("Invalid row or column range for indexPut.");
        }

        // Validate dimensions of the input data
        if (data.rows_ != (end_row - start_row) || data.cols_ != (end_col - start_col) || 
            data.batch_size_ != batch_size_) {
            throw std::invalid_argument("Input data dimensions do not match the target range.");
        }

        // Calculate the offset for the primal data
        ComplexT* target_primal_data = primal_data_ + start_row * cols_ + start_col;

        // Update primal data
        for (int b = 0; b < batch_size_; ++b) {
            hipMemcpy2DAsync(target_primal_data + b * rows_ * cols_,
                            cols_ * sizeof(ComplexT),
                            data.primal_data_ + b * data.rows_ * data.cols_,
                            data.cols_ * sizeof(ComplexT),
                            data.cols_ * sizeof(ComplexT),
                            data.rows_,
                            hipMemcpyDeviceToDevice,
                            stream_);
        }

        // Synchronize to ensure data transfer is complete
        hipStreamSynchronize(stream_);
    }


    MatrixDense<T> multiply(const MatrixDense<T>& other) const {
        // Validate dimensions
        if (cols_ != other.rows_ || batch_size_ != other.batch_size_) {
            throw std::invalid_argument("Matrix dimensions do not match for multiplication.");
        }

        // Create the result matrix
        MatrixDense<T> result(batch_size_, rows_, other.cols_);

        // Scaling factors for cuBLAS
        ComplexT alpha = ComplexT(1.0, 0.0);
        ComplexT beta = ComplexT(0.0, 0.0);

        // Perform batched matrix multiplication
        hipblasStatus_t status = hipblasZgemmStridedBatched(
            handle_,
            HIPBLAS_OP_N,           // No transpose for this matrix
            HIPBLAS_OP_N,           // No transpose for the other matrix
            rows_,                 // Number of rows of the output matrix
            other.cols_,           // Number of columns of the output matrix
            cols_,                 // Shared dimension (this.cols_ == other.rows_)
            &alpha,                // Scaling factor for the multiplication
            primal_data_,          // Pointer to this matrix data
            rows_,                 // Leading dimension of this matrix
            rows_ * cols_,         // Stride between consecutive matrices in the batch
            other.primal_data_,    // Pointer to other matrix data
            other.rows_,           // Leading dimension of the other matrix
            other.rows_ * other.cols_, // Stride between consecutive matrices in the batch
            &beta,                 // Scaling factor for the result matrix
            result.primal_data_,   // Pointer to result matrix data
            result.rows_,          // Leading dimension of the result matrix
            result.rows_ * result.cols_, // Stride between consecutive matrices in the batch
            batch_size_            // Number of matrices in the batch
        );

        if (status != HIPBLAS_STATUS_SUCCESS) {
            throw std::runtime_error("cuBLAS matrix multiplication failed.");
        }

        return result;
    }

    VectorDenseCuda<T> matrixVectorProduct(const VectorDenseCuda<T>& vector) const {
        // Validate dimensions
        if (cols_ != vector.size() || batch_size_ != vector.batch_size()) {
            throw std::invalid_argument("Matrix and vector dimensions do not match for multiplication.");
        }

        // Create the result vector
        VectorDenseCuda<T> result(batch_size_, rows_);

        // Scaling factors for cuBLAS
        ComplexT alpha = ComplexT(1.0, 0.0);
        ComplexT beta = ComplexT(0.0, 0.0);

        // Perform batched matrix-vector multiplication
        for (int b = 0; b < batch_size_; ++b) {
            hipblasStatus_t status = hipblasZgemv(
                handle_,
                HIPBLAS_OP_N,                            // No transpose for this matrix
                rows_, cols_,                           // Dimensions of the matrix
                &alpha,                                 // Scaling factor for multiplication
                primal_data_ + b * rows_ * cols_,       // Pointer to the matrix for this batch
                rows_,                                  // Leading dimension of the matrix
                vector.primal_data() + b * vector.size(), // Pointer to the vector for this batch
                1,                                      // Stride for the vector
                &beta,                                  // Scaling factor for the result
                result.primal_data() + b * rows_,       // Pointer to the result vector for this batch
                1                                       // Stride for the result vector
            );

            if (status != HIPBLAS_STATUS_SUCCESS) {
                throw std::runtime_error("cuBLAS matrix-vector multiplication failed.");
            }
        }

        return result;
    }

    MatrixDense<T> transpose() const {
        MatrixDense<T> result(batch_size_, cols_, rows_);
        ComplexT alpha = ComplexT(1.0, 0.0);
        ComplexT beta = ComplexT(0.0, 0.0);

        for (int b = 0; b < batch_size_; ++b) {
            hipblasZgeam(handle_,
                        HIPBLAS_OP_T, HIPBLAS_OP_T,
                        cols_, rows_,
                        &alpha,
                        primal_data_ + b * rows_ * cols_, rows_,
                        &beta,
                        nullptr, cols_,
                        result.primal_data_ + b * cols_ * rows_, cols_);
        }

        return result;
    }

    MatrixDense<T> elementwiseAdd(const MatrixDense<T>& other) const {
        if (batch_size_ != other.batch_size_ || rows_ != other.rows_ || cols_ != other.cols_) {
            throw std::invalid_argument("Tensor dimensions do not match for addition.");
        }

        MatrixDense<T> result(batch_size_, rows_, cols_);
            int total_elements = batch_size_ * rows_ * cols_;

            thrust::transform(thrust::device_pointer_cast(primal_data_),
                            thrust::device_pointer_cast(primal_data_ + total_elements),
                            thrust::device_pointer_cast(other.primal_data_),
                            thrust::device_pointer_cast(result.primal_data_),
                            thrust::plus<ComplexT>());

            return result;
    }

    MatrixDense<T> square() const {
        // Create a new MatrixDense object to store the result
        MatrixDense<T> result(batch_size_, rows_, cols_);

        // Calculate the total number of elements in the tensor
        int total_elements = batch_size_ * rows_ * cols_;

        // Use thrust to perform element-wise squaring of the tensor
        thrust::transform(
            thrust::device_pointer_cast(primal_data_), 
            thrust::device_pointer_cast(primal_data_ + total_elements),
            thrust::device_pointer_cast(result.primal_data()),
            [] __device__(ComplexT x) { return x * x; });

        return result;
    }

    MatrixDense<T> upperTriangular() const {
        // Create a result matrix with the same dimensions
        MatrixDense<T> result(batch_size_, rows_, cols_);

        size_t total_elements = rows_ * cols_;

        for (int b = 0; b < batch_size_; ++b) {
            // Primal data pointers for this batch
            ComplexT* batch_primal_src = primal_data_ + b * total_elements;
            ComplexT* batch_primal_dst = result.primal_data_ + b * total_elements;

            // Apply the upper triangular operation
            thrust::for_each(
                thrust::counting_iterator<int>(0),
                thrust::counting_iterator<int>(rows_ * cols_),
                [=] __device__(int idx) {
                    int row = idx / cols_;
                    int col = idx % cols_;
                    batch_primal_dst[idx] = (row <= col) ? batch_primal_src[idx] : ComplexT(0.0, 0.0);
                });
        }

        return result;
    }

    MatrixDense<T> lowerTriangular() const {
        // Create a result matrix with the same dimensions
        MatrixDense<T> result(batch_size_, rows_, cols_);

        size_t total_elements = rows_ * cols_;

        for (int b = 0; b < batch_size_; ++b) {
            // Primal data pointers for this batch
            ComplexT* batch_primal_src = primal_data_ + b * total_elements;
            ComplexT* batch_primal_dst = result.primal_data_ + b * total_elements;

            // Apply the lower triangular operation
            thrust::for_each(
                thrust::counting_iterator<int>(0),
                thrust::counting_iterator<int>(rows_ * cols_),
                [=] __device__(int idx) {
                    int row = idx / cols_;
                    int col = idx % cols_;
                    batch_primal_dst[idx] = (row >= col) ? batch_primal_src[idx] : ComplexT(0.0, 0.0);
                });
        }

        return result;
    }




private:
    void initializeHandles() {
        if (hipblasCreate(&handle_) != HIPBLAS_STATUS_SUCCESS) {
            throw std::runtime_error("Failed to create cuBLAS handle.");
        }
        if (hipStreamCreate(&stream_) != hipSuccess) {
            hipblasDestroy(handle_);
            throw std::runtime_error("Failed to create CUDA stream.");
        }
        hipblasSetStream(handle_, stream_);
    }

public:
    // Disable copy constructor and copy assignment
    MatrixDense(const MatrixDense&) = delete;
    MatrixDense& operator=(const MatrixDense&) = delete;

    // Enable move constructor and move assignment
    MatrixDense(MatrixDense&&) noexcept = default;
    MatrixDense& operator=(MatrixDense&&) noexcept = default;

    // Getters for dimensions
    int batch_size() const { return batch_size_; }
    int rows() const { return rows_; }
    int cols() const { return cols_; }

    // Getters for data pointers
    ComplexT* primal_data() { return primal_data_; }
    const ComplexT* primal_data() const { return primal_data_; }
};




template <typename T>
class MatrixDualDense {
private:
    using ComplexT = std::complex<T>;

    // Dimensions
    int batch_size_;
    int rows_;
    int cols_;
    int dual_dim_;

    // Primal and Dual Data
    ComplexT* primal_data_; // Device-side primal part
    ComplexT* dual_data_;   // Device-side dual part
    bool owns_memory_;      // Indicates if memory is managed internally

    // cuBLAS handle
    hipblasHandle_t handle_;
    hipStream_t stream_;

public:
    // Constructor with external memory
    MatrixDualDense(int batch_size, int rows, int cols, int dual_dim, ComplexT* primal_data, ComplexT* dual_data)
        : batch_size_(batch_size), rows_(rows), cols_(cols), dual_dim_(dual_dim),
          primal_data_(primal_data), dual_data_(dual_data), owns_memory_(false) {
        if (!primal_data_ || !dual_data_) {
            throw std::invalid_argument("Primal or dual data pointer is null");
        }

        initializeHandles();
    }

    // Constructor with internal memory allocation
    MatrixDualDense(int batch_size, int rows, int cols, int dual_dim)
        : batch_size_(batch_size), rows_(rows), cols_(cols), dual_dim_(dual_dim), owns_memory_(true) {
        if (batch_size <= 0 || rows <= 0 || cols <= 0 || dual_dim <= 0) {
            throw std::invalid_argument("All dimensions must be positive.");
        }

        size_t primal_size = batch_size * rows * cols * sizeof(ComplexT);
        size_t dual_size = batch_size * rows * cols * dual_dim * sizeof(ComplexT);

        if (hipMalloc(&primal_data_, primal_size) != hipSuccess) {
            throw std::runtime_error("Failed to allocate GPU memory for primal data.");
        }

        if (hipMalloc(&dual_data_, dual_size) != hipSuccess) {
            hipFree(primal_data_);
            throw std::runtime_error("Failed to allocate GPU memory for dual data.");
        }

        initializeHandles();
    }

    ~MatrixDualDense() {
        if (owns_memory_) {
            if (primal_data_) hipFree(primal_data_);
            if (dual_data_) hipFree(dual_data_);
        }
        hipblasDestroy(handle_);
        hipStreamDestroy(stream_);
    }

    void initialize(const ComplexT* primal, const ComplexT* dual, size_t primal_size, size_t dual_size) {
        if (primal_size != batch_size_ * rows_ * cols_ || dual_size != batch_size_ * rows_ * cols_ * dual_dim_) {
            throw std::invalid_argument("Input sizes do not match tensor dimensions.");
        }

        if (primal) {
            hipMemcpyAsync(primal_data_, primal, primal_size * sizeof(ComplexT), hipMemcpyHostToDevice, stream_);
        }
        if (dual) {
            hipMemcpyAsync(dual_data_, dual, dual_size * sizeof(ComplexT), hipMemcpyHostToDevice, stream_);
        }
        hipStreamSynchronize(stream_);
    }

    template <typename U>
    MatrixDualDense<U> indexGet(int start_row, int end_row, int start_col, int end_col) const {
        // Validate row and column ranges
        if (start_row < 0 || end_row > rows_ || start_row >= end_row ||
            start_col < 0 || end_col > cols_ || start_col >= end_col) {
            throw std::invalid_argument("Invalid row or column range for indexGet.");
        }

        // Dimensions of the submatrix
        int sub_rows = end_row - start_row;
        int sub_cols = end_col - start_col;

        // Calculate offsets for the primal and dual data
        ComplexT* sub_primal_data = primal_data_ + start_row * cols_ + start_col;
        ComplexT* sub_dual_data = dual_data_ + start_row * cols_ * dual_dim_ + start_col * dual_dim_;

        // Create a new MatrixDualDense instance sharing the data with the original
        return MatrixDualDense<T>(batch_size_, sub_rows, sub_cols, dual_dim_, sub_primal_data, sub_dual_data);
    }

    template <typename U>
    void indexPut(int start_row, int end_row, int start_col, int end_col, const MatrixDualDense<U>& data) {
        // Validate row and column ranges
        if (start_row < 0 || end_row > rows_ || start_row >= end_row ||
            start_col < 0 || end_col > cols_ || start_col >= end_col) {
            throw std::invalid_argument("Invalid row or column range for indexPut.");
        }

        // Validate dimensions of the input data
        if (data.rows_ != (end_row - start_row) || data.cols_ != (end_col - start_col) || 
            data.dual_dim_ != dual_dim_ || data.batch_size_ != batch_size_) {
            throw std::invalid_argument("Input data dimensions do not match the target range.");
        }

        // Calculate offsets for the primal and dual data
        ComplexT* target_primal_data = primal_data_ + start_row * cols_ + start_col;
        ComplexT* target_dual_data = dual_data_ + start_row * cols_ * dual_dim_ + start_col * dual_dim_;

        // Calculate data sizes
        size_t primal_size = data.rows_ * data.cols_ * sizeof(ComplexT);
        size_t dual_size = data.rows_ * data.cols_ * dual_dim_ * sizeof(ComplexT);

        // Update primal data
        for (int b = 0; b < batch_size_; ++b) {
            hipMemcpy2DAsync(target_primal_data + b * rows_ * cols_,
                            cols_ * sizeof(ComplexT),
                            data.primal_data_ + b * data.rows_ * data.cols_,
                            data.cols_ * sizeof(ComplexT),
                            data.cols_ * sizeof(ComplexT),
                            data.rows_,
                            hipMemcpyDeviceToDevice,
                            stream_);
        }

        // Update dual data
        for (int b = 0; b < batch_size_; ++b) {
            hipMemcpy2DAsync(target_dual_data + b * rows_ * cols_ * dual_dim_,
                            cols_ * dual_dim_ * sizeof(ComplexT),
                            data.dual_data_ + b * data.rows_ * data.cols_ * dual_dim_,
                            data.cols_ * dual_dim_ * sizeof(ComplexT),
                            data.cols_ * dual_dim_ * sizeof(ComplexT),
                            data.rows_,
                            hipMemcpyDeviceToDevice,
                            stream_);
        }

        // Synchronize to ensure data transfer is complete
        hipStreamSynchronize(stream_);
    }    

    void square() {
        // Element-wise square for the primal part
        size_t total_primal_elements = batch_size_ * rows_ * cols_;
        size_t total_dual_elements = batch_size_ * rows_ * cols_ * dual_dim_;

        // Kernel to compute element-wise square
        auto squareKernel = [] __device__(ComplexT x) -> ComplexT {
            return x * x;
        };

        // Launch a CUDA kernel to square the primal part
        thrust::device_ptr<ComplexT> primal_ptr(primal_data_);
        thrust::transform(thrust::device, primal_ptr, primal_ptr + total_primal_elements, primal_ptr, squareKernel);

        // Update the dual part according to the product rule:
        // If u = f(x) and v = f'(x), then square(u) has derivative: 2 * u * v.
        auto dualKernel = [] __device__(ComplexT u, ComplexT v) -> ComplexT {
            return ComplexT(2.0, 0.0) * u * v;
        };

        // Process the dual part
        for (int d = 0; d < dual_dim_; ++d) {
            ComplexT* dual_ptr = dual_data_ + d * batch_size_ * rows_ * cols_;
            thrust::device_ptr<ComplexT> dual_thrust_ptr(dual_ptr);
            thrust::transform(thrust::device, primal_ptr, primal_ptr + total_primal_elements,
                            dual_thrust_ptr, dual_thrust_ptr, dualKernel);
        }

        hipStreamSynchronize(stream_);
    }




    template <typename U>
    MatrixDualDense<U> sum(int dimension) const {
        if (dimension != 1 && dimension != 2) {
            throw std::invalid_argument("Dimension must be 1 (rows) or 2 (columns).");
        }

        // Determine the new dimensions after summing along the specified axis
        int new_rows = (dimension == 1) ? 1 : rows_;
        int new_cols = (dimension == 2) ? 1 : cols_;

        // Create the resulting MatrixDualDense object
        MatrixDualDense<T> result(batch_size_, new_rows, new_cols, dual_dim_);

        // Allocate memory for temporary host buffers
        size_t primal_size = rows_ * cols_;
        size_t dual_size = rows_ * cols_ * dual_dim_;

        // Perform the summation along the specified dimension
        for (int b = 0; b < batch_size_; ++b) {
            if (dimension == 1) {
                // Summing along rows
                for (int c = 0; c < cols_; ++c) {
                    ComplexT sum_primal = ComplexT(0.0, 0.0);
                    std::vector<ComplexT> sum_dual(dual_dim_, ComplexT(0.0, 0.0));

                    for (int r = 0; r < rows_; ++r) {
                        int idx = b * rows_ * cols_ + r * cols_ + c;
                        sum_primal += primal_data_[idx];

                        for (int d = 0; d < dual_dim_; ++d) {
                            int dual_idx = b * rows_ * cols_ * dual_dim_ + r * cols_ * dual_dim_ + c * dual_dim_ + d;
                            sum_dual[d] += dual_data_[dual_idx];
                        }
                    }

                    // Store the result in the output matrix
                    int result_idx = b * new_rows * new_cols + c;
                    result.primal_data_[result_idx] = sum_primal;

                    for (int d = 0; d < dual_dim_; ++d) {
                        int result_dual_idx = b * new_rows * new_cols * dual_dim_ + c * dual_dim_ + d;
                        result.dual_data_[result_dual_idx] = sum_dual[d];
                    }
                }
            } else if (dimension == 2) {
                // Summing along columns
                for (int r = 0; r < rows_; ++r) {
                    ComplexT sum_primal = ComplexT(0.0, 0.0);
                    std::vector<ComplexT> sum_dual(dual_dim_, ComplexT(0.0, 0.0));

                    for (int c = 0; c < cols_; ++c) {
                        int idx = b * rows_ * cols_ + r * cols_ + c;
                        sum_primal += primal_data_[idx];

                        for (int d = 0; d < dual_dim_; ++d) {
                            int dual_idx = b * rows_ * cols_ * dual_dim_ + r * cols_ * dual_dim_ + c * dual_dim_ + d;
                            sum_dual[d] += dual_data_[dual_idx];
                        }
                    }

                    // Store the result in the output matrix
                    int result_idx = b * new_rows * new_cols + r;
                    result.primal_data_[result_idx] = sum_primal;

                    for (int d = 0; d < dual_dim_; ++d) {
                        int result_dual_idx = b * new_rows * new_cols * dual_dim_ + r * dual_dim_ + d;
                        result.dual_data_[result_dual_idx] = sum_dual[d];
                    }
                }
            }
        }

        hipStreamSynchronize(stream_);
        return result;
    }

    VectorDualDenseCuda<T> squeeze(int dim) const {
        // Check the validity of the dimension
        if (dim < 1 || dim > 2) {
            throw std::invalid_argument("Dimension to squeeze must be 1 (rows) or 2 (columns).");
        }

        // Ensure the specified dimension has size 1
        if ((dim == 1 && rows_ != 1) || (dim == 2 && cols_ != 1)) {
            throw std::invalid_argument("Cannot squeeze a dimension with size greater than 1.");
        }

        // Determine the size of the resulting vector
        int vector_size = (dim == 1) ? cols_ : rows_;

        // Calculate the pointer to primal and dual data
        ComplexT* squeezed_primal_data = primal_data_;
        ComplexT* squeezed_dual_data = dual_data_;

        // Create and return a VectorDual object
        return VectorDualDenseCuda<T>(vector_size, dual_dim_, squeezed_primal_data, squeezed_dual_data);
    }

    template <typename U>
    VectorDualDenseCuda<U> matrixVectorProduct(const VectorDualDenseCuda<U>& vector) const {
        // Validate dimensions
        if (cols_ != vector.size_ || batch_size_ != vector.batch_size_ || dual_dim_ != vector.dual_dim_) {
            throw std::invalid_argument("Matrix and vector dimensions do not match for multiplication.");
        }

        // Result vector
        VectorDualDenseCuda<T> result(batch_size_, rows_, dual_dim_);

        int matrix_primal_size = rows_ * cols_;
        int vector_primal_size = vector.size_;
        int result_primal_size = rows_;

        int matrix_dual_size = matrix_primal_size * dual_dim_;
        int vector_dual_size = vector_primal_size * dual_dim_;
        int result_dual_size = result_primal_size * dual_dim_;

        ComplexT alpha = ComplexT(1.0, 0.0);
        ComplexT beta = ComplexT(0.0, 0.0);

        // Perform matrix-vector multiplication for primal part
        for (int b = 0; b < batch_size_; ++b) {
            hipblasZgemv(handle_,
                        HIPBLAS_OP_N,
                        rows_, cols_,
                        &alpha,
                        primal_data_ + b * matrix_primal_size, rows_,
                        vector.primal_data_ + b * vector_primal_size, 1,
                        &beta,
                        result.primal_data_ + b * result_primal_size, 1);
        }

        // Perform matrix-vector multiplication for dual part
        for (int b = 0; b < batch_size_; ++b) {
            for (int d = 0; d < dual_dim_; ++d) {
                // Matrix * Dual(Vector)
                hipblasZgemv(handle_,
                            HIPBLAS_OP_N,
                            rows_, cols_,
                            &alpha,
                            primal_data_ + b * matrix_primal_size, rows_,
                            vector.dual_data_ + b * vector_primal_size * dual_dim_ + d * vector_primal_size, 1,
                            &beta,
                            result.dual_data_ + b * result_primal_size * dual_dim_ + d * result_primal_size, 1);

                // Dual(Matrix) * Vector
                hipblasZgemv(handle_,
                            HIPBLAS_OP_N,
                            rows_, cols_,
                            &alpha,
                            dual_data_ + b * matrix_dual_size + d * matrix_primal_size, rows_,
                            vector.primal_data_ + b * vector_primal_size, 1,
                            &alpha, // Accumulate
                            result.dual_data_ + b * result_primal_size * dual_dim_ + d * result_primal_size, 1);
            }
        }

        return result;
    }


    MatrixDualDense<T> transpose() const {
        MatrixDualDense<T> result(batch_size_, cols_, rows_, dual_dim_);
        ComplexT alpha = ComplexT(1.0, 0.0);
        ComplexT beta = ComplexT(0.0, 0.0);

        for (int b = 0; b < batch_size_; ++b) {
            hipblasZgeam(handle_,
                        HIPBLAS_OP_T, HIPBLAS_OP_T,
                        cols_, rows_,
                        &alpha,
                        primal_data_ + b * rows_ * cols_, rows_,
                        &beta,
                        nullptr, cols_,
                        result.primal_data_ + b * cols_ * rows_, cols_);
        }

        for (int b = 0; b < batch_size_; ++b) {
            for (int d = 0; d < dual_dim_; ++d) {
                hipblasZgeam(handle_,
                            HIPBLAS_OP_T, HIPBLAS_OP_T,
                            cols_, rows_,
                            &alpha,
                            dual_data_ + b * rows_ * cols_ * dual_dim_ + d * rows_ * cols_, rows_,
                            &beta,
                            nullptr, cols_,
                            result.dual_data_ + b * cols_ * rows_ * dual_dim_ + d * cols_ * rows_, cols_);
            }
        }

        return result;
    }

    // Method to generate an upper triangular matrix
    MatrixDualDense<T> upperTriangular() const {
        // Create a new matrix for the result
        MatrixDualDense<T> result(batch_size_, rows_, cols_, dual_dim_);

        size_t total_elements = rows_ * cols_;

        for (int b = 0; b < batch_size_; ++b) {
            // Initialize primal part to upper triangular
            ComplexT* batch_primal_src = primal_data_ + b * total_elements;
            ComplexT* batch_primal_dst = result.primal_data_ + b * total_elements;

            thrust::for_each(
                thrust::counting_iterator<int>(0),
                thrust::counting_iterator<int>(rows_ * cols_),
                [=] __device__(int idx) {
                    int row = idx / cols_;
                    int col = idx % cols_;
                    batch_primal_dst[idx] = (row <= col) ? batch_primal_src[idx] : ComplexT(0.0, 0.0);
                });

            // Initialize dual part to zero
            ComplexT* batch_dual_dst = result.dual_data_ + b * total_elements * dual_dim_;
            thrust::fill(thrust::device_pointer_cast(batch_dual_dst),
                         thrust::device_pointer_cast(batch_dual_dst + total_elements * dual_dim_),
                         ComplexT(0.0, 0.0));
        }

        return result;
    }


    // Method to generate a lower triangular matrix
    MatrixDualDense<T> lowerTriangular() const {
            // Create a new matrix for the result
            MatrixDualDense<T> result(batch_size_, rows_, cols_, dual_dim_);

            size_t total_elements = rows_ * cols_;

            for (int b = 0; b < batch_size_; ++b) {
                // Initialize primal part to lower triangular
                ComplexT* batch_primal_src = primal_data_ + b * total_elements;
                ComplexT* batch_primal_dst = result.primal_data_ + b * total_elements;

                thrust::for_each(
                    thrust::counting_iterator<int>(0),
                    thrust::counting_iterator<int>(rows_ * cols_),
                    [=] __device__(int idx) {
                        int row = idx / cols_;
                        int col = idx % cols_;
                        batch_primal_dst[idx] = (row >= col) ? batch_primal_src[idx] : ComplexT(0.0, 0.0);
                    });

                // Initialize dual part to zero
                ComplexT* batch_dual_dst = result.dual_data_ + b * total_elements * dual_dim_;
                thrust::fill(thrust::device_pointer_cast(batch_dual_dst),
                            thrust::device_pointer_cast(batch_dual_dst + total_elements * dual_dim_),
                            ComplexT(0.0, 0.0));
            }

            return result;
        }

    template <typename U>
    MatrixDualDense<U> matrixMultiply(const MatrixDualDense<U>& other) const {
        // Validate dimensions
        if (cols_ != other.rows_ || batch_size_ != other.batch_size_ || dual_dim_ != other.dual_dim_) {
            throw std::invalid_argument("Matrix dimensions do not match for multiplication.");
        }

        // Create the result matrix
        MatrixDualDense<T> result(batch_size_, rows_, other.cols_, dual_dim_);

        ComplexT alpha = ComplexT(1.0, 0.0);
        ComplexT beta = ComplexT(0.0, 0.0);

        // Perform batched matrix multiplication for primal and dual parts
        for (int b = 0; b < batch_size_; ++b) {
            // Primal part
            hipblasZgemm(handle_,
                        HIPBLAS_OP_N, HIPBLAS_OP_N,
                        rows_, other.cols_, cols_,
                        &alpha,
                        primal_data_ + b * rows_ * cols_, rows_,
                        other.primal_data_ + b * other.rows_ * other.cols_, other.rows_,
                        &beta,
                        result.primal_data_ + b * rows_ * other.cols_, rows_);

            // Dual part
            for (int d = 0; d < dual_dim_; ++d) {
                // Primal * Dual(other)
                hipblasZgemm(handle_,
                            HIPBLAS_OP_N, HIPBLAS_OP_N,
                            rows_, other.cols_, cols_,
                            &alpha,
                            primal_data_ + b * rows_ * cols_, rows_,
                            other.dual_data_ + b * other.rows_ * other.cols_ * dual_dim_ + d * other.rows_ * other.cols_, other.rows_,
                            &beta,
                            result.dual_data_ + b * rows_ * other.cols_ * dual_dim_ + d * rows_ * other.cols_, rows_);

                // Dual(Matrix) * Primal(other)
                hipblasZgemm(handle_,
                            HIPBLAS_OP_N, HIPBLAS_OP_N,
                            rows_, other.cols_, cols_,
                            &alpha,
                            dual_data_ + b * rows_ * cols_ * dual_dim_ + d * rows_ * cols_, rows_,
                            other.primal_data_ + b * other.rows_ * other.cols_, other.rows_,
                            &alpha, // Accumulate
                            result.dual_data_ + b * rows_ * other.cols_ * dual_dim_ + d * rows_ * other.cols_, rows_);
            }
        }

        hipStreamSynchronize(stream_);
        return result;
    }


    template <typename U>
    MatrixDualDense<U> elementwiseMultiply(const MatrixDualDense<U>& other) const {
        // Validate dimensions
        if (batch_size_ != other.batch_size_ || rows_ != other.rows_ || cols_ != other.cols_ || dual_dim_ != other.dual_dim_) {
            throw std::invalid_argument("Tensor dimensions do not match for elementwise multiplication.");
        }

        // Create result tensor
        MatrixDualDense<T> result(batch_size_, rows_, cols_, dual_dim_);

        int total_primal_elements = batch_size_ * rows_ * cols_;
        int total_dual_elements = total_primal_elements * dual_dim_;

        // Elementwise multiplication for primal part
        thrust::transform(
            thrust::device_pointer_cast(primal_data_),
            thrust::device_pointer_cast(primal_data_ + total_primal_elements),
            thrust::device_pointer_cast(other.primal_data_),
            thrust::device_pointer_cast(result.primal_data_),
            thrust::multiplies<ComplexT>());

        // Elementwise multiplication for dual part using product rule
        thrust::for_each(
            thrust::counting_iterator<int>(0),
            thrust::counting_iterator<int>(total_dual_elements),
            [=] __device__(int idx) {
                int primal_idx = idx / dual_dim_;
                int dual_idx = idx % dual_dim_;

                result.dual_data_[idx] =
                    dual_data_[idx] * other.primal_data_[primal_idx] +
                    primal_data_[primal_idx] * other.dual_data_[idx];
            });

        return result;
    };

    

private:
    void initializeHandles() {
        if (hipblasCreate(&handle_) != HIPBLAS_STATUS_SUCCESS) {
            throw std::runtime_error("Failed to create cuBLAS handle.");
        }
        if (hipStreamCreate(&stream_) != hipSuccess) {
            hipblasDestroy(handle_);
            throw std::runtime_error("Failed to create CUDA stream.");
        }
        hipblasSetStream(handle_, stream_);
    }
};


template <typename T>
class MatrixHyperDualDense {
private:
    using ComplexT = std::complex<T>;

    // Dimensions
    int batch_size_;
    int rows_;
    int cols_;
    int dual_dim_;

    // Primal, Dual, and Hyper-Dual Data
    ComplexT* primal_data_;     // [M, N, L]
    ComplexT* dual_data_;       // [M, N, L, D]
    ComplexT* hyper_dual_data_; // [M, N, L, D, D]
    bool owns_memory_;          // Indicates if memory is managed internally

    // cuBLAS handle
    hipblasHandle_t handle_;
    hipStream_t stream_;

public:
    // Constructor with external memory
    MatrixHyperDualDense(int batch_size, int rows, int cols, int dual_dim,
                           ComplexT* primal_data, ComplexT* dual_data, ComplexT* hyper_dual_data)
        : batch_size_(batch_size), rows_(rows), cols_(cols), dual_dim_(dual_dim),
          primal_data_(primal_data), dual_data_(dual_data), hyper_dual_data_(hyper_dual_data),
          owns_memory_(false) {
        if (!primal_data_ || !dual_data_ || !hyper_dual_data_) {
            throw std::invalid_argument("Primal, dual, or hyper-dual data pointer is null");
        }

        initializeHandles();
    }

    // Constructor with internal memory allocation
    MatrixHyperDualDense(int batch_size, int rows, int cols, int dual_dim)
        : batch_size_(batch_size), rows_(rows), cols_(cols), dual_dim_(dual_dim), owns_memory_(true) {
        if (batch_size <= 0 || rows <= 0 || cols <= 0 || dual_dim <= 0) {
            throw std::invalid_argument("All dimensions must be positive.");
        }

        size_t primal_size = batch_size * rows * cols * sizeof(ComplexT);
        size_t dual_size = batch_size * rows * cols * dual_dim * sizeof(ComplexT);
        size_t hyper_dual_size = batch_size * rows * cols * dual_dim * dual_dim * sizeof(ComplexT);

        if (hipMalloc(&primal_data_, primal_size) != hipSuccess) {
            throw std::runtime_error("Failed to allocate GPU memory for primal data.");
        }

        if (hipMalloc(&dual_data_, dual_size) != hipSuccess) {
            hipFree(primal_data_);
            throw std::runtime_error("Failed to allocate GPU memory for dual data.");
        }

        if (hipMalloc(&hyper_dual_data_, hyper_dual_size) != hipSuccess) {
            hipFree(primal_data_);
            hipFree(dual_data_);
            throw std::runtime_error("Failed to allocate GPU memory for hyper-dual data.");
        }

        initializeHandles();
    }

    ~MatrixHyperDualDense() {
        if (owns_memory_) {
            if (primal_data_) hipFree(primal_data_);
            if (dual_data_) hipFree(dual_data_);
            if (hyper_dual_data_) hipFree(hyper_dual_data_);
        }
        hipblasDestroy(handle_);
        hipStreamDestroy(stream_);
    }

    void initialize(const ComplexT* primal, const ComplexT* dual, const ComplexT* hyper_dual,
                    size_t primal_size, size_t dual_size, size_t hyper_dual_size) {
        if (primal_size != batch_size_ * rows_ * cols_ ||
            dual_size != batch_size_ * rows_ * cols_ * dual_dim_ ||
            hyper_dual_size != batch_size_ * rows_ * cols_ * dual_dim_ * dual_dim_) {
            throw std::invalid_argument("Input sizes do not match tensor dimensions.");
        }

        if (primal) {
            hipMemcpyAsync(primal_data_, primal, primal_size * sizeof(ComplexT), hipMemcpyHostToDevice, stream_);
        }
        if (dual) {
            hipMemcpyAsync(dual_data_, dual, dual_size * sizeof(ComplexT), hipMemcpyHostToDevice, stream_);
        }
        if (hyper_dual) {
            hipMemcpyAsync(hyper_dual_data_, hyper_dual, hyper_dual_size * sizeof(ComplexT), hipMemcpyHostToDevice, stream_);
        }
        hipStreamSynchronize(stream_);
    }

    template <typename U>
    MatrixHyperDualDense<U> indexGet(int start_row, int end_row, int start_col, int end_col) const {
        // Validate row and column ranges
        if (start_row < 0 || end_row > rows_ || start_row >= end_row ||
            start_col < 0 || end_col > cols_ || start_col >= end_col) {
            throw std::invalid_argument("Invalid row or column range for indexGet.");
        }

        // Dimensions of the submatrix
        int sub_rows = end_row - start_row;
        int sub_cols = end_col - start_col;

        // Calculate offsets for the primal, dual, and hyper-dual data
        ComplexT* sub_primal_data = primal_data_ + start_row * cols_ + start_col;
        ComplexT* sub_dual_data = dual_data_ + start_row * cols_ * dual_dim_ + start_col * dual_dim_;
        ComplexT* sub_hyper_dual_data = hyper_dual_data_ +
                                        start_row * cols_ * dual_dim_ * dual_dim_ +
                                        start_col * dual_dim_ * dual_dim_;

        // Create a new MatrixHyperDualDense instance sharing the data with the original
        return MatrixHyperDualDense<T>(batch_size_, sub_rows, sub_cols, dual_dim_,
                                    sub_primal_data, sub_dual_data, sub_hyper_dual_data);
    }


    template <typename U>
    void indexPut(int start_row, int end_row, int start_col, int end_col, const MatrixHyperDualDense<U>& data) {
        // Validate row and column ranges
        if (start_row < 0 || end_row > rows_ || start_row >= end_row ||
            start_col < 0 || end_col > cols_ || start_col >= end_col) {
            throw std::invalid_argument("Invalid row or column range for indexPut.");
        }

        // Validate dimensions of the input data
        if (data.rows_ != (end_row - start_row) || data.cols_ != (end_col - start_col) ||
            data.dual_dim_ != dual_dim_ || data.batch_size_ != batch_size_) {
            throw std::invalid_argument("Input data dimensions do not match the target range.");
        }

        // Calculate offsets for the primal, dual, and hyper-dual data
        ComplexT* target_primal_data = primal_data_ + start_row * cols_ + start_col;
        ComplexT* target_dual_data = dual_data_ + start_row * cols_ * dual_dim_ + start_col * dual_dim_;
        ComplexT* target_hyper_dual_data = hyper_dual_data_ +
                                        start_row * cols_ * dual_dim_ * dual_dim_ +
                                        start_col * dual_dim_ * dual_dim_;

        // Update primal data
        for (int b = 0; b < batch_size_; ++b) {
            hipMemcpy2DAsync(target_primal_data + b * rows_ * cols_,
                            cols_ * sizeof(ComplexT),
                            data.primal_data_ + b * data.rows_ * data.cols_,
                            data.cols_ * sizeof(ComplexT),
                            data.cols_ * sizeof(ComplexT),
                            data.rows_,
                            hipMemcpyDeviceToDevice,
                            stream_);
        }

        // Update dual data
        for (int b = 0; b < batch_size_; ++b) {
            hipMemcpy2DAsync(target_dual_data + b * rows_ * cols_ * dual_dim_,
                            cols_ * dual_dim_ * sizeof(ComplexT),
                            data.dual_data_ + b * data.rows_ * data.cols_ * dual_dim_,
                            data.cols_ * dual_dim_ * sizeof(ComplexT),
                            data.cols_ * dual_dim_ * sizeof(ComplexT),
                            data.rows_,
                            hipMemcpyDeviceToDevice,
                            stream_);
        }

        // Update hyper-dual data
        for (int b = 0; b < batch_size_; ++b) {
            hipMemcpy2DAsync(target_hyper_dual_data + b * rows_ * cols_ * dual_dim_ * dual_dim_,
                            cols_ * dual_dim_ * dual_dim_ * sizeof(ComplexT),
                            data.hyper_dual_data_ + b * data.rows_ * data.cols_ * dual_dim_ * dual_dim_,
                            data.cols_ * dual_dim_ * dual_dim_ * sizeof(ComplexT),
                            data.cols_ * dual_dim_ * dual_dim_ * sizeof(ComplexT),
                            data.rows_,
                            hipMemcpyDeviceToDevice,
                            stream_);
        }

        // Synchronize to ensure data transfer is complete
        hipStreamSynchronize(stream_);
    }

    MatrixHyperDualDense<T> transpose() const {
        MatrixHyperDualDense<T> result(batch_size_, cols_, rows_, dual_dim_);
        ComplexT alpha = ComplexT(1.0, 0.0);
        ComplexT beta = ComplexT(0.0, 0.0);

        // Transpose primal part
        for (int b = 0; b < batch_size_; ++b) {
            hipblasZgeam(handle_,
                        HIPBLAS_OP_T, HIPBLAS_OP_T,
                        cols_, rows_,
                        &alpha,
                        primal_data_ + b * rows_ * cols_, rows_,
                        &beta,
                        nullptr, cols_,
                        result.primal_data_ + b * cols_ * rows_, cols_);
        }

        // Transpose dual part
        for (int b = 0; b < batch_size_; ++b) {
            for (int d = 0; d < dual_dim_; ++d) {
                hipblasZgeam(handle_,
                            HIPBLAS_OP_T, HIPBLAS_OP_T,
                            cols_, rows_,
                            &alpha,
                            dual_data_ + b * rows_ * cols_ * dual_dim_ + d * rows_ * cols_, rows_,
                            &beta,
                            nullptr, cols_,
                            result.dual_data_ + b * cols_ * rows_ * dual_dim_ + d * cols_ * rows_, cols_);
            }
        }

        // Transpose hyper-dual part
        for (int b = 0; b < batch_size_; ++b) {
            for (int d1 = 0; d1 < dual_dim_; ++d1) {
                for (int d2 = 0; d2 < dual_dim_; ++d2) {
                    hipblasZgeam(handle_,
                                HIPBLAS_OP_T, HIPBLAS_OP_T,
                                cols_, rows_,
                                &alpha,
                                hyper_dual_data_ + b * rows_ * cols_ * dual_dim_ * dual_dim_ +
                                    d1 * rows_ * cols_ * dual_dim_ + d2 * rows_ * cols_,
                                rows_,
                                &beta,
                                nullptr, cols_,
                                result.hyper_dual_data_ + b * cols_ * rows_ * dual_dim_ * dual_dim_ +
                                    d1 * cols_ * rows_ * dual_dim_ + d2 * cols_ * rows_,
                                cols_);
                }
            }
        }

        return result;
    }

    MatrixHyperDualDense<T> upperTriangular() const {
        // Create a result matrix with the same dimensions
        MatrixHyperDualDense<T> result(batch_size_, rows_, cols_, dual_dim_);

        size_t total_elements = rows_ * cols_;

        for (int b = 0; b < batch_size_; ++b) {
            // Primal data pointers for this batch
            ComplexT* batch_primal_src = primal_data_ + b * total_elements;
            ComplexT* batch_primal_dst = result.primal_data_ + b * total_elements;

            // Apply the upper triangular operation to the primal part
            thrust::for_each(
                thrust::counting_iterator<int>(0),
                thrust::counting_iterator<int>(rows_ * cols_),
                [=] __device__(int idx) {
                    int row = idx / cols_;
                    int col = idx % cols_;
                    batch_primal_dst[idx] = (row <= col) ? batch_primal_src[idx] : ComplexT(0.0, 0.0);
                });

            // Dual part is zero-initialized
            ComplexT* batch_dual_dst = result.dual_data_ + b * total_elements * dual_dim_;
            thrust::fill(thrust::device_pointer_cast(batch_dual_dst),
                        thrust::device_pointer_cast(batch_dual_dst + total_elements * dual_dim_),
                        ComplexT(0.0, 0.0));

            // Hyper-dual part is zero-initialized
            ComplexT* batch_hyper_dual_dst = result.hyper_dual_data_ + b * total_elements * dual_dim_ * dual_dim_;
            thrust::fill(thrust::device_pointer_cast(batch_hyper_dual_dst),
                        thrust::device_pointer_cast(batch_hyper_dual_dst + total_elements * dual_dim_ * dual_dim_),
                        ComplexT(0.0, 0.0));
        }

        return result;
    }

    MatrixHyperDualDense<T> lowerTriangular() const {
        // Create a result matrix with the same dimensions
        MatrixHyperDualDense<T> result(batch_size_, rows_, cols_, dual_dim_);

        size_t total_elements = rows_ * cols_;

        for (int b = 0; b < batch_size_; ++b) {
            // Primal data pointers for this batch
            ComplexT* batch_primal_src = primal_data_ + b * total_elements;
            ComplexT* batch_primal_dst = result.primal_data_ + b * total_elements;

            // Apply the lower triangular operation to the primal part
            thrust::for_each(
                thrust::counting_iterator<int>(0),
                thrust::counting_iterator<int>(rows_ * cols_),
                [=] __device__(int idx) {
                    int row = idx / cols_;
                    int col = idx % cols_;
                    batch_primal_dst[idx] = (row >= col) ? batch_primal_src[idx] : ComplexT(0.0, 0.0);
                });

            // Dual part is zero-initialized
            ComplexT* batch_dual_dst = result.dual_data_ + b * total_elements * dual_dim_;
            thrust::fill(thrust::device_pointer_cast(batch_dual_dst),
                        thrust::device_pointer_cast(batch_dual_dst + total_elements * dual_dim_),
                        ComplexT(0.0, 0.0));

            // Hyper-dual part is zero-initialized
            ComplexT* batch_hyper_dual_dst = result.hyper_dual_data_ + b * total_elements * dual_dim_ * dual_dim_;
            thrust::fill(thrust::device_pointer_cast(batch_hyper_dual_dst),
                        thrust::device_pointer_cast(batch_hyper_dual_dst + total_elements * dual_dim_ * dual_dim_),
                        ComplexT(0.0, 0.0));
        }

        return result;
    }
    template <typename U>
    MatrixHyperDualDense<U> matrixMultiply(const MatrixHyperDualDense<U>& other) const {
        // Validate dimensions
        if (cols_ != other.rows_ || batch_size_ != other.batch_size_ || dual_dim_ != other.dual_dim_) {
            throw std::invalid_argument("Matrix dimensions do not match for multiplication.");
        }

        // Create the result matrix
        MatrixHyperDualDense<T> result(batch_size_, rows_, other.cols_, dual_dim_);

        ComplexT alpha = ComplexT(1.0, 0.0);
        ComplexT beta = ComplexT(0.0, 0.0);

        for (int b = 0; b < batch_size_; ++b) {
            // Real part: Primal * Primal
            hipblasZgemm(handle_,
                        HIPBLAS_OP_N, HIPBLAS_OP_N,
                        rows_, other.cols_, cols_,
                        &alpha,
                        primal_data_ + b * rows_ * cols_, rows_,
                        other.primal_data_ + b * other.rows_ * other.cols_, other.rows_,
                        &beta,
                        result.primal_data_ + b * rows_ * other.cols_, rows_);

            // Dual part
            for (int d = 0; d < dual_dim_; ++d) {
                // Primal * Dual(other) + Dual * Primal(other)
                hipblasZgemm(handle_,
                            HIPBLAS_OP_N, HIPBLAS_OP_N,
                            rows_, other.cols_, cols_,
                            &alpha,
                            primal_data_ + b * rows_ * cols_, rows_,
                            other.dual_data_ + b * other.rows_ * other.cols_ * dual_dim_ + d * other.rows_ * other.cols_, other.rows_,
                            &beta,
                            result.dual_data_ + b * rows_ * other.cols_ * dual_dim_ + d * rows_ * other.cols_, rows_);

                hipblasZgemm(handle_,
                            HIPBLAS_OP_N, HIPBLAS_OP_N,
                            rows_, other.cols_, cols_,
                            &alpha,
                            dual_data_ + b * rows_ * cols_ * dual_dim_ + d * rows_ * cols_, rows_,
                            other.primal_data_ + b * other.rows_ * other.cols_, other.rows_,
                            &alpha, // Accumulate
                            result.dual_data_ + b * rows_ * other.cols_ * dual_dim_ + d * rows_ * other.cols_, rows_);
            }

            // Hyper-dual part
            for (int d1 = 0; d1 < dual_dim_; ++d1) {
                for (int d2 = 0; d2 < dual_dim_; ++d2) {
                    // (Primal * Hyper-Dual(other)) + (Dual * Dual(other)) + (Hyper-Dual * Primal(other))
                    hipblasZgemm(handle_,
                                HIPBLAS_OP_N, HIPBLAS_OP_N,
                                rows_, other.cols_, cols_,
                                &alpha,
                                primal_data_ + b * rows_ * cols_, rows_,
                                other.hyper_dual_data_ + b * other.rows_ * other.cols_ * dual_dim_ * dual_dim_ +
                                    d1 * other.rows_ * other.cols_ * dual_dim_ + d2 * other.rows_ * other.cols_, other.rows_,
                                &beta,
                                result.hyper_dual_data_ + b * rows_ * other.cols_ * dual_dim_ * dual_dim_ +
                                    d1 * rows_ * other.cols_ * dual_dim_ + d2 * rows_ * other.cols_, rows_);

                    hipblasZgemm(handle_,
                                HIPBLAS_OP_N, HIPBLAS_OP_N,
                                rows_, other.cols_, cols_,
                                &alpha,
                                dual_data_ + b * rows_ * cols_ * dual_dim_ + d1 * rows_ * cols_, rows_,
                                other.dual_data_ + b * other.rows_ * other.cols_ * dual_dim_ + d2 * other.rows_ * other.cols_, other.rows_,
                                &alpha, // Accumulate
                                result.hyper_dual_data_ + b * rows_ * other.cols_ * dual_dim_ * dual_dim_ +
                                    d1 * rows_ * other.cols_ * dual_dim_ + d2 * rows_ * other.cols_, rows_);

                    hipblasZgemm(handle_,
                                HIPBLAS_OP_N, HIPBLAS_OP_N,
                                rows_, other.cols_, cols_,
                                &alpha,
                                hyper_dual_data_ + b * rows_ * cols_ * dual_dim_ * dual_dim_ +
                                    d1 * rows_ * cols_ * dual_dim_ + d2 * rows_ * cols_, rows_,
                                other.primal_data_ + b * other.rows_ * other.cols_, other.rows_,
                                &alpha, // Accumulate
                                result.hyper_dual_data_ + b * rows_ * other.cols_ * dual_dim_ * dual_dim_ +
                                    d1 * rows_ * other.cols_ * dual_dim_ + d2 * rows_ * other.cols_, rows_);
                }
            }
        }

        hipStreamSynchronize(stream_);
        return result;
    }

    template <typename U>
    VectorHyperDualDense<U> matrixVectorProduct(const VectorHyperDualDense<U>& vector) const {
        // Validate dimensions
        if (cols_ != vector.size() || batch_size_ != vector.batch_size() || dual_dim_ != vector.dual_dim()) {
            throw std::invalid_argument("Matrix and vector dimensions do not match for multiplication.");
        }

        // Create the result vector
        VectorHyperDualDense<T> result(batch_size_, rows_, dual_dim_);

        ComplexT alpha = ComplexT(1.0, 0.0);
        ComplexT beta = ComplexT(0.0, 0.0);

        for (int b = 0; b < batch_size_; ++b) {
            // Real part: Primal * Primal(vector)
            hipblasZgemv(handle_,
                        HIPBLAS_OP_N,
                        rows_, cols_,
                        &alpha,
                        primal_data_ + b * rows_ * cols_, rows_,
                        vector.primal_data() + b * vector.size(), 1,
                        &beta,
                        result.primal_data() + b * rows_, 1);

            // Dual part
            for (int d = 0; d < dual_dim_; ++d) {
                // Primal * Dual(vector) + Dual * Primal(vector)
                hipblasZgemv(handle_,
                            HIPBLAS_OP_N,
                            rows_, cols_,
                            &alpha,
                            primal_data_ + b * rows_ * cols_, rows_,
                            vector.dual_data() + b * vector.size() * dual_dim_ + d * vector.size(), 1,
                            &beta,
                            result.dual_data() + b * rows_ * dual_dim_ + d * rows_, 1);

                hipblasZgemv(handle_,
                            HIPBLAS_OP_N,
                            rows_, cols_,
                            &alpha,
                            dual_data_ + b * rows_ * cols_ * dual_dim_ + d * rows_ * cols_, rows_,
                            vector.primal_data() + b * vector.size(), 1,
                            &alpha, // Accumulate
                            result.dual_data() + b * rows_ * dual_dim_ + d * rows_, 1);
            }

            // Hyper-dual part
            for (int d1 = 0; d1 < dual_dim_; ++d1) {
                for (int d2 = 0; d2 < dual_dim_; ++d2) {
                    // (Primal * Hyper-Dual(vector)) + (Dual * Dual(vector)) + (Hyper-Dual * Primal(vector))
                    hipblasZgemv(handle_,
                                HIPBLAS_OP_N,
                                rows_, cols_,
                                &alpha,
                                primal_data_ + b * rows_ * cols_, rows_,
                                vector.hyper_dual_data() + b * vector.size() * dual_dim_ * dual_dim_ +
                                    d1 * vector.size() * dual_dim_ + d2 * vector.size(), 1,
                                &beta,
                                result.hyper_dual_data() + b * rows_ * dual_dim_ * dual_dim_ +
                                    d1 * rows_ * dual_dim_ + d2 * rows_, 1);

                    hipblasZgemv(handle_,
                                HIPBLAS_OP_N,
                                rows_, cols_,
                                &alpha,
                                dual_data_ + b * rows_ * cols_ * dual_dim_ + d1 * rows_ * cols_, rows_,
                                vector.dual_data() + b * vector.size() * dual_dim_ + d2 * vector.size(), 1,
                                &alpha, // Accumulate
                                result.hyper_dual_data() + b * rows_ * dual_dim_ * dual_dim_ +
                                    d1 * rows_ * dual_dim_ + d2 * rows_, 1);

                    hipblasZgemv(handle_,
                                HIPBLAS_OP_N,
                                rows_, cols_,
                                &alpha,
                                hyper_dual_data_ + b * rows_ * cols_ * dual_dim_ * dual_dim_ +
                                    d1 * rows_ * cols_ * dual_dim_ + d2 * rows_ * cols_, rows_,
                                vector.primal_data() + b * vector.size(), 1,
                                &alpha, // Accumulate
                                result.hyper_dual_data() + b * rows_ * dual_dim_ * dual_dim_ +
                                    d1 * rows_ * dual_dim_ + d2 * rows_, 1);
                }
            }
        }

        hipStreamSynchronize(stream_);
        return result;
    }


private:
    void initializeHandles() {
        if (hipblasCreate(&handle_) != HIPBLAS_STATUS_SUCCESS) {
            throw std::runtime_error("Failed to create cuBLAS handle.");
        }
        if (hipStreamCreate(&stream_) != hipSuccess) {
            hipblasDestroy(handle_);
            throw std::runtime_error("Failed to create CUDA stream.");
        }
        hipblasSetStream(handle_, stream_);
    }

public:
    // Disable copy constructor and copy assignment
    MatrixHyperDualDense(const MatrixHyperDualDense&) = delete;
    MatrixHyperDualDense& operator=(const MatrixHyperDualDense&) = delete;

    // Enable move constructor and move assignment
    MatrixHyperDualDense(MatrixHyperDualDense&&) noexcept = default;
    MatrixHyperDualDense& operator=(MatrixHyperDualDense&&) noexcept = default;

    // Getters for dimensions
    int batch_size() const { return batch_size_; }
    int rows() const { return rows_; }
    int cols() const { return cols_; }
    int dual_dim() const { return dual_dim_; }

    // Getters for data pointers
    ComplexT* primal_data() { return primal_data_; }
    ComplexT* dual_data() { return dual_data_; }
    ComplexT* hyper_dual_data() { return hyper_dual_data_; }

    const ComplexT* primal_data() const { return primal_data_; }
    const ComplexT* dual_data() const { return dual_data_; }
    const ComplexT* hyper_dual_data() const { return hyper_dual_data_; }
}; // class MatrixHyperDualDense






}  // namespace janus
#endif // _CU_DUAL_TENSOR_HPP